#include "hip/hip_runtime.h"
/*
 * To change this license header, choose License Headers in Project Properties.
 * To change this template file, choose Tools | Templates
 * and open the template in the editor.
 */
#include "numerical.h"
#include "octree.h"
#include "mesh.h"
#include "geometry.h"
#include <hipblas.h>
#include <hip/hip_math_constants.h>
#include <hipsolver.h>
#include <hipfft/hipfft.h>
#include <float.h>
#include <string.h>
#include <stdio.h>
#include <stdlib.h>

//air density and speed of sound
__constant__ float density = 1.2041;

__constant__ float speed = 343.21;

//Integral points and weights
__constant__ float INTPT[INTORDER]; 

__constant__ float INTWGT[INTORDER];

float intpt[INTORDER];
float intwgt[INTORDER];

#ifndef NUM_EXTRAP_PER_LAUNCH
#define NUM_EXTRAP_PER_LAUNCH 1024
#endif

#ifndef REF_SOUND_PRESSURE
#define REF_SOUND_PRESSURE 0.00002
#endif

void vecd2f(const vec3d* vec, const int len, vec3f* vecf)
{
    for(int i=0;i<len;i++) {
        for(int j=0;j<3;j++) {
            vecf[i].coords[j] = vec[i].coords[j];
        }
    }
}

int genGaussParams(const int n, float* pt, float* wgt) 
{
    int i, j;
    double t;
    gsl_vector *v = gsl_vector_alloc(n);
    for(i=0;i<n-1;i++) {
        gsl_vector_set(v,i,sqrt(pow(2*(i+1),2)-1));
    }
    for(i=0;i<n-1;i++) {
        t = gsl_vector_get(v,i);
        gsl_vector_set(v,i,(i+1)/t);
    }
    gsl_matrix *A = gsl_matrix_alloc(n,n);
    gsl_matrix *B = gsl_matrix_alloc(n,n);
    for(i=0;i<n;i++) {
        for(j=0;j<n;j++) {
            gsl_matrix_set(A,i,j,0);
            if(i==j) {
                gsl_matrix_set(B,i,j,1);
            } else {
                gsl_matrix_set(B,i,j,0);
            }
        }
    }
    for(i=0;i<n-1;i++) {
        t = gsl_vector_get(v,i);
        gsl_matrix_set(A,i+1,i,t);
        gsl_matrix_set(A,i,i+1,t);
    }
    gsl_eigen_symmv_workspace * wsp = gsl_eigen_symmv_alloc(n);
    HOST_CALL(gsl_eigen_symmv(A,v,B,wsp));
    for(i=0;i<n;i++) {
        pt[i] = gsl_vector_get(v,i);
        t = gsl_matrix_get(B,0,i);
        wgt[i] = 2*pow(t,2);
    }
    gsl_vector_free(v);
    gsl_matrix_free(A);
    gsl_matrix_free(B);
    return EXIT_SUCCESS;
}

int cuGenGaussParams(const int n, float* pt, float* wgt)
{
    hipsolverHandle_t handle;
    CUSOLVER_CALL(hipsolverDnCreate(&handle));
    
    // allocate memory for vector v of length n
    float *v = (float*)malloc(n*sizeof(float));
    
    // set the vector v
    for(int i=0;i<n-1;i++) {
        v[i] = sqrt(pow(2*(i+1),2)-1);
    }
    for(int i=0;i<n-1;i++) {
        float t = v[i];
        v[i] = (i+1)/t;
    }
    //printf("The vector v is set properly.\n");
    
    float *A = (float*)malloc(n*n*sizeof(float));
    memset(A,0,n*n*sizeof(float));
    
    // set up matrix A
    for(int i=0;i<n-1;i++) {
        float t = v[i];
        A[IDXC0(i+1,i,n)] = t;
        A[IDXC0(i,i+1,n)] = t;
    }
    
    //printf("The matrix A is set properly.\n");
    
    float *A_d, *Lambda_d;
    CUDA_CALL(hipMalloc(&A_d,n*n*sizeof(float)));
    //printf("A_d allocated.\n");
    CUDA_CALL(hipMemcpy(A_d,A,n*n*sizeof(float),hipMemcpyHostToDevice));
    //printf("A copied to A_d.\n");
    CUDA_CALL(hipMalloc(&Lambda_d,n*sizeof(float)));
    //printf("Lambda_d allocated successfully.\n");
    
    int lwork;
    hipsolverEigMode_t jobz = HIPSOLVER_EIG_MODE_VECTOR; // compute eigenvalues and eigenvectors.
    hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;
    CUSOLVER_CALL(hipsolverDnSsyevd_bufferSize(handle,jobz,
            uplo,n,A_d,n,Lambda_d,&lwork));
    //printf("Buffer is set up.\n");
    float *work_d;
    CUDA_CALL(hipMalloc(&work_d,lwork*sizeof(float)));
    int *devInfo;
    CUDA_CALL(hipMalloc(&devInfo,sizeof(int)));
    CUSOLVER_CALL(hipsolverDnSsyevd(handle,jobz,uplo,n,A_d,n,Lambda_d,work_d,lwork,devInfo));
    //printf("Eigenvalues and eigenvectors found.\n");
    float *Lambda = (float*)malloc(n*sizeof(float));
    CUDA_CALL(hipMemcpy(A,A_d,n*n*sizeof(float),hipMemcpyDeviceToHost));
    CUDA_CALL(hipMemcpy(Lambda,Lambda_d,n*sizeof(float),hipMemcpyDeviceToHost));
    
    memcpy(pt,Lambda,n*sizeof(float));
    for(int i=0;i<n;i++) {
        float t = A[IDXC0(0,i,n)];
        wgt[i] = 2*pow(t,2);
    }
    
    if(A_d) {
        CUDA_CALL(hipFree(A_d));
    }
    if(Lambda_d) {
        CUDA_CALL(hipFree(Lambda_d));
    }
    if(work_d) {
        CUDA_CALL(hipFree(work_d));
    }
    if(devInfo) {
        CUDA_CALL(hipFree(devInfo));
    }
    if(handle) {
        CUSOLVER_CALL(hipsolverDnDestroy(handle));
    }
    
    free(v);
    free(Lambda);
    free(A);
    
    
    return EXIT_SUCCESS;
}

int gaussPtsToDevice(const float *evalPt, const float *wgt) 
{
    CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(INTPT),evalPt,INTORDER*sizeof(float),0,hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(INTWGT),wgt,INTORDER*sizeof(float),0,hipMemcpyHostToDevice));
    return EXIT_SUCCESS;
}

void printMat(const float* mat, const int numRow, const int numCol, const int lda)
{
    for(int i=0;i<numRow;i++) {
        for(int j=0;j<numCol;j++) {
            printf("%f ",mat[IDXC0(i,j,lda)]);
        }
        printf("\n");
    }
}

void printMat(const double* mat, const int numRow, const int numCol, const int lda)
{
    for(int i=0;i<numRow;i++) {
        for(int j=0;j<numCol;j++) {
            printf("%lf ",mat[IDXC0(i,j,lda)]);
        }
        printf("\n");
    }
}

void matRowSwap(double* mat, const int numCol, const int lda, const int i, const int j)
{
    /*switches the ith and the jth row of a matrix mat*/
    if(i!=j) {
        double temp;
        for(int idx=0;idx<numCol;idx++) {
            temp = mat[IDXC0(i,idx,lda)];
            mat[IDXC0(i,idx,lda)] = mat[IDXC0(j,idx,lda)];
            mat[IDXC0(j,idx,lda)] = temp;
        }
    }
    
}

void scaRowMul(double* mat, const int numCol, const int lda, const int ridx, const double c)
{
    for(int i=0;i<numCol;i++) {
        mat[IDXC0(ridx,i,lda)] = c*mat[IDXC0(ridx,i,lda)];
    }
}

void subScaRowFromRow(double* mat, const int numCol, const int lda, const int i, const int j, const double c)
{
    /*subtract row j from row i*/
    for(int idx=0;idx<numCol;idx++) {
        mat[IDXC0(i,idx,lda)] = mat[IDXC0(i,idx,lda)]-c*mat[IDXC0(j,idx,lda)];
    }
}

__host__ __device__ void findVecMaxAbs(const double* vec, const int idx_min, const int idx_max, double& val, int& idx)
{
    idx = idx_min;
    val = abs(vec[idx_min]);
    for(int i=idx_min+1;i<=idx_max;i++) {
        if(abs(vec[i]) > val) {
            val = abs(vec[i]);
            idx = i;
        }
    }
        
}
void GaussElim(double* mat, const int numRow, const int numCol, const int lda)
{
    double temp;
    int pr=0, pc=0, idx; //index for pivot row and pivot column
    while(pr<numRow && pc<numCol) {
        findVecMaxAbs(&mat[IDXC0(0,pc,lda)],pr,numRow-1,temp,idx);
        if(temp<EPS) { //no pivot point in the current column
            pc++;
        }
        else { //idx is the row index for the row of interest
            double coeff;
            matRowSwap(mat,numCol,lda,pr,idx);
            printf("Swapped rows: \n");
            printMat(mat,numRow,numCol,lda);
            for(int i=pr+1;i<numRow;i++) {
                coeff = mat[IDXC0(i,pc,lda)]/mat[IDXC0(pr,pc,lda)];
                mat[IDXC0(i,pc,lda)] = 0.0;
                for(int j=pc+1;j<numCol;j++) {
                    mat[IDXC0(i,j,lda)] = mat[IDXC0(i,j,lda)]-coeff*mat[IDXC0(pr,j,lda)];
                }
            }
            printf("Subtracted: \n");
            printMat(mat,numRow,numCol,lda);
            printf("\n");
            pr++;
            pc++;
        }
    }
}

void print_float_mat(const float *A, const int numRow, const int numCol, const int lda) 
{
    for(int i=0;i<numRow;i++) {
        for(int j=0;j<numCol;j++) {
            printf("%f ",A[IDXC0(i,j,lda)]);
        }
        printf("\n");
    }
}

void print_cuFloatComplex_mat(const hipFloatComplex *A, const int numRow, const int numCol, 
        const int lda)
{
    for(int i=0;i<numRow;i++) {
        for(int j=0;j<numCol;j++) {
            printf("(%f,%f) ",hipCrealf(A[IDXC0(i,j,lda)]),hipCimagf(A[IDXC0(i,j,lda)]));
        }
        printf("\n");
    }
}

__host__ __device__ void printVec(const vec3f* pt, const int numPt)
{
    for(int i=0;i<numPt;i++) {
        printf("(%f,%f,%f), ",pt[i].coords[0],pt[i].coords[1],pt[i].coords[2]);
    }
    printf("\n");
}

__host__ __device__ void printVec(const vec3d* pt, const int numPt)
{
    for(int i=0;i<numPt;i++) {
        printf("(%f,%f,%f), ",pt[i].coords[0],pt[i].coords[1],pt[i].coords[2]);
    }
    printf("\n");
}

__host__ __device__ float vecDotMul(const vec3f& u, const vec3f& v)
{
    return u.coords[0]*v.coords[0]+u.coords[1]*v.coords[1]+u.coords[2]*v.coords[2];
}

__host__ __device__ double vecDotMul(const vec3d& u, const vec3d& v)
{
    return u.coords[0]*v.coords[0]+u.coords[1]*v.coords[1]+u.coords[2]*v.coords[2];
}

__host__ __device__ float vecDotMul(const vec2f& u, const vec2f& v)
{
    return u.coords[0]*v.coords[0]+u.coords[1]+v.coords[1];
}

__host__ __device__ double vecDotMul(const vec2d& u, const vec2d& v)
{
    return u.coords[0]*v.coords[0]+u.coords[1]+v.coords[1];
}

__host__ __device__ float vecNorm(const vec3f& v)
{
    return sqrtf(vecDotMul(v,v));
}

__host__ __device__ double vecNorm(const vec3d& v)
{
    return sqrt(vecDotMul(v,v));
}

__host__ __device__ vec3f vecCrossMul(const vec3f& a, const vec3f& b)
{
    vec3f temp;
    temp.coords[0] = a.coords[1]*b.coords[2]-a.coords[2]*b.coords[1];
    temp.coords[1] = -(a.coords[0]*b.coords[2]-a.coords[2]*b.coords[0]);
    temp.coords[2] = a.coords[0]*b.coords[1]-a.coords[1]*b.coords[0];
    return temp;
}

__host__ __device__ vec3d vecCrossMul(const vec3d& a, const vec3d& b)
{
    vec3d temp;
    temp.coords[0] = a.coords[1]*b.coords[2]-a.coords[2]*b.coords[1];
    temp.coords[1] = -(a.coords[0]*b.coords[2]-a.coords[2]*b.coords[0]);
    temp.coords[2] = a.coords[0]*b.coords[1]-a.coords[1]*b.coords[0];
    return temp;
}

__host__ __device__ vec3d vecNrmlz(const vec3d& v)
{
    double nrm = sqrt(vecDotMul(v,v));
    return scaVecMul(1.0/nrm,v);
}

__host__ __device__ vec3f vecNrmlz(const vec3f& v)
{
    float nrm = sqrt(vecDotMul(v,v));
    return scaVecMul(1.0/nrm,v);
}

__host__ __device__ int vecEqual(const vec3f& v1, const vec3f& v2)
{
    vec3f v = vecSub(v1,v2);
    if(vecNorm(v) < EPS) {
        return 1;
    } else {
        return 0;
    }
}

__host__ __device__ int vecEqual(const vec3d& v1, const vec3d& v2)
{
    vec3d v = vecSub(v1,v2);
    if(vecNorm(v) < EPS) {
        return 1;
    } else {
        return 0;
    }
}

__host__ __device__ vec3f scaVecMul(const float& lambda, const vec3f& v)
{
    vec3f result;
    for(int i=0;i<3;i++) {
        result.coords[i] = lambda*v.coords[i];
    }
    return result;
}

__host__ __device__ vec3d scaVecMul(const double& lambda, const vec3d& v)
{
    vec3d result;
    for(int i=0;i<3;i++) {
        result.coords[i] = lambda*v.coords[i];
    }
    return result;
}

__host__ __device__ vec2d scaVecMul(const double& lambda, const vec2d& v)
{
    vec2d result;
    for(int i=0;i<2;i++) {
        result.coords[i] = lambda*v.coords[i];
    }
    return result;
}

__host__ __device__ vec2f scaVecMul(const float& lambda, const vec2f& v)
{
    vec2f result;
    for(int i=0;i<2;i++) {
        result.coords[i] = lambda*v.coords[i];
    }
    return result;
}

__host__ __device__ vec3f vecAdd(const vec3f& u, const vec3f& v)
{
    vec3f result;
    for(int i=0;i<3;i++) {
        result.coords[i] = u.coords[i]+v.coords[i];
    }
    return result;
}

__host__ __device__ vec3d vecAdd(const vec3d& u, const vec3d& v)
{
    vec3d result;
    for(int i=0;i<3;i++) {
        result.coords[i] = u.coords[i]+v.coords[i];
    }
    return result;
}

__host__ __device__ vec2d vecAdd(const vec2d& u, const vec2d& v)
{
    vec2d result;
    for(int i=0;i<2;i++) {
        result.coords[i] = u.coords[i]+v.coords[i];
    }
    return result;
}

__host__ __device__ vec2f vecAdd(const vec2f& u, const vec2f& v)
{
    vec2f result;
    for(int i=0;i<2;i++) {
        result.coords[i] = u.coords[i]+v.coords[i];
    }
    return result;
}

__host__ __device__ vec3d vecSub(const vec3d& u, const vec3d& v)
{
    vec3d result;
    for(int i=0;i<3;i++) {
        result.coords[i] = u.coords[i]-v.coords[i];
    }
    return result;
}

__host__ __device__ vec3f vecSub(const vec3f& u, const vec3f& v)
{
    vec3f result;
    for(int i=0;i<3;i++) {
        result.coords[i] = u.coords[i]-v.coords[i];
    }
    return result;
}

__host__ __device__ vec2d vecSub(const vec2d& u, const vec2d& v)
{
    vec2d result;
    for(int i=0;i<2;i++) {
        result.coords[i] = u.coords[i]-v.coords[i];
    }
    return result;
}

__host__ __device__ vec2f vecSub(const vec2f& u, const vec2f& v)
{
    vec2f result;
    for(int i=0;i<2;i++) {
        result.coords[i] = u.coords[i]-v.coords[i];
    }
    return result;
}

__host__ __device__ vec3d triCentroid(vec3d nod[3])
{
    vec3d ctr_23 = scaVecMul(0.5,vecAdd(nod[1],nod[2]));
    vec3d centroid = vecAdd(nod[0],scaVecMul(2.0/3.0,vecSub(ctr_23,nod[0])));
    return centroid;
}

__host__ __device__ bool ray_intersect_triangle(const vec3f O, const vec3f dir, 
        const vec3f nod[3])
{
    /*vert0 is chosen as reference point*/
    vec3f E1, E2;
    E1 = vecSub(nod[1],nod[0]);
    E2 = vecSub(nod[2],nod[0]);
    /*cross product of dir and v0 to v1*/
    vec3f P = vecCrossMul(dir,E2);
    float det = vecDotMul(P,E1);
    if(abs(det)<EPS) {
        return false;
    }
    /*Computation of parameter u*/
    vec3f T = vecSub(O,nod[0]);
    float u = 1.0f/det*vecDotMul(P,T);
    if(u<0 || u>1) {
        return false;
    }
    /*Computation of parameter v*/
    vec3f Q = vecCrossMul(T,E1);
    float v = 1.0f/det*vecDotMul(Q,dir);
    if(v<0 || u+v>1) {
        return false;
    }
    /*Computation of parameter t*/
    float t = 1.0f/det*vecDotMul(Q,E2);
    if(t<EPS) {
        return false;
    }
    return true;
}

__global__ void rayTrisInt(const vec3f pt_s, const vec3f dir, const vec3f *nod, 
        const tri_elem *elem, const int numElem, bool *flag)
{
    // decides if a point pnt is in a closed surface elem
    int idx = blockDim.x*blockIdx.x+threadIdx.x;
    if(idx<numElem) {
        vec3f pt[3];
        for(int i=0;i<3;i++) {
            pt[i].coords[0] = nod[elem[idx].nod[i]].coords[0];
            pt[i].coords[1] = nod[elem[idx].nod[i]].coords[1];
            pt[i].coords[2] = nod[elem[idx].nod[i]].coords[2];
        }
        flag[idx] = ray_intersect_triangle(pt_s,dir,pt);
    }
}

__global__ void distPntPnts(const vec3f pt, const vec3f *nod, const int numNod, float *dist) {
    int idx = blockDim.x*blockIdx.x + threadIdx.x;
    if(idx < numNod) {
        dist[idx] = __fsqrt_rn((pt.coords[0]-nod[idx].coords[0])*(pt.coords[0]-nod[idx].coords[0])
                +(pt.coords[1]-nod[idx].coords[1])*(pt.coords[1]-nod[idx].coords[1])
                +(pt.coords[2]-nod[idx].coords[2])*(pt.coords[2]-nod[idx].coords[2]));
    }
}

__host__ __device__ float convRand(const float lb, const float ub, const float randNumber) {
    float result = (ub-lb)*randNumber+lb;
    return result;
}

bool inBdry(const bool *flag, const int numFlag) {
    int sum = 0;;
    for(int i=0;i<numFlag;i++) {
        if(flag[i]) {
            sum++;
        }
    }
    if(sum%2==0) {
        return false;
    } else {
        return true;
    }
}

int genCHIEF(const vec3f* pt, const int numPt, const tri_elem* elem, const int numElem, 
        vec3f* pCHIEF, const int numCHIEF) {
    int i, cnt;
    float *dist_h = (float*)malloc(numPt*sizeof(float));
    float minDist; //minimum distance between the chief point to all surface nod
    float *dist_d;
    CUDA_CALL(hipMalloc((void**)&dist_d, numPt*sizeof(float)));
    vec3f dir; 
    
    //transfer the point cloud to GPU
    vec3f *pt_d;
    CUDA_CALL(hipMalloc((void**)&pt_d,numPt*sizeof(vec3f))); //point cloud allocated on device
    CUDA_CALL(hipMemcpy(pt_d,pt,numPt*sizeof(vec3f),hipMemcpyHostToDevice)); //point cloud copied to device
    
    //transfer the element cloud to GPU
    tri_elem *elem_d;
    CUDA_CALL(hipMalloc((void**)&elem_d,numElem*sizeof(tri_elem))); //elements allcoated on device
    CUDA_CALL(hipMemcpy(elem_d,elem,numElem*sizeof(tri_elem),hipMemcpyHostToDevice)); //elements copied to device
    
    //create a flag array on CPU and on GPU
    bool *flag_h = (bool*)malloc(numElem*sizeof(bool));
    bool *flag_d;
    CUDA_CALL(hipMalloc((void**)&flag_d,numElem*sizeof(bool))); //memory for flags allocated on device

    unsigned long long seed = 0;
    int blockWidth = 32;
    int gridWidth;
    float xrand, yrand, zrand, unifRandNum[3];
    vec3f chief;
    
    //Find the bounding box
    float xb[2], yb[2], zb[2];
    findBB(pt,numPt,0,xb,yb,zb);
    float threshold_inner = 0.01*min(min(xb[1]-xb[0],yb[1]-yb[0]),zb[1]-zb[0]);
    
    //create a handle to hiprand
    hiprandGenerator_t gen;
    CURAND_CALL(hiprandCreateGeneratorHost(&gen,HIPRAND_RNG_PSEUDO_DEFAULT)); //construct generator
    cnt = 0; // initialize count for number of points generated
    while(cnt<numCHIEF) {
        do
        {
            //set seed
            CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(gen,seed++));
            CURAND_CALL(hiprandGenerateUniform(gen,unifRandNum,3)); //generate a uniformly distributed random number
            //generate the direction
            for(i=0;i<3;i++) {
                dir.coords[i] = unifRandNum[i];
            }
            //Convert the rand numbers into a point in the bounding box
            xrand = convRand(xb[0],xb[1],unifRandNum[0]);
            yrand = convRand(yb[0],yb[1],unifRandNum[1]);
            zrand = convRand(zb[0],zb[1],unifRandNum[2]);
            chief.coords[0] = xrand;
            chief.coords[1] = yrand;
            chief.coords[2] = zrand;
            //(&chief,1);
            gridWidth = (numElem+blockWidth-1)/blockWidth;
            rayTrisInt<<<gridWidth,blockWidth>>>(chief,dir,pt_d,elem_d,numElem,flag_d);
            gridWidth = (numPt+blockWidth-1)/blockWidth;
            distPntPnts<<<gridWidth,blockWidth>>>(chief,pt_d,numPt,dist_d);
            CUDA_CALL(hipMemcpy(dist_h,dist_d,numPt*sizeof(float),hipMemcpyDeviceToHost));
            //printFltMat(dist_h,1,numPt,1);
            CUDA_CALL(hipMemcpy(flag_h,flag_d,numElem*sizeof(bool),hipMemcpyDeviceToHost));
            minDist = dist_h[0];
            for(i=1;i<numPt;i++) {
                if(dist_h[i]<minDist) {
                    minDist = dist_h[i];
                }
            }
            //printf("The minimum distance is %f, threshold is %f\n",dist_min,threshold_inner);
            //printf("inSurf: %d\n", inSurf(flags_h, numElem));
        } while (!inBdry(flag_h,numElem) || minDist<threshold_inner);
        pCHIEF[cnt] = chief;
        cnt++;
    }
    CURAND_CALL(hiprandDestroyGenerator(gen));
    free(flag_h);
    free(dist_h);
    CUDA_CALL(hipFree(pt_d));
    CUDA_CALL(hipFree(elem_d));
    CUDA_CALL(hipFree(flag_d));
    CUDA_CALL(hipFree(dist_d));
    
    //printPts(pCHIEF,numCHIEF);
    
    return EXIT_SUCCESS;
}

inline __device__ void crossNorm(const vec3f a, const vec3f b, vec3f *norm, float *length) 
{
    vec3f c;
    c.coords[0] = a.coords[1]*b.coords[2]-a.coords[2]*b.coords[1];
    c.coords[1] = a.coords[2]*b.coords[0]-a.coords[0]*b.coords[2];
    c.coords[2] = a.coords[0]*b.coords[1]-a.coords[1]*b.coords[0];

    *length = __fsqrt_rn((c.coords[0]*c.coords[0])+(c.coords[1]*c.coords[1])+(c.coords[2]*c.coords[2]));

    norm->coords[0] = c.coords[0] / *length;
    norm->coords[1] = c.coords[1] / *length;
    norm->coords[2] = c.coords[2] / *length;
}

__device__ void g_h_c_nsgl(const float k, const vec3f x, const vec3f p[3], 
        hipFloatComplex gCoeff[3], hipFloatComplex hCoeff[3], float *cCoeff) {
    //Initalization of g, h and c
    //printf("(%f,%f,%f)\n",p[0].coords[0],p[0].coords[1],p[0].coords[2]);
    for(int i=0;i<3;i++) {
        gCoeff[i] = make_hipFloatComplex(0,0);
        hCoeff[i] = make_hipFloatComplex(0,0);
    }
    *cCoeff = 0;
    
    //Local variables
    float eta1, eta2, wn, wm, xi1, xi2, xi3, rho, theta, vertCrossProd, temp, 
            temp_gh[3], omega = k*speed, pPsiLpn2, radius, prpn2;
    vec3f y, normal, rVec;
    hipFloatComplex Psi, pPsipn2;
    crossNorm(
    {
        p[0].coords[0]-p[2].coords[0],p[0].coords[1]-p[2].coords[1],p[0].coords[2]-p[2].coords[2]
    },
    {
        p[1].coords[0]-p[2].coords[0],p[1].coords[1]-p[2].coords[1],p[1].coords[2]-p[2].coords[2]
    },&normal,&vertCrossProd);
    vertCrossProd = vertCrossProd*0.25f;
    //printf("%f\n",normal.coords[0]);
    const float prodRhoOmega = density*omega;
    const float fourPI = 4.0f*PI;
    const float recipFourPI = 1.0f/fourPI;
    //printf("%f\n",k);
    for(int n=0;n<INTORDER;n++) {
        eta2 = INTPT[n];
        wn = INTWGT[n];
        theta = 0.5f+0.5f*eta2;
        for(int m=0;m<INTORDER;m++) {
            eta1 = INTPT[m];
            wm = INTWGT[m];
            rho = 0.5f+0.5f*eta1;
            temp = wn*wm*rho*vertCrossProd;
            
            xi1 = rho*(1-theta);
            xi2 = rho-xi1;
            xi3 = 1-xi1-xi2;
            //printf("xi1 = %f, xi2 = %f\n",xi1,xi2);
            y= {
                p[0].coords[0]*xi1+p[1].coords[0]*xi2+p[2].coords[0]*xi3, 
                p[0].coords[1]*xi1+p[1].coords[1]*xi2+p[2].coords[1]*xi3, 
                p[0].coords[2]*xi1+p[1].coords[2]*xi2+p[2].coords[2]*xi3
            };
            //printf("x: (%f,%f,%f), y: (%f,%f,%f)\n",x.coords[0],x.coords[1],x.coords[2],
            //        y.coords[0],y.coords[1],y.coords[2]);
            rVec = vecSub(y,x);
            radius = __fsqrt_rn(rVec.coords[0]*rVec.coords[0]+rVec.coords[1]*rVec.coords[1]
                    +rVec.coords[2]*rVec.coords[2]);
            //printf("radius = %f\n",radius);
            prpn2 = ((y.coords[0]-x.coords[0])*normal.coords[0]+(y.coords[1]-x.coords[1])*normal.coords[1]
                    +(y.coords[2]-x.coords[2])*normal.coords[2])/radius;
            //printf("prpn2=%f\n",prpn2);
            pPsiLpn2 = -recipFourPI/(radius*radius)*prpn2;
            //printf("%f\n",pPsiLpn2);
            Psi = make_hipFloatComplex(__cosf(-k*radius)/(fourPI*radius),__sinf(-k*radius)/(fourPI*radius));
            pPsipn2 = hipCmulf(Psi,make_hipFloatComplex(-1.0f/radius,-k));
            pPsipn2 = make_hipFloatComplex(prpn2*hipCrealf(pPsipn2),prpn2*hipCimagf(pPsipn2));
            temp_gh[0] = temp*xi1;
            temp_gh[1] = temp*xi2;
            temp_gh[2] = temp*xi3;
            
            gCoeff[0] = hipCaddf(gCoeff[0],make_hipFloatComplex(temp_gh[0]*hipCrealf(Psi),temp_gh[0]*hipCimagf(Psi)));
            gCoeff[1] = hipCaddf(gCoeff[1],make_hipFloatComplex(temp_gh[1]*hipCrealf(Psi),temp_gh[1]*hipCimagf(Psi)));
            gCoeff[2] = hipCaddf(gCoeff[2],make_hipFloatComplex(temp_gh[2]*hipCrealf(Psi),temp_gh[2]*hipCimagf(Psi)));
            
            hCoeff[0] = hipCaddf(hCoeff[0],make_hipFloatComplex(temp_gh[0]*hipCrealf(pPsipn2),temp_gh[0]*hipCimagf(pPsipn2)));
            hCoeff[1] = hipCaddf(hCoeff[1],make_hipFloatComplex(temp_gh[1]*hipCrealf(pPsipn2),temp_gh[1]*hipCimagf(pPsipn2)));
            hCoeff[2] = hipCaddf(hCoeff[2],make_hipFloatComplex(temp_gh[2]*hipCrealf(pPsipn2),temp_gh[2]*hipCimagf(pPsipn2)));
            
            *cCoeff += temp*pPsiLpn2;
        }
    }
    gCoeff[0] = make_hipFloatComplex(-prodRhoOmega*hipCimagf(gCoeff[0]),prodRhoOmega*hipCrealf(gCoeff[0]));
    gCoeff[1] = make_hipFloatComplex(-prodRhoOmega*hipCimagf(gCoeff[1]),prodRhoOmega*hipCrealf(gCoeff[1]));
    gCoeff[2] = make_hipFloatComplex(-prodRhoOmega*hipCimagf(gCoeff[2]),prodRhoOmega*hipCrealf(gCoeff[2]));
}

__device__ void g_h_c_sgl(const float k, const vec3f x_sgl1, const vec3f x_sgl2, 
        const vec3f x_sgl3, const vec3f p[3], 
        hipFloatComplex gCoeff_sgl1[3], hipFloatComplex hCoeff_sgl1[3], float *cCoeff_sgl1,
        hipFloatComplex gCoeff_sgl2[3], hipFloatComplex hCoeff_sgl2[3], float *cCoeff_sgl2,
        hipFloatComplex gCoeff_sgl3[3], hipFloatComplex hCoeff_sgl3[3], float *cCoeff_sgl3) 
{
    //Initalization of g, h and c
    for(int i=0;i<3;i++) {
        gCoeff_sgl1[i] = make_hipFloatComplex(0,0);
        hCoeff_sgl1[i] = make_hipFloatComplex(0,0);
        gCoeff_sgl2[i] = make_hipFloatComplex(0,0);
        hCoeff_sgl2[i] = make_hipFloatComplex(0,0);
        gCoeff_sgl3[i] = make_hipFloatComplex(0,0);
        hCoeff_sgl3[i] = make_hipFloatComplex(0,0);
    }
    *cCoeff_sgl1 = 0;
    *cCoeff_sgl2 = 0;
    *cCoeff_sgl3 = 0;
    
    //Local variables
    float eta1, eta2, wn, wm, xi1_sgl1, xi2_sgl1, xi3_sgl1, xi1_sgl2, xi2_sgl2, xi3_sgl2,
            xi1_sgl3, xi2_sgl3, xi3_sgl3, rho, theta, vertCrossProd, temp, 
            temp_gh[3], omega = k*speed, pPsiLpn2, radius, prpn2;
    vec3f y_sgl1, y_sgl2, y_sgl3, normal, rVec;
    hipFloatComplex Psi, pPsipn2;
    crossNorm(
    {
        p[0].coords[0]-p[2].coords[0],p[0].coords[1]-p[2].coords[1],p[0].coords[2]-p[2].coords[2]
    },
    {
        p[1].coords[0]-p[2].coords[0],p[1].coords[1]-p[2].coords[1],p[1].coords[2]-p[2].coords[2]
    },&normal,&vertCrossProd);
    vertCrossProd = vertCrossProd*0.25f;
    //printf("vert: %f\n",vertCrossProd);
    
    //printf("normal=(%f,%f,%f)\n",normal.coords[0],normal.coords[1],normal.coords[2]);
    const float prodRhoOmega = density*omega;
    const float fourPI = 4.0f*PI;
    const float recipFourPI = 1.0/fourPI;
    //printf("density*omega = %f\n",prodRhoOmega);
    //Compute integrals for g, h and c
    for(int n=0;n<INTORDER;n++) {
        eta2 = INTPT[n];
        wn = INTWGT[n];
        theta = 0.5f+0.5f*eta2;
        for(int m=0;m<INTORDER;m++) {
            eta1 = INTPT[m];
            wm = INTWGT[m];
            rho = 0.5f+0.5f*eta1;
            temp = wn*wm*rho*vertCrossProd;
            
            xi1_sgl3 = rho*(1-theta);
            xi2_sgl3 = rho-xi1_sgl3; //rho*theta
            xi3_sgl3 = 1-xi1_sgl3-xi2_sgl3;
            
            xi1_sgl1 = 1-rho;
            xi2_sgl1 = rho-xi2_sgl3; //rho-rho*theta
            xi3_sgl1 = 1-xi1_sgl1-xi2_sgl1;
            
            xi1_sgl2 = xi2_sgl3; //rho*theta
            xi2_sgl2 = 1-rho;
            xi3_sgl2 = 1-xi1_sgl2-xi2_sgl2;
            
            
            
            //printf("xi1 = %f, xi2 = %f\n",xi1,xi2);
            y_sgl1= {
                p[0].coords[0]*xi1_sgl1+p[1].coords[0]*xi2_sgl1+p[2].coords[0]*xi3_sgl1, 
                p[0].coords[1]*xi1_sgl1+p[1].coords[1]*xi2_sgl1+p[2].coords[1]*xi3_sgl1, 
                p[0].coords[2]*xi1_sgl1+p[1].coords[2]*xi2_sgl1+p[2].coords[2]*xi3_sgl1
            };
            y_sgl2= {
                p[0].coords[0]*xi1_sgl2+p[1].coords[0]*xi2_sgl2+p[2].coords[0]*xi3_sgl2, 
                p[0].coords[1]*xi1_sgl2+p[1].coords[1]*xi2_sgl2+p[2].coords[1]*xi3_sgl2, 
                p[0].coords[2]*xi1_sgl2+p[1].coords[2]*xi2_sgl2+p[2].coords[2]*xi3_sgl2
            };
            y_sgl3= {
                p[0].coords[0]*xi1_sgl3+p[1].coords[0]*xi2_sgl3+p[2].coords[0]*xi3_sgl3, 
                p[0].coords[1]*xi1_sgl3+p[1].coords[1]*xi2_sgl3+p[2].coords[1]*xi3_sgl3, 
                p[0].coords[2]*xi1_sgl3+p[1].coords[2]*xi2_sgl3+p[2].coords[2]*xi3_sgl3
            };
            
            //update coefficients with singularity on node 1
            rVec = vecSub(y_sgl1,x_sgl1);
            radius = sqrtf(rVec.coords[0]*rVec.coords[0]+rVec.coords[1]*rVec.coords[1]+rVec.coords[2]*rVec.coords[2]);
            //printf("radius = %f\n",radius);
            prpn2 = ((y_sgl1.coords[0]-x_sgl1.coords[0])*normal.coords[0]+(y_sgl1.coords[1]-x_sgl1.coords[1])*normal.coords[1]
                    +(y_sgl1.coords[2]-x_sgl1.coords[2])*normal.coords[2])/radius;
            //printf("prpn2=%f\n",prpn2);
            pPsiLpn2 = -recipFourPI/(radius*radius)*prpn2;
            //printf("%f\n",pPsiLpn2);
            Psi = make_hipFloatComplex(__cosf(-k*radius)/(fourPI*radius),__sinf(-k*radius)/(fourPI*radius));
            pPsipn2 = hipCmulf(Psi,make_hipFloatComplex(-1.0f/radius,-k));
            pPsipn2 = make_hipFloatComplex(prpn2*hipCrealf(pPsipn2),prpn2*hipCimagf(pPsipn2));
            temp_gh[0] = temp*xi1_sgl1;
            temp_gh[1] = temp*xi2_sgl1;
            temp_gh[2] = temp*xi3_sgl1;
            
            gCoeff_sgl1[0] = hipCaddf(gCoeff_sgl1[0],make_hipFloatComplex(temp_gh[0]*hipCrealf(Psi),temp_gh[0]*hipCimagf(Psi)));
            gCoeff_sgl1[1] = hipCaddf(gCoeff_sgl1[1],make_hipFloatComplex(temp_gh[1]*hipCrealf(Psi),temp_gh[1]*hipCimagf(Psi)));
            gCoeff_sgl1[2] = hipCaddf(gCoeff_sgl1[2],make_hipFloatComplex(temp_gh[2]*hipCrealf(Psi),temp_gh[2]*hipCimagf(Psi)));
            
            hCoeff_sgl1[0] = hipCaddf(hCoeff_sgl1[0],make_hipFloatComplex(temp_gh[0]*hipCrealf(pPsipn2),temp_gh[0]*hipCimagf(pPsipn2)));
            hCoeff_sgl1[1] = hipCaddf(hCoeff_sgl1[1],make_hipFloatComplex(temp_gh[1]*hipCrealf(pPsipn2),temp_gh[1]*hipCimagf(pPsipn2)));
            hCoeff_sgl1[2] = hipCaddf(hCoeff_sgl1[2],make_hipFloatComplex(temp_gh[2]*hipCrealf(pPsipn2),temp_gh[2]*hipCimagf(pPsipn2)));
            
            *cCoeff_sgl1 += temp*pPsiLpn2;
            
            //update coefficients with singularity on node 2
            rVec = vecSub(y_sgl2,x_sgl2);
            radius = sqrtf(rVec.coords[0]*rVec.coords[0]+rVec.coords[1]*rVec.coords[1]+rVec.coords[2]*rVec.coords[2]);
            //printf("radius = %f\n",radius);
            prpn2 = ((y_sgl1.coords[0]-x_sgl1.coords[0])*normal.coords[0]+(y_sgl1.coords[1]-x_sgl1.coords[1])*normal.coords[1]
                    +(y_sgl1.coords[2]-x_sgl1.coords[2])*normal.coords[2])/radius;
            //printf("prpn2=%f\n",prpn2);
            pPsiLpn2 = -recipFourPI/(radius*radius)*prpn2;
            //printf("%f\n",pPsiLpn2);
            Psi = make_hipFloatComplex(__cosf(-k*radius)/(fourPI*radius),__sinf(-k*radius)/(fourPI*radius));
            pPsipn2 = hipCmulf(Psi,make_hipFloatComplex(-1.0f/radius,-k));
            pPsipn2 = make_hipFloatComplex(prpn2*hipCrealf(pPsipn2),prpn2*hipCimagf(pPsipn2));
            temp_gh[0] = temp*xi1_sgl2;
            temp_gh[1] = temp*xi2_sgl2;
            temp_gh[2] = temp*xi3_sgl2;
            
            gCoeff_sgl2[0] = hipCaddf(gCoeff_sgl2[0],make_hipFloatComplex(temp_gh[0]*hipCrealf(Psi),temp_gh[0]*hipCimagf(Psi)));
            gCoeff_sgl2[1] = hipCaddf(gCoeff_sgl2[1],make_hipFloatComplex(temp_gh[1]*hipCrealf(Psi),temp_gh[1]*hipCimagf(Psi)));
            gCoeff_sgl2[2] = hipCaddf(gCoeff_sgl2[2],make_hipFloatComplex(temp_gh[2]*hipCrealf(Psi),temp_gh[2]*hipCimagf(Psi)));
            
            hCoeff_sgl2[0] = hipCaddf(hCoeff_sgl2[0],make_hipFloatComplex(temp_gh[0]*hipCrealf(pPsipn2),temp_gh[0]*hipCimagf(pPsipn2)));
            hCoeff_sgl2[1] = hipCaddf(hCoeff_sgl2[1],make_hipFloatComplex(temp_gh[1]*hipCrealf(pPsipn2),temp_gh[1]*hipCimagf(pPsipn2)));
            hCoeff_sgl2[2] = hipCaddf(hCoeff_sgl2[2],make_hipFloatComplex(temp_gh[2]*hipCrealf(pPsipn2),temp_gh[2]*hipCimagf(pPsipn2)));
            
            *cCoeff_sgl2 += temp*pPsiLpn2;
            
            //update coefficients with singularity on node 3
            rVec = vecSub(y_sgl3,x_sgl3);
            radius = sqrtf(rVec.coords[0]*rVec.coords[0]+rVec.coords[1]*rVec.coords[1]+rVec.coords[2]*rVec.coords[2]);
            //printf("radius = %f\n",radius);
            prpn2 = ((y_sgl1.coords[0]-x_sgl1.coords[0])*normal.coords[0]+(y_sgl1.coords[1]-x_sgl1.coords[1])*normal.coords[1]
                    +(y_sgl1.coords[2]-x_sgl1.coords[2])*normal.coords[2])/radius;
            //printf("prpn2=%f\n",prpn2);
            pPsiLpn2 = -recipFourPI/(radius*radius)*prpn2;
            //printf("%f\n",pPsiLpn2);
            Psi = make_hipFloatComplex(__cosf(-k*radius)/(fourPI*radius),__sinf(-k*radius)/(fourPI*radius));
            pPsipn2 = hipCmulf(Psi,make_hipFloatComplex(-1.0f/radius,-k));
            pPsipn2 = make_hipFloatComplex(prpn2*hipCrealf(pPsipn2),prpn2*hipCimagf(pPsipn2));
            temp_gh[0] = temp*xi1_sgl3;
            temp_gh[1] = temp*xi2_sgl3;
            temp_gh[2] = temp*xi3_sgl3;
            
            gCoeff_sgl3[0] = hipCaddf(gCoeff_sgl3[0],make_hipFloatComplex(temp_gh[0]*hipCrealf(Psi),temp_gh[0]*hipCimagf(Psi)));
            gCoeff_sgl3[1] = hipCaddf(gCoeff_sgl3[1],make_hipFloatComplex(temp_gh[1]*hipCrealf(Psi),temp_gh[1]*hipCimagf(Psi)));
            gCoeff_sgl3[2] = hipCaddf(gCoeff_sgl3[2],make_hipFloatComplex(temp_gh[2]*hipCrealf(Psi),temp_gh[2]*hipCimagf(Psi)));
            
            hCoeff_sgl3[0] = hipCaddf(hCoeff_sgl3[0],make_hipFloatComplex(temp_gh[0]*hipCrealf(pPsipn2),temp_gh[0]*hipCimagf(pPsipn2)));
            hCoeff_sgl3[1] = hipCaddf(hCoeff_sgl3[1],make_hipFloatComplex(temp_gh[1]*hipCrealf(pPsipn2),temp_gh[1]*hipCimagf(pPsipn2)));
            hCoeff_sgl3[2] = hipCaddf(hCoeff_sgl3[2],make_hipFloatComplex(temp_gh[2]*hipCrealf(pPsipn2),temp_gh[2]*hipCimagf(pPsipn2)));
            
            *cCoeff_sgl3 += temp*pPsiLpn2;
        }
    }
    gCoeff_sgl1[0] = make_hipFloatComplex(-prodRhoOmega*hipCimagf(gCoeff_sgl1[0]),prodRhoOmega*hipCrealf(gCoeff_sgl1[0]));
    gCoeff_sgl1[1] = make_hipFloatComplex(-prodRhoOmega*hipCimagf(gCoeff_sgl1[1]),prodRhoOmega*hipCrealf(gCoeff_sgl1[1]));
    gCoeff_sgl1[2] = make_hipFloatComplex(-prodRhoOmega*hipCimagf(gCoeff_sgl1[2]),prodRhoOmega*hipCrealf(gCoeff_sgl1[2]));
    
    gCoeff_sgl2[0] = make_hipFloatComplex(-prodRhoOmega*hipCimagf(gCoeff_sgl2[0]),prodRhoOmega*hipCrealf(gCoeff_sgl2[0]));
    gCoeff_sgl2[1] = make_hipFloatComplex(-prodRhoOmega*hipCimagf(gCoeff_sgl2[1]),prodRhoOmega*hipCrealf(gCoeff_sgl2[1]));
    gCoeff_sgl2[2] = make_hipFloatComplex(-prodRhoOmega*hipCimagf(gCoeff_sgl2[2]),prodRhoOmega*hipCrealf(gCoeff_sgl2[2]));
    
    gCoeff_sgl3[0] = make_hipFloatComplex(-prodRhoOmega*hipCimagf(gCoeff_sgl3[0]),prodRhoOmega*hipCrealf(gCoeff_sgl3[0]));
    gCoeff_sgl3[1] = make_hipFloatComplex(-prodRhoOmega*hipCimagf(gCoeff_sgl3[1]),prodRhoOmega*hipCrealf(gCoeff_sgl3[1]));
    gCoeff_sgl3[2] = make_hipFloatComplex(-prodRhoOmega*hipCimagf(gCoeff_sgl3[2]),prodRhoOmega*hipCrealf(gCoeff_sgl3[2]));
}

__host__ __device__ hipFloatComplex ptSrc(const float k, const float amp, const vec3f srcLoc, const vec3f evalLoc)
{
    float fourPI = 4.0f*PI;
    vec3f rVec = vecSub(evalLoc,srcLoc);
    float radius = sqrtf(rVec.coords[0]*rVec.coords[0]+rVec.coords[1]*rVec.coords[1]+rVec.coords[2]*rVec.coords[2]);
    return make_hipFloatComplex(amp*cosf(-k*radius)/(fourPI*radius),amp*sinf(-k*radius)/(fourPI*radius));
}

__host__ __device__ hipFloatComplex mpSrc(const float k, const float qs, const vec3f src, const vec3f eval)
{
    vec3f vec = vecSub(eval,src);
    float radius = sqrtf(vec.coords[0]*vec.coords[0]+vec.coords[1]*vec.coords[1]+vec.coords[2]*vec.coords[2]);
    hipFloatComplex result = make_hipFloatComplex(0,RHO_AIR*SPEED_SOUND*k*qs/(4*PI));
    result = hipCmulf(result,make_hipFloatComplex(cos(-k*radius)/radius,sin(-k*radius)/radius));
    return result;
}

__host__ __device__ hipFloatComplex dirSrc(const float k, const float strength, const vec3f dir, const vec3f evalLoc)
{
    float theta = -k*vecDotMul(dir,evalLoc);
    return make_hipFloatComplex(strength*cosf(theta),strength*sinf(theta));
}

// compute non-singular relationship between points and elements
__global__ void atomicPtsElems_nsgl(const float k, const vec3f *pt, const int numNod, 
        const int idxPntStart, const int idxPntEnd, const tri_elem *elem, const int numElem, 
        hipFloatComplex *A, const int lda, hipFloatComplex *B, const int numSrc, const int ldb) {
    int xIdx = blockIdx.x*blockDim.x+threadIdx.x; //Index for points
    int yIdx = blockIdx.y*blockDim.y+threadIdx.y; //Index for elements
    //The thread with indices xIdx and yIdx process the point xIdx and elem yIdx
    if(xIdx>=idxPntStart && xIdx<=idxPntEnd && yIdx<numElem && xIdx!=elem[yIdx].nod[0] 
            && xIdx!=elem[yIdx].nod[1] && xIdx!=elem[yIdx].nod[2]) {
        int i, j;
        hipFloatComplex hCoeff[3], gCoeff[3], bc, pCoeffs[3], temp;
        float cCoeff;
        vec3f triNod[3];
        triNod[0] = pt[elem[yIdx].nod[0]];
        triNod[1] = pt[elem[yIdx].nod[1]];
        triNod[2] = pt[elem[yIdx].nod[2]];
        g_h_c_nsgl(k,pt[xIdx],triNod,gCoeff,hCoeff,&cCoeff);
        
        //Update the A matrix
        bc = hipCdivf(elem[yIdx].bc[0],elem[yIdx].bc[1]);
        for(i=0;i<3;i++) {
            pCoeffs[i] = hipCsubf(hCoeff[i],hipCmulf(bc,gCoeff[i]));
        }
        
        for(i=0;i<3;i++) {
            //atomicFloatComplexAdd(&A[IDXC0(xIdx,elem[yIdx].nod[i],lda)],pCoeffs[i]);
            atomicAdd(&A[IDXC0(xIdx,elem[yIdx].nod[i],lda)].x,hipCrealf(pCoeffs[i]));
            atomicAdd(&A[IDXC0(xIdx,elem[yIdx].nod[i],lda)].y,hipCimagf(pCoeffs[i]));
        }
        
        //Update from C coefficients
        if(xIdx<numNod) {
            //atomicFloatComplexSub(&A[IDXC0(xIdx,xIdx,lda)],make_hipFloatComplex(cCoeff,0));
            atomicAdd(&A[IDXC0(xIdx,xIdx,lda)].x,-cCoeff);
        }
        
        //Update the B matrix
        bc = hipCdivf(elem[yIdx].bc[2],elem[yIdx].bc[1]);
        //printf("bc: \n");
        //printComplexMatrix(&bc,1,1,1);
        for(i=0;i<numSrc;i++) {
            for(j=0;j<3;j++) {
                //atomicFloatComplexSub(&B[IDXC0(xIdx,i,ldb)],hipCmulf(bc,gCoeff[j]));
                temp = hipCmulf(bc,gCoeff[j]);
                atomicAdd(&B[IDXC0(xIdx,i,ldb)].x,-hipCrealf(temp));
                atomicAdd(&B[IDXC0(xIdx,i,ldb)].y,-hipCimagf(temp));
            }
        }
    }
}

__global__ void atomicPtsElems_sgl(const float k, const vec3f *pt, const tri_elem *elem, 
        const int numElem, hipFloatComplex *A, const int lda, hipFloatComplex *B, 
        const int numSrc, const int ldb) {
    int idx = blockIdx.x*blockDim.x+threadIdx.x;
    if(idx < numElem) {
        int i, j;
        hipFloatComplex hCoeff_sgl1[3], hCoeff_sgl2[3], hCoeff_sgl3[3], 
                gCoeff_sgl1[3], gCoeff_sgl2[3], gCoeff_sgl3[3], pCoeffs_sgl1[3], 
                pCoeffs_sgl2[3], pCoeffs_sgl3[3], bc, temp;
        float cCoeff_sgl1, cCoeff_sgl2, cCoeff_sgl3;
        
        vec3f nod[3];
        for(i=0;i<3;i++) {
            nod[i] = pt[elem[idx].nod[i]];
        }
        // Compute h and g coefficients
        g_h_c_sgl(k,pt[elem[idx].nod[0]],pt[elem[idx].nod[1]],pt[elem[idx].nod[2]],
                nod,gCoeff_sgl1,hCoeff_sgl1,&cCoeff_sgl1,gCoeff_sgl2,hCoeff_sgl2,&cCoeff_sgl2,
                gCoeff_sgl3,hCoeff_sgl3,&cCoeff_sgl3);
        
        //Compute p coefficients
        bc = hipCdivf(elem[idx].bc[0],elem[idx].bc[1]);
        for(j=0;j<3;j++) {
            pCoeffs_sgl1[j] = hipCsubf(hCoeff_sgl1[j],hipCmulf(bc,gCoeff_sgl1[j]));
            pCoeffs_sgl2[j] = hipCsubf(hCoeff_sgl2[j],hipCmulf(bc,gCoeff_sgl2[j]));
            pCoeffs_sgl3[j] = hipCsubf(hCoeff_sgl3[j],hipCmulf(bc,gCoeff_sgl3[j]));
        }
        
        //Update matrix A using pCoeffs
        for(j=0;j<3;j++) {
            //atomicFloatComplexAdd(&A[IDXC0(elem[idx].nod[0],elem[idx].nod[j],lda)],
            //        pCoeffs_sgl1[j]);
            atomicAdd(&A[IDXC0(elem[idx].nod[0],elem[idx].nod[j],lda)].x,
                    hipCrealf(pCoeffs_sgl1[j]));
            atomicAdd(&A[IDXC0(elem[idx].nod[0],elem[idx].nod[j],lda)].y,
                    hipCimagf(pCoeffs_sgl1[j]));
            //atomicFloatComplexAdd(&A[IDXC0(elem[idx].nod[1],elem[idx].nod[j],lda)],
            //        pCoeffs_sgl2[j]);
            atomicAdd(&A[IDXC0(elem[idx].nod[1],elem[idx].nod[j],lda)].x,
                    hipCrealf(pCoeffs_sgl2[j]));
            atomicAdd(&A[IDXC0(elem[idx].nod[1],elem[idx].nod[j],lda)].y,
                    hipCimagf(pCoeffs_sgl2[j]));
            //atomicFloatComplexAdd(&A[IDXC0(elem[idx].nod[2],elem[idx].nod[j],lda)],
            //        pCoeffs_sgl3[j]);
            atomicAdd(&A[IDXC0(elem[idx].nod[2],elem[idx].nod[j],lda)].x,
                    hipCrealf(pCoeffs_sgl3[j]));
            atomicAdd(&A[IDXC0(elem[idx].nod[2],elem[idx].nod[j],lda)].y,
                    hipCimagf(pCoeffs_sgl3[j]));
        }
        
        //atomicFloatComplexSub(&A[IDXC0(elem[idx].nod[0],elem[idx].nod[0],lda)],
        //        make_hipFloatComplex(cCoeff_sgl1,0));
        atomicAdd(&A[IDXC0(elem[idx].nod[0],elem[idx].nod[0],lda)].x,
                -cCoeff_sgl1);
        //atomicFloatComplexSub(&A[IDXC0(elem[idx].nod[1],elem[idx].nod[1],lda)],
        //        make_hipFloatComplex(cCoeff_sgl2,0));
        atomicAdd(&A[IDXC0(elem[idx].nod[1],elem[idx].nod[1],lda)].x,
                -cCoeff_sgl2);
        //atomicFloatComplexSub(&A[IDXC0(elem[idx].nod[2],elem[idx].nod[2],lda)],
        //        make_hipFloatComplex(cCoeff_sgl3,0));
        atomicAdd(&A[IDXC0(elem[idx].nod[2],elem[idx].nod[2],lda)].x,
                -cCoeff_sgl3);
        
        //Update matrix B using g Coefficients
        bc = hipCdivf(elem[idx].bc[2],elem[idx].bc[1]);
        for(i=0;i<numSrc;i++) {
            for(j=0;j<3;j++) {
                //atomicFloatComplexSub(&B[IDXC0(elem[idx].nod[0],i,ldb)],
                //        hipCmulf(bc,gCoeff_sgl1[j]));
                temp = hipCmulf(bc,gCoeff_sgl1[j]);
                atomicAdd(&B[IDXC0(elem[idx].nod[0],i,ldb)].x,-hipCrealf(temp));
                atomicAdd(&B[IDXC0(elem[idx].nod[0],i,ldb)].y,-hipCimagf(temp));
                //atomicFloatComplexSub(&B[IDXC0(elem[idx].nod[1],i,ldb)],
                //        hipCmulf(bc,gCoeff_sgl2[j]));
                temp = hipCmulf(bc,gCoeff_sgl2[j]);
                atomicAdd(&B[IDXC0(elem[idx].nod[1],i,ldb)].x,-hipCrealf(temp));
                atomicAdd(&B[IDXC0(elem[idx].nod[1],i,ldb)].y,-hipCimagf(temp));
                //atomicFloatComplexSub(&B[IDXC0(elem[idx].nod[2],i,ldb)],
                //        hipCmulf(bc,gCoeff_sgl3[j]));
                temp = hipCmulf(bc,gCoeff_sgl3[j]);
                atomicAdd(&B[IDXC0(elem[idx].nod[2],i,ldb)].x,-hipCrealf(temp));
                atomicAdd(&B[IDXC0(elem[idx].nod[2],i,ldb)].y,-hipCimagf(temp));
            }
        }
    }
}

int atomicGenSystem(const float k, const tri_elem *elem, const int numElem, 
        const vec3f *nod, const int numNod, const vec3f *chief, const int numCHIEF, 
        const vec3f *src, const int numSrc, hipFloatComplex *A, const int lda, 
        hipFloatComplex *B, const int ldb) {
    int i, j;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    //Move elements to GPU
    tri_elem *elem_d;
    CUDA_CALL(hipMalloc(&elem_d,numElem*sizeof(tri_elem)));
    CUDA_CALL(hipMemcpy(elem_d,elem,numElem*sizeof(tri_elem),hipMemcpyHostToDevice));
    
    //Move points to GPU
    vec3f *pt_h = (vec3f*)malloc((numNod+numCHIEF)*sizeof(vec3f));
    for(i=0;i<numNod;i++) {
        pt_h[i] = nod[i];
    }
    for(i=0;i<numCHIEF;i++) {
        pt_h[numNod+i] = chief[i];
    }
    
    vec3f *pt_d;
    CUDA_CALL(hipMalloc(&pt_d,(numNod+numCHIEF)*sizeof(vec3f)));
    CUDA_CALL(hipMemcpy(pt_d,pt_h,(numNod+numCHIEF)*sizeof(vec3f),hipMemcpyHostToDevice));
    
    //Initialization of A
    for(i=0;i<numNod+numCHIEF;i++) {
        for(j=0;j<numNod;j++) {
            if(i==j) {
                A[IDXC0(i,j,lda)] = make_hipFloatComplex(1,0);
            } else {
                A[IDXC0(i,j,lda)] = make_hipFloatComplex(0,0);
            }
        }
    }
    
    //Initialization of B
    for(i=0;i<numNod+numCHIEF;i++) {
        for(j=0;j<numSrc;j++) {
            B[IDXC0(i,j,ldb)] = ptSrc(k,STRENGTH,src[j],pt_h[i]);
        }
    }
    
    hipFloatComplex *A_d, *B_d;
    CUDA_CALL(hipMalloc(&A_d,(numNod+numCHIEF)*numNod*sizeof(hipFloatComplex)));
    CUDA_CALL(hipMemcpy(A_d,A,(numNod+numCHIEF)*numNod*sizeof(hipFloatComplex),hipMemcpyHostToDevice));
    
    CUDA_CALL(hipMalloc(&B_d,(numNod+numCHIEF)*numSrc*sizeof(hipFloatComplex)));
    CUDA_CALL(hipMemcpy(B_d,B,(numNod+numCHIEF)*numSrc*sizeof(hipFloatComplex),hipMemcpyHostToDevice));
    
    int xNumBlocks, xWidth = 16, yNumBlocks, yWidth = 16;
    xNumBlocks = (numNod+numCHIEF+xWidth-1)/xWidth;
    yNumBlocks = (numElem+yWidth-1)/yWidth;
    dim3 gridLayout, blockLayout;
    gridLayout.x = xNumBlocks;
    gridLayout.y = yNumBlocks;
    
    blockLayout.x = xWidth;
    blockLayout.y = yWidth;
    
    hipEventRecord(start);
    atomicPtsElems_nsgl<<<gridLayout,blockLayout>>>(k,pt_d,numNod,0,numNod+numCHIEF-1,
            elem_d,numElem,A_d,lda,B_d,numSrc,ldb);
    
    //CUDA_CALL(hipMemcpy(A,A_d,(numNod+numCHIEF)*numNod*sizeof(hipFloatComplex),hipMemcpyDeviceToHost));
    //printCuFloatComplexMat(A,numNod+numCHIEF,numNod,numNod+numCHIEF);
    atomicPtsElems_sgl<<<yNumBlocks,yWidth>>>(k,pt_d,elem_d,numElem,A_d,lda,B_d,numSrc,ldb);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds,start,stop);
    printf("Elapsed system generation time: %f milliseconds.\n",milliseconds);
    CUDA_CALL(hipMemcpy(A,A_d,(numNod+numCHIEF)*numNod*sizeof(hipFloatComplex),hipMemcpyDeviceToHost));
    CUDA_CALL(hipMemcpy(B,B_d,(numNod+numCHIEF)*numSrc*sizeof(hipFloatComplex),hipMemcpyDeviceToHost));
    
    
    CUDA_CALL(hipFree(A_d));
    CUDA_CALL(hipFree(B_d));
    CUDA_CALL(hipFree(elem_d));
    CUDA_CALL(hipFree(pt_d));
    
    return EXIT_SUCCESS;
}

int qrSolver(const hipFloatComplex *A, const int mA, const int nA, const int ldA, 
        hipFloatComplex *B, const int nB, const int ldB) {
    hipEvent_t start, stop;
    CUDA_CALL(hipEventCreate(&start));
    CUDA_CALL(hipEventCreate(&stop));
    hipsolverHandle_t cusolverH = NULL;
    CUSOLVER_CALL(hipsolverDnCreate(&cusolverH));
    
    
    hipFloatComplex *A_d;
    CUDA_CALL(hipMalloc(&A_d,ldA*nA*sizeof(hipFloatComplex)));
    CUDA_CALL(hipMemcpy(A_d,A,ldA*nA*sizeof(hipFloatComplex),hipMemcpyHostToDevice));
    
    hipFloatComplex *B_d;
    CUDA_CALL(hipMalloc(&B_d,ldB*nB*sizeof(hipFloatComplex)));
    CUDA_CALL(hipMemcpy(B_d,B,ldB*nB*sizeof(hipFloatComplex),hipMemcpyHostToDevice));
    
    //A = QR
    int lwork;
    CUSOLVER_CALL(hipsolverDnCgeqrf_bufferSize(cusolverH,mA,nA,A_d,ldA,&lwork));
    
    hipFloatComplex *workspace_d;
    CUDA_CALL(hipMalloc(&workspace_d,lwork*sizeof(hipFloatComplex)));
    hipFloatComplex *tau_d;
    CUDA_CALL(hipMalloc(&tau_d,max(mA,nA)*sizeof(hipFloatComplex)));
    int *deviceInfo_d, deviceInfo;
    CUDA_CALL(hipMalloc(&deviceInfo_d,sizeof(int)));
    
    CUDA_CALL(hipEventRecord(start));
    CUSOLVER_CALL(hipsolverDnCgeqrf(cusolverH,mA,nA,A_d,ldA,tau_d,workspace_d,lwork,
            deviceInfo_d));
    CUDA_CALL(hipMemcpy(&deviceInfo,deviceInfo_d,sizeof(int),hipMemcpyDeviceToHost));
    
    //B = (Q^H)*B
    CUSOLVER_CALL(hipsolverDnCunmqr(cusolverH,HIPBLAS_SIDE_LEFT,HIPBLAS_OP_C,mA,nB,
            nA,A_d,ldA,tau_d,B_d,ldB,workspace_d,lwork,deviceInfo_d));
    CUDA_CALL(hipMemcpy(&deviceInfo,deviceInfo_d,sizeof(int),hipMemcpyDeviceToHost));
    
    //Solve Rx = B
    hipFloatComplex alpha = make_hipFloatComplex(1,0);
    hipblasHandle_t cublasH;
    CUBLAS_CALL(hipblasCreate(&cublasH));
    CUBLAS_CALL(hipblasCtrsm(cublasH,HIPBLAS_SIDE_LEFT,HIPBLAS_FILL_MODE_UPPER,
            HIPBLAS_OP_N,HIPBLAS_DIAG_NON_UNIT,nA,nB,&alpha,A_d,ldA,B_d,ldB));
    CUDA_CALL(hipEventRecord(stop));
    
    CUDA_CALL(hipMemcpy(B,B_d,ldB*nB*sizeof(hipFloatComplex),hipMemcpyDeviceToHost));
    CUDA_CALL(hipEventSynchronize(stop));
    
    float milliseconds = 0;
    CUDA_CALL(hipEventElapsedTime(&milliseconds,start,stop));
    printf("Elapsed system solving time: %f milliseconds.\n",milliseconds);
    CUDA_CALL(hipEventDestroy(start));
    CUDA_CALL(hipEventDestroy(stop));
    CUDA_CALL(hipFree(A_d));
    CUDA_CALL(hipFree(B_d));
    CUDA_CALL(hipFree(tau_d));
    CUDA_CALL(hipFree(workspace_d));
    CUDA_CALL(hipFree(deviceInfo_d));
    CUBLAS_CALL(hipblasDestroy(cublasH));
    CUSOLVER_CALL(hipsolverDnDestroy(cusolverH));
    
    return EXIT_SUCCESS;
}

int bemSolver_pt(const float k, const tri_elem *elem, const int numElem, 
        const vec3f *nod, const int numNod, const vec3f *chief, const int numCHIEF, 
        const vec3f *src, const float* strength, const int numSrc, hipFloatComplex *B, const int ldb)
{
    int i, j;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    //Move elements to GPU
    tri_elem *elem_d;
    CUDA_CALL(hipMalloc(&elem_d,numElem*sizeof(tri_elem)));
    CUDA_CALL(hipMemcpy(elem_d,elem,numElem*sizeof(tri_elem),hipMemcpyHostToDevice));
    
    //Move points to GPU
    // vec3f *pt_h = (vec3f*)malloc((numNod+numCHIEF)*sizeof(vec3f));
    // for(i=0;i<numNod;i++) {
    //     pt_h[i] = nod[i];
    // }
    // for(i=0;i<numCHIEF;i++) {
    //     pt_h[numNod+i] = chief[i];
    // }
    
    vec3f *pt_d;
    CUDA_CALL(hipMalloc(&pt_d, (numNod + numCHIEF) * sizeof(vec3f)));
    CUDA_CALL(hipMemcpy(pt_d, nod, numNod * sizeof(vec3f),hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpy(pt_d + numNod, chief, numCHIEF * sizeof(vec3f),hipMemcpyHostToDevice));
    
    CUDA_CALL(hipEventRecord(start));
    //Generate the system
    hipFloatComplex *A = (hipFloatComplex*)malloc((numNod+numCHIEF)*numNod*sizeof(hipFloatComplex));
    
    memset(A,0,(numNod+numCHIEF)*numNod*sizeof(hipFloatComplex));

    for(i=0;i<numNod;i++) 
    {
        A[IDXC0(i,i,numNod+numCHIEF)] = make_hipFloatComplex(1,0);
    }
    
    //Initialization of B
    for(i=0;i<numNod+numCHIEF;i++) 
    {
        for(j=0;j<numSrc;j++) 
        {
            if(i < numNod)
                B[IDXC0(i,j,ldb)] = ptSrc(k,strength[j],src[j],nod[i]);
            else
                B[IDXC0(i,j,ldb)] = ptSrc(k,strength[j],src[j],chief[i - numNod]);
        }
    }
    
    hipFloatComplex *A_d, *B_d;
    CUDA_CALL(hipMalloc(&A_d,(numNod+numCHIEF)*numNod*sizeof(hipFloatComplex)));
    CUDA_CALL(hipMemcpy(A_d,A,(numNod+numCHIEF)*numNod*sizeof(hipFloatComplex),hipMemcpyHostToDevice));
    
    CUDA_CALL(hipMalloc(&B_d,(numNod+numCHIEF)*numSrc*sizeof(hipFloatComplex)));
    CUDA_CALL(hipMemcpy(B_d,B,(numNod+numCHIEF)*numSrc*sizeof(hipFloatComplex),hipMemcpyHostToDevice));
    
    int xNumBlocks, xWidth = 16, yNumBlocks, yWidth = 16;
    xNumBlocks = (numNod+numCHIEF+xWidth-1)/xWidth;
    yNumBlocks = (numElem+yWidth-1)/yWidth;
    dim3 gridLayout, blockLayout;
    gridLayout.x = xNumBlocks;
    gridLayout.y = yNumBlocks;
    
    blockLayout.x = xWidth;
    blockLayout.y = yWidth;
    
    atomicPtsElems_nsgl<<<gridLayout,blockLayout>>>(k,pt_d,numNod,0,numNod+numCHIEF-1,
            elem_d,numElem,A_d,numNod+numCHIEF,B_d,numSrc,ldb);
    atomicPtsElems_sgl<<<yNumBlocks,yWidth>>>(k,pt_d,elem_d,numElem,A_d,numNod+numCHIEF,
            B_d,numSrc,ldb);
    
    //Solving the system
    hipsolverHandle_t cusolverH = NULL;
    CUSOLVER_CALL(hipsolverDnCreate(&cusolverH));
    
    //A = QR
    int lwork;
    CUSOLVER_CALL(hipsolverDnCgeqrf_bufferSize(cusolverH,numNod+numCHIEF,numNod,A_d
            ,numNod+numCHIEF,&lwork));
    
    hipFloatComplex *workspace_d;
    CUDA_CALL(hipMalloc(&workspace_d,lwork*sizeof(hipFloatComplex)));
    hipFloatComplex *tau_d;
    CUDA_CALL(hipMalloc(&tau_d,(numNod+numCHIEF)*sizeof(hipFloatComplex)));
    int *deviceInfo_d, deviceInfo;
    CUDA_CALL(hipMalloc(&deviceInfo_d,sizeof(int)));
    
    
    CUSOLVER_CALL(hipsolverDnCgeqrf(cusolverH,numNod+numCHIEF,numNod,A_d,numNod+numCHIEF,
            tau_d,workspace_d,lwork,deviceInfo_d));
    CUDA_CALL(hipMemcpy(&deviceInfo,deviceInfo_d,sizeof(int),hipMemcpyDeviceToHost));
    
    //B = (Q^H)*B
    CUSOLVER_CALL(hipsolverDnCunmqr(cusolverH,HIPBLAS_SIDE_LEFT,HIPBLAS_OP_C,numNod+numCHIEF,numSrc,
            numNod,A_d,numNod+numCHIEF,tau_d,B_d,ldb,workspace_d,lwork,deviceInfo_d));
    CUDA_CALL(hipMemcpy(&deviceInfo,deviceInfo_d,sizeof(int),hipMemcpyDeviceToHost));
    
    //Solve Rx = B
    hipFloatComplex alpha = make_hipFloatComplex(1,0);
    hipblasHandle_t cublasH;
    CUBLAS_CALL(hipblasCreate(&cublasH));
    CUBLAS_CALL(hipblasCtrsm(cublasH,HIPBLAS_SIDE_LEFT,HIPBLAS_FILL_MODE_UPPER,
            HIPBLAS_OP_N,HIPBLAS_DIAG_NON_UNIT,numNod,numSrc,&alpha,A_d,numNod+numCHIEF,B_d,ldb));
    CUDA_CALL(hipEventRecord(stop));
    CUDA_CALL(hipEventSynchronize(stop));
    CUDA_CALL(hipMemcpy(B,B_d,ldb*numSrc*sizeof(hipFloatComplex),hipMemcpyDeviceToHost));
    
    float milliseconds = 0;
    CUDA_CALL(hipEventElapsedTime(&milliseconds,start,stop));
    printf("Elapsed system solving time: %f milliseconds.\n",milliseconds);
    
    //release memory
    CUDA_CALL(hipEventDestroy(start));
    CUDA_CALL(hipEventDestroy(stop));
    CUDA_CALL(hipFree(A_d));
    CUDA_CALL(hipFree(B_d));
    CUDA_CALL(hipFree(tau_d));
    CUDA_CALL(hipFree(workspace_d));
    CUDA_CALL(hipFree(deviceInfo_d));
    CUBLAS_CALL(hipblasDestroy(cublasH));
    CUSOLVER_CALL(hipsolverDnDestroy(cusolverH));
    CUDA_CALL(hipFree(elem_d));
    CUDA_CALL(hipFree(pt_d));
    free(A);
    return EXIT_SUCCESS;
}

int bemSolver_mp(const float k, const tri_elem *elem, const int numElem, 
        const vec3f *nod, const int numNod, const vec3f *chief, const int numCHIEF, 
        const vec3f *src, const float* strength, const int numSrc, hipFloatComplex *B, const int ldb)
{
    int i, j;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    //Move elements to GPU
    tri_elem *elem_d;
    CUDA_CALL(hipMalloc(&elem_d,numElem*sizeof(tri_elem)));
    CUDA_CALL(hipMemcpy(elem_d,elem,numElem*sizeof(tri_elem),hipMemcpyHostToDevice));
    
    //Move points to GPU
    // vec3f *pt_h = (vec3f*)malloc((numNod+numCHIEF)*sizeof(vec3f));
    // for(i=0;i<numNod;i++) {
    //     pt_h[i] = nod[i];
    // }
    // for(i=0;i<numCHIEF;i++) {
    //     pt_h[numNod+i] = chief[i];
    // }
    
    vec3f *pt_d;
    CUDA_CALL(hipMalloc(&pt_d, (numNod + numCHIEF) * sizeof(vec3f)));
    CUDA_CALL(hipMemcpy(pt_d, nod, numNod * sizeof(vec3f),hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpy(pt_d + numNod, chief, numCHIEF * sizeof(vec3f),hipMemcpyHostToDevice));
    
    CUDA_CALL(hipEventRecord(start));
    //Generate the system
    hipFloatComplex *A = (hipFloatComplex*)malloc((numNod+numCHIEF)*numNod*sizeof(hipFloatComplex));
    
    memset(A,0,(numNod+numCHIEF)*numNod*sizeof(hipFloatComplex));
    memset(B,0,(numNod+numCHIEF)*numSrc*sizeof(hipFloatComplex));

    for(i=0;i<numNod;i++) 
    {
        A[IDXC0(i,i,numNod+numCHIEF)] = make_hipFloatComplex(1,0);
    }
    
    //Initialization of B
    for(i=0;i<numNod+numCHIEF;i++) 
    {
        for(j=0;j<numSrc;j++) 
        {
            if(i < numNod)
                B[IDXC0(i,j,ldb)] = mpSrc(k,strength[j],src[j],nod[i]);
            else
                B[IDXC0(i,j,ldb)] = mpSrc(k,strength[j],src[j],chief[i-numNod]);
        }
    }
    
    hipFloatComplex *A_d, *B_d;
    CUDA_CALL(hipMalloc(&A_d,(numNod+numCHIEF)*numNod*sizeof(hipFloatComplex)));
    CUDA_CALL(hipMemcpy(A_d,A,(numNod+numCHIEF)*numNod*sizeof(hipFloatComplex),hipMemcpyHostToDevice));
    
    CUDA_CALL(hipMalloc(&B_d,(numNod+numCHIEF)*numSrc*sizeof(hipFloatComplex)));
    CUDA_CALL(hipMemcpy(B_d,B,(numNod+numCHIEF)*numSrc*sizeof(hipFloatComplex),hipMemcpyHostToDevice));
    
    int xNumBlocks, xWidth = 16, yNumBlocks, yWidth = 16;
    xNumBlocks = (numNod+numCHIEF+xWidth-1)/xWidth;
    yNumBlocks = (numElem+yWidth-1)/yWidth;
    dim3 gridLayout, blockLayout;
    gridLayout.x = xNumBlocks;
    gridLayout.y = yNumBlocks;
    
    blockLayout.x = xWidth;
    blockLayout.y = yWidth;
    
    atomicPtsElems_nsgl<<<gridLayout,blockLayout>>>(k,pt_d,numNod,0,numNod+numCHIEF-1,
            elem_d,numElem,A_d,numNod+numCHIEF,B_d,numSrc,ldb);
    atomicPtsElems_sgl<<<yNumBlocks,yWidth>>>(k,pt_d,elem_d,numElem,A_d,numNod+numCHIEF,
            B_d,numSrc,ldb);
    
    //Solving the system
    hipsolverHandle_t cusolverH = NULL;
    CUSOLVER_CALL(hipsolverDnCreate(&cusolverH));
    
    //A = QR
    int lwork;
    CUSOLVER_CALL(hipsolverDnCgeqrf_bufferSize(cusolverH,numNod+numCHIEF,numNod,A_d
            ,numNod+numCHIEF,&lwork));
    
    hipFloatComplex *workspace_d;
    CUDA_CALL(hipMalloc(&workspace_d,lwork*sizeof(hipFloatComplex)));
    hipFloatComplex *tau_d;
    CUDA_CALL(hipMalloc(&tau_d,(numNod+numCHIEF)*sizeof(hipFloatComplex)));
    int *deviceInfo_d, deviceInfo;
    CUDA_CALL(hipMalloc(&deviceInfo_d,sizeof(int)));
    
    
    CUSOLVER_CALL(hipsolverDnCgeqrf(cusolverH,numNod+numCHIEF,numNod,A_d,numNod+numCHIEF,
            tau_d,workspace_d,lwork,deviceInfo_d));
    CUDA_CALL(hipMemcpy(&deviceInfo,deviceInfo_d,sizeof(int),hipMemcpyDeviceToHost));
    
    //B = (Q^H)*B
    CUSOLVER_CALL(hipsolverDnCunmqr(cusolverH,HIPBLAS_SIDE_LEFT,HIPBLAS_OP_C,numNod+numCHIEF,numSrc,
            numNod,A_d,numNod+numCHIEF,tau_d,B_d,ldb,workspace_d,lwork,deviceInfo_d));
    CUDA_CALL(hipMemcpy(&deviceInfo,deviceInfo_d,sizeof(int),hipMemcpyDeviceToHost));
    
    //Solve Rx = B
    hipFloatComplex alpha = make_hipFloatComplex(1,0);
    hipblasHandle_t cublasH;
    CUBLAS_CALL(hipblasCreate(&cublasH));
    CUBLAS_CALL(hipblasCtrsm(cublasH,HIPBLAS_SIDE_LEFT,HIPBLAS_FILL_MODE_UPPER,
            HIPBLAS_OP_N,HIPBLAS_DIAG_NON_UNIT,numNod,numSrc,&alpha,A_d,numNod+numCHIEF,B_d,ldb));
    CUDA_CALL(hipEventRecord(stop));
    CUDA_CALL(hipEventSynchronize(stop));
    CUDA_CALL(hipMemcpy(B,B_d,ldb*numSrc*sizeof(hipFloatComplex),hipMemcpyDeviceToHost));
    
    float milliseconds = 0;
    CUDA_CALL(hipEventElapsedTime(&milliseconds,start,stop));
    printf("Elapsed system solving time: %f milliseconds.\n",milliseconds);
    
    //release memory
    CUDA_CALL(hipEventDestroy(start));
    CUDA_CALL(hipEventDestroy(stop));
    CUDA_CALL(hipFree(A_d));
    CUDA_CALL(hipFree(B_d));
    CUDA_CALL(hipFree(tau_d));
    CUDA_CALL(hipFree(workspace_d));
    CUDA_CALL(hipFree(deviceInfo_d));
    CUBLAS_CALL(hipblasDestroy(cublasH));
    CUSOLVER_CALL(hipsolverDnDestroy(cusolverH));
    CUDA_CALL(hipFree(elem_d));
    CUDA_CALL(hipFree(pt_d));
    free(A);
    return EXIT_SUCCESS;
}

int bemSolver_dir(const float k, const tri_elem *elem, const int numElem, 
        const vec3f *nod, const int numNod, const vec3f *chief, const int numCHIEF, 
        const vec3f *dir, const float* strength, const int numSrc, hipFloatComplex *B, const int ldb)
{
    int i, j;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    //Move elements to GPU
    tri_elem *elem_d;
    CUDA_CALL(hipMalloc(&elem_d,numElem*sizeof(tri_elem)));
    CUDA_CALL(hipMemcpy(elem_d,elem,numElem*sizeof(tri_elem),hipMemcpyHostToDevice));
    
    //Move points to GPU
    // vec3f *pt_h = (vec3f*)malloc((numNod+numCHIEF)*sizeof(vec3f));
    // for(i=0;i<numNod;i++) {
    //     pt_h[i] = nod[i];
    // }
    // for(i=0;i<numCHIEF;i++) {
    //     pt_h[numNod+i] = chief[i];
    // }
    
    vec3f *pt_d;
    CUDA_CALL(hipMalloc(&pt_d,(numNod+numCHIEF)*sizeof(vec3f)));
    CUDA_CALL(hipMemcpy(pt_d,nod,numNod*sizeof(vec3f),hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpy(pt_d+numNod,chief,numCHIEF*sizeof(vec3f),hipMemcpyHostToDevice));
    
    CUDA_CALL(hipEventRecord(start));
    //Generate the system
    hipFloatComplex *A = (hipFloatComplex*)malloc((numNod+numCHIEF)*numNod*sizeof(hipFloatComplex));
    memset(A,0,(numNod+numCHIEF)*numNod*sizeof(hipFloatComplex));

    for(i=0;i<numNod;i++) 
    {
        A[IDXC0(i,i,numNod+numCHIEF)] = make_hipFloatComplex(1,0);
    }
    
    //Initialization of B
    for(i=0;i<numNod+numCHIEF;i++) 
    {
        for(j=0;j<numSrc;j++) 
        {
            if(i < numNod)
                //B[IDXC0(i,j,ldb)] = ptSrc(k,STRENGTH,src[j],nod[i]);
                B[IDXC0(i,j,ldb)] = dirSrc(k,strength[j],dir[j],nod[i]);
            else
                //B[IDXC0(i,j,ldb)] = ptSrc(k,STRENGTH,src[j],chief[i - numNod]);
                B[IDXC0(i,j,ldb)] = dirSrc(k,strength[j],dir[j],chief[i-numNod]);
        }
    }
    
    hipFloatComplex *A_d, *B_d;
    CUDA_CALL(hipMalloc(&A_d,(numNod+numCHIEF)*numNod*sizeof(hipFloatComplex)));
    CUDA_CALL(hipMemcpy(A_d,A,(numNod+numCHIEF)*numNod*sizeof(hipFloatComplex),hipMemcpyHostToDevice));
    
    CUDA_CALL(hipMalloc(&B_d,(numNod+numCHIEF)*numSrc*sizeof(hipFloatComplex)));
    CUDA_CALL(hipMemcpy(B_d,B,(numNod+numCHIEF)*numSrc*sizeof(hipFloatComplex),hipMemcpyHostToDevice));
    
    int xNumBlocks, xWidth = 16, yNumBlocks, yWidth = 16;
    xNumBlocks = (numNod+numCHIEF+xWidth-1)/xWidth;
    yNumBlocks = (numElem+yWidth-1)/yWidth;
    dim3 gridLayout, blockLayout;
    gridLayout.x = xNumBlocks;
    gridLayout.y = yNumBlocks;
    
    blockLayout.x = xWidth;
    blockLayout.y = yWidth;
    
    atomicPtsElems_nsgl<<<gridLayout,blockLayout>>>(k,pt_d,numNod,0,numNod+numCHIEF-1,
            elem_d,numElem,A_d,numNod+numCHIEF,B_d,numSrc,ldb);
    atomicPtsElems_sgl<<<yNumBlocks,yWidth>>>(k,pt_d,elem_d,numElem,A_d,numNod+numCHIEF,
            B_d,numSrc,ldb);
    
    //Solving the system
    hipsolverHandle_t cusolverH = NULL;
    CUSOLVER_CALL(hipsolverDnCreate(&cusolverH));
    
    //A = QR
    int lwork;
    CUSOLVER_CALL(hipsolverDnCgeqrf_bufferSize(cusolverH,numNod+numCHIEF,numNod,A_d
            ,numNod+numCHIEF,&lwork));
    
    hipFloatComplex *workspace_d;
    CUDA_CALL(hipMalloc(&workspace_d,lwork*sizeof(hipFloatComplex)));
    hipFloatComplex *tau_d;
    CUDA_CALL(hipMalloc(&tau_d,(numNod+numCHIEF)*sizeof(hipFloatComplex)));
    int *deviceInfo_d, deviceInfo;
    CUDA_CALL(hipMalloc(&deviceInfo_d,sizeof(int)));
    
    
    CUSOLVER_CALL(hipsolverDnCgeqrf(cusolverH,numNod+numCHIEF,numNod,A_d,numNod+numCHIEF,
            tau_d,workspace_d,lwork,deviceInfo_d));
    CUDA_CALL(hipMemcpy(&deviceInfo,deviceInfo_d,sizeof(int),hipMemcpyDeviceToHost));
    
    //B = (Q^H)*B
    CUSOLVER_CALL(hipsolverDnCunmqr(cusolverH,HIPBLAS_SIDE_LEFT,HIPBLAS_OP_C,numNod+numCHIEF,numSrc,
            numNod,A_d,numNod+numCHIEF,tau_d,B_d,ldb,workspace_d,lwork,deviceInfo_d));
    CUDA_CALL(hipMemcpy(&deviceInfo,deviceInfo_d,sizeof(int),hipMemcpyDeviceToHost));
    
    //Solve Rx = B
    hipFloatComplex alpha = make_hipFloatComplex(1,0);
    hipblasHandle_t cublasH;
    CUBLAS_CALL(hipblasCreate(&cublasH));
    CUBLAS_CALL(hipblasCtrsm(cublasH,HIPBLAS_SIDE_LEFT,HIPBLAS_FILL_MODE_UPPER,
            HIPBLAS_OP_N,HIPBLAS_DIAG_NON_UNIT,numNod,numSrc,&alpha,A_d,numNod+numCHIEF,B_d,ldb));
    CUDA_CALL(hipEventRecord(stop));
    CUDA_CALL(hipEventSynchronize(stop));
    CUDA_CALL(hipMemcpy(B,B_d,ldb*numSrc*sizeof(hipFloatComplex),hipMemcpyDeviceToHost));
    
    float milliseconds = 0;
    CUDA_CALL(hipEventElapsedTime(&milliseconds,start,stop));
    printf("Elapsed system solving time: %f milliseconds.\n",milliseconds);
    
    //release memory
    CUDA_CALL(hipEventDestroy(start));
    CUDA_CALL(hipEventDestroy(stop));
    CUDA_CALL(hipFree(A_d));
    CUDA_CALL(hipFree(B_d));
    CUDA_CALL(hipFree(tau_d));
    CUDA_CALL(hipFree(workspace_d));
    CUDA_CALL(hipFree(deviceInfo_d));
    CUBLAS_CALL(hipblasDestroy(cublasH));
    CUSOLVER_CALL(hipsolverDnDestroy(cusolverH));
    CUDA_CALL(hipFree(elem_d));
    CUDA_CALL(hipFree(pt_d));
    free(A);
    return EXIT_SUCCESS;
}

__host__ gsl_complex gsl_sf_bessel_hl(const int l, const double s)
{
    double x = gsl_sf_bessel_jl(l,s);
    double y = gsl_sf_bessel_yl(l,s);
    gsl_complex z = gsl_complex_rect(x,y);
    return z;
}

double jprime(const int n, const double r)
{
    double result;
    if(n == 0) {
        result = -gsl_sf_bessel_jl(1,r);
    } else {
        result = gsl_sf_bessel_jl(n-1,r)-(n+1)*gsl_sf_bessel_jl(n,r)/r;
    }
    return result;
}

gsl_complex hprime(const int n, const double r)
{
    gsl_complex result;
    if(n == 0) {
        result = gsl_complex_negative(gsl_sf_bessel_hl(1,r));
    } else {
        result = gsl_complex_sub(gsl_sf_bessel_hl(n-1,r),gsl_complex_mul_real(gsl_sf_bessel_hl(n,r),(n+1)/r));
    }
    return result;
}

__host__ __device__ vec3f sph2vec(const sph3f s)
{
    float r = s.coords[0], theta = s.coords[1], phi = s.coords[2];
    float x = r*sinf(theta)*cosf(phi), y = r*sinf(theta)*sinf(phi), z = r*cosf(theta);
    vec3f result;
    result.coords[0] = x;
    result.coords[1] = y;
    result.coords[2] = z;
    return result;
}

__host__ __device__ vec3d sph2vec(const sph3d s)
{
    double r = s.coords[0], theta = s.coords[1], phi = s.coords[2];
    double x = r*sin(theta)*cos(phi), y = r*sin(theta)*sin(phi), z = r*cos(theta);
    vec3d result;
    result.coords[0] = x;
    result.coords[1] = y;
    result.coords[2] = z;
    return result;
}

__host__ __device__ sph3f vec2sph(const vec3f s)
{
    sph3f temp;
    temp.coords[0] = sqrtf(powf(s.coords[0],2)+powf(s.coords[1],2)+powf(s.coords[2],2));
    temp.coords[1] = acosf(s.coords[2]/(temp.coords[0]));
    temp.coords[2] = atan2f(s.coords[1],s.coords[0]);
    return temp;
}

__host__ __device__ sph3d vec2sph(const vec3d s)
{
    sph3d temp;
    temp.coords[0] = sqrt(pow(s.coords[0],2)+pow(s.coords[1],2)+pow(s.coords[2],2));
    temp.coords[1] = acos(s.coords[2]/(temp.coords[0]));
    temp.coords[2] = atan2(s.coords[1],s.coords[0]);
    return temp;
}

__global__ void g_h_elem_extraps(const float wavNum, const vec3f nod[3], const vec3f* ptExtrap, 
        const int numExtrap, hipFloatComplex* g, hipFloatComplex* h, float* c)
{
    /*generate all g and h coefficients of a single element and all extrapolation point
     wavNum: the current wave number
     nod: 3 nodes on the element
     ptExtrap: an array for extrapolation points
     g: an array for all g coefficients, of length 3*numExtrap
     h: an array for all h coefficiens, of length 3*numExtrap
     c: an array for c coefficients, of length 3*numExtrap*/
    int idx = blockIdx.x*blockDim.x+threadIdx.x;
    if(idx<numExtrap) {
        g_h_c_nsgl(wavNum,ptExtrap[idx],nod,&g[3*idx],&h[3*idx],&c[3*idx]);
    }
}

__global__ void UpdateExrapSglElem(const float wavNum, const tri_elem elem, const vec3f* pt, 
        const vec3f* pt_extrap, const int numExtrap, const int numSrc, const hipFloatComplex* B, 
        const int ldb, hipFloatComplex* prsr)
{
    /* update the pressure array according to a single element 
       elem: the element
       g: an array for current g coefficients
       h: an array for current h coefficients
       B: pressure on the surface of the object
       ldb: the dimension of the B matrix
       prsr: an array for pressure of extrapolation points */
    
    int idx_extrap = blockIdx.x*blockDim.x+threadIdx.x; // index for extrapolation
    int idx_src = blockIdx.y*blockDim.y+threadIdx.y; // index for source
    
    if(idx_extrap<numExtrap && idx_src<numSrc) {
        vec3f nod[3];
        hipFloatComplex gCoeff[3], hCoeff[3], temp[3];
        float cCoeff;
        for(int i=0;i<3;i++) {
            nod[i] = pt[elem.nod[i]];
        }
        g_h_c_nsgl(wavNum,pt_extrap[idx_extrap],nod,gCoeff,hCoeff,&cCoeff); //compute coefficients
        for(int i=0;i<3;i++) {
            temp[i] = make_hipFloatComplex(0,0); // initialize temp
        }
        
        for(int i=0;i<3;i++) {
            temp[0] = hipCaddf(temp[0],gCoeff[i]);
        }
        temp[0] = hipCmulf(temp[0],hipCdivf(elem.bc[2],elem.bc[1]));
        
        for(int i=0;i<3;i++) {
            temp[1] = hipCaddf(temp[1],hipCmulf(hCoeff[i],B[IDXC0(elem.nod[i],idx_src,ldb)]));
        }
        
        for(int i=0;i<3;i++) {
            temp[2] = hipCaddf(temp[2],hipCmulf(gCoeff[i],B[IDXC0(elem.nod[i],idx_src,ldb)]));
        }
        temp[2] = hipCmulf(temp[2],hipCdivf(elem.bc[0],elem.bc[1]));
        
        temp[2] = hipCsubf(hipCsubf(temp[2],temp[1]),temp[0]);
        
        prsr[idx_src*numExtrap+idx_extrap] = hipCaddf(prsr[idx_src*numExtrap+idx_extrap],temp[2]);
        
    }
}

__device__ hipFloatComplex extrapolation_dir(const float wavNum, const vec3f x, 
        const tri_elem* elem, const int numElem, const vec3f* pt, 
        const hipFloatComplex* p, const float& strength, const vec3f& dir)
{
    /*field extrapolation from the surface to a single point in free space
     wavNum: wave number
     elem: pointer for all elements
     pt: pointer for all points
     x: the point in free space
     dir: the direction of the plane wave*/
    hipFloatComplex result = dirSrc(wavNum,strength,dir,x);
    hipFloatComplex temp;
    for(int i=0;i<numElem;i++) {
        vec3f nod[3];
        for(int j=0;j<3;j++) {
            nod[j] = pt[elem[i].nod[j]];
        }
        hipFloatComplex gCoeff[3], hCoeff[3]; 
        float cCoeff[3];
        g_h_c_nsgl(wavNum,x,nod,gCoeff,hCoeff,cCoeff);
        for(int j=0;j<3;j++) {
            temp = hipCdivf(elem[i].bc[2],elem[i].bc[1]);
            temp = hipCmulf(temp,gCoeff[j]);
            result = hipCsubf(result,temp);
            temp = hipCdivf(elem[i].bc[0],elem[i].bc[1]);
            temp = hipCmulf(temp,gCoeff[j]);
            temp = hipCsubf(hCoeff[j],temp);
            temp = hipCmulf(temp,p[elem[i].nod[j]]);
            result = hipCsubf(result,temp);
        }
    }
    return result;
}

__device__ hipFloatComplex extrapolation_pt(const float wavNum, const vec3f x, 
        const tri_elem* elem, const int numElem, const vec3f* pt, 
        const hipFloatComplex* p, const float strength, const vec3f src)
{
    /*field extrapolation from the surface to a single point in free space
     x: the single point in free space
     elem: pointer to mesh elements
     pt: pointer to mesh nod and chief points
     p: surface pressure
     strength: intensity of the source
     src: source location*/
    hipFloatComplex result = ptSrc(wavNum,strength,src,x);
    hipFloatComplex temp;
    vec3f nod[3];
    hipFloatComplex gCoeff[3], hCoeff[3]; 
    float cCoeff[3];
    for(int i=0;i<numElem;i++) {
        for(int j=0;j<3;j++) {
            nod[j] = pt[elem[i].nod[j]];
        }
        g_h_c_nsgl(wavNum,x,nod,gCoeff,hCoeff,cCoeff);
        for(int j=0;j<3;j++) {
            temp = hipCdivf(elem[i].bc[2],elem[i].bc[1]);
            temp = hipCmulf(temp,gCoeff[j]);
            result = hipCsubf(result,temp);
            temp = hipCdivf(elem[i].bc[0],elem[i].bc[1]);
            temp = hipCmulf(temp,gCoeff[j]);
            temp = hipCsubf(hCoeff[j],temp);
            temp = hipCmulf(temp,p[elem[i].nod[j]]);
            /*
            if(isnan(hipCrealf(temp)) || isnan(hipCimagf(temp))) {
                printf("problem with temp.\n");
                if(isnan(hipCrealf(p[elem[i].nod[j]])) || isnan(hipCimagf(p[elem[i].nod[j]]))) {
                    printf("%d elem, pressure array issue\n",i);
                    printf("nodes: %d, %d, %d\n",elem[i].nod[0],elem[i].nod[1],elem[i].nod[2]);
                    
                }
                return make_hipFloatComplex(0,0);
            }
            */ 
            result = hipCsubf(result,temp);
            /*
            if(isnan(hipCrealf(result)) || isnan(hipCimagf(result))) {
                printf("result issue.\n");
                return make_hipFloatComplex(0,0);
            }
            */ 
        }
    }
    return result;
}

__device__ hipFloatComplex extrapolation_mp(const float wavNum, const vec3f x, 
        const tri_elem* elem, const int numElem, const vec3f* pt, 
        const hipFloatComplex* p, const float strength, const vec3f src)
{
    /*field extrapolation from the surface to a single monopole in free space
     x: the single point in free space
     elem: pointer to mesh elements
     pt: pointer to mesh nod and chief points
     p: surface pressure
     strength: intensity of the source
     src: source location
     return: sound pressure at the extrapolation point*/
    hipFloatComplex result = mpSrc(wavNum,strength,src,x);
    hipFloatComplex temp;
    for(int i=0;i<numElem;i++) {
        vec3f nod[3];
        for(int j=0;j<3;j++) {
            nod[j] = pt[elem[i].nod[j]];
        }
        hipFloatComplex gCoeff[3], hCoeff[3]; 
        float cCoeff[3];
        g_h_c_nsgl(wavNum,x,nod,gCoeff,hCoeff,cCoeff);
        for(int j=0;j<3;j++) {
            temp = hipCdivf(elem[i].bc[2],elem[i].bc[1]);
            temp = hipCmulf(temp,gCoeff[j]);
            result = hipCsubf(result,temp);
            temp = hipCdivf(elem[i].bc[0],elem[i].bc[1]);
            temp = hipCmulf(temp,gCoeff[j]);
            temp = hipCsubf(hCoeff[j],temp);
            temp = hipCmulf(temp,p[elem[i].nod[j]]);
            result = hipCsubf(result,temp);
        }
    }
    return result;
}

__global__ void extrap_dir_sgl_src(const float wavNum, const vec3f* expPt, const int numExpPt,
        const tri_elem* elem, const int numElem, const vec3f* pt, const hipFloatComplex* p, 
        const float strength, const vec3f dir, hipFloatComplex *p_exp)
{
    /*
     extrapolation from surface pressure to multiple points in free space, a single source at a time
     wavNum: wave number
     expPt: extrapolation points in free space
     p: surface pressure
     dir: direction of the plane wave
     p_exp: pressure at the extrapolation points
     */
    int idx = blockIdx.x*blockDim.x+threadIdx.x;
    if(idx < numExpPt) {
        p_exp[idx] = extrapolation_dir(wavNum,expPt[idx],elem,numElem,pt,p,strength,dir);
    }
}

__global__ void extrap_dir_multi_src(const float wavNum, const vec3f* pt_extrap, const int numExtrap, 
        const tri_elem* elem, const int numElem, const vec3f* pt, const hipFloatComplex* B, 
        const int ldb, const float* strength, const vec3f* src, const int numSrc, hipFloatComplex* prsr)
{
    int idx_src = blockIdx.x*blockDim.x+threadIdx.x;
    int idx_extrap = blockIdx.y*blockDim.y+threadIdx.y;
    if(idx_src<numSrc && idx_extrap<numExtrap) {
        int idx_prsr = idx_src*numExtrap+idx_extrap;
        prsr[idx_prsr] = extrapolation_dir(wavNum,pt_extrap[idx_extrap],elem,numElem,
                pt,&B[idx_src*ldb],strength[idx_src],src[idx_src]);
    }
}

__global__ void extrap_pt_sgl_src(const float wavNum, const vec3f* expPt, const int numExpPt,
        const tri_elem* elem, const int numElem, const vec3f* pt, const hipFloatComplex* p, 
        const float strength, const vec3f src, hipFloatComplex *p_exp)
{
    /*extrapolation from surface pressure to multiple points in free space
     wavNum: wave number
     expPt:  extrapolation  points in free space
     p: surface pressure
     src: location of the source
     p_exp: pressure at the extrapolation points*/
    int idx = blockIdx.x*blockDim.x+threadIdx.x;
    if(idx < numExpPt) {
        p_exp[idx] = extrapolation_pt(wavNum,expPt[idx],elem,numElem,pt,p,strength,src);
        /*
        if(idx==0) {
            printf("(%f,%f)\n",hipCrealf(p_exp[idx]),hipCimagf(p_exp[idx]));
            printf("strength: %f\n",strength);
            printf("source: (%f,%f,%f)\n",src.coords[0],src.coords[1],src.coords[2]);
            printf("extrapolation point: (%f,%f,%f)\n",expPt[idx].coords[0],expPt[idx].coords[1],
                    expPt[idx].coords[2]);
        }
        */ 
    }
}

__global__ void extrap_pt_multi_src(const float wavNum, const vec3f* pt_extrap, const int numExtrap, 
        const tri_elem* elem, const int numElem, const vec3f* pt, const hipFloatComplex* B, 
        const int ldb, const float* strength, const vec3f* src, const int numSrc, hipFloatComplex* prsr)
{
    int idx_src = blockIdx.x*blockDim.x+threadIdx.x;
    int idx_extrap = blockIdx.y*blockDim.y+threadIdx.y;
    if(idx_src<numSrc && idx_extrap<numExtrap) {
        int idx_prsr = idx_src*numExtrap+idx_extrap;
        prsr[idx_prsr] = extrapolation_pt(wavNum,pt_extrap[idx_extrap],elem,numElem,
                pt,&B[idx_src*ldb],strength[idx_src],src[idx_src]);
    }
}

__global__ void extrap_mp_sgl_src(const float wavNum, const vec3f* expPt, const int numExpPt,
        const tri_elem* elem, const int numElem, const vec3f* pt, const hipFloatComplex* p, 
        const float strength, const vec3f src, hipFloatComplex *p_exp)
{
    int idx = blockIdx.x*blockDim.x+threadIdx.x;
    if(idx < numExpPt) {
        p_exp[idx] = extrapolation_mp(wavNum,expPt[idx],elem,numElem,pt,p,strength,src);
    }
}

__global__ void extrap_mp_multi_src(const float wavNum, const vec3f* pt_extrap, const int numExtrap, 
        const tri_elem* elem, const int numElem, const vec3f* pt, const hipFloatComplex* B, 
        const int ldb, const float* strength, const vec3f* src, const int numSrc, hipFloatComplex* prsr)
{
    int idx_src = blockIdx.x*blockDim.x+threadIdx.x;
    int idx_extrap = blockIdx.y*blockDim.y+threadIdx.y;
    if(idx_src<numSrc && idx_extrap<numExtrap) {
        int idx_prsr = idx_src*numExtrap+idx_extrap;
        prsr[idx_prsr] = extrapolation_mp(wavNum,pt_extrap[idx_extrap],elem,numElem,
                pt,&B[idx_src*ldb],strength[idx_src],src[idx_src]);
    }
}

int field_extrapolation_single_dir(const float wavNum, const vec3f* expPt, const int numExpPt, 
        const tri_elem* elem, const int numElem, const vec3f* pt, const int numPt, 
        const hipFloatComplex* p, const float strength, const vec3f dir, hipFloatComplex *pExp)
{
    /*extrapolation of acoustic field from surface pressure
     wavNum: wave number
     expPt: extrapolation points in free space
     elem: pointer to mesh elements
     pt: pointer to mesh nod and chief points
     p: surface pressure
     strength: intensity of the sound source
     dir: direction of the plane wave
     pExp: pressure at extrapolation points*/
    int width = 16, numBlock = (numExpPt+width-1)/width;
    
    // allocate memory on GPU and copy data to GPU memory
    vec3f *expPt_d, *pt_d;
    tri_elem *elem_d;
    hipFloatComplex *p_d, *pExp_d;
    
    CUDA_CALL(hipMalloc(&expPt_d,numExpPt*sizeof(vec3f)));
    CUDA_CALL(hipMemcpy(expPt_d,expPt,numExpPt*sizeof(vec3f),hipMemcpyHostToDevice));
    
    CUDA_CALL(hipMalloc(&pt_d,numPt*sizeof(vec3f)));
    CUDA_CALL(hipMemcpy(pt_d,pt,numPt*sizeof(vec3f),hipMemcpyHostToDevice));
    
    CUDA_CALL(hipMalloc(&elem_d,numElem*sizeof(tri_elem)));
    CUDA_CALL(hipMemcpy(elem_d,elem,numElem*sizeof(tri_elem),hipMemcpyHostToDevice));
    
    CUDA_CALL(hipMalloc(&p_d,numPt*sizeof(hipFloatComplex)));
    CUDA_CALL(hipMemcpy(p_d,p,numPt*sizeof(hipFloatComplex),hipMemcpyHostToDevice));
    
    CUDA_CALL(hipMalloc(&pExp_d,numExpPt*sizeof(hipFloatComplex)));
    
    extrap_dir_sgl_src<<<numBlock,width>>>(wavNum,expPt_d,numExpPt,elem_d,numElem,pt_d,p_d,
            strength,dir,pExp_d);
    
    CUDA_CALL(hipMemcpy(pExp,pExp_d,numExpPt*sizeof(hipFloatComplex),hipMemcpyDeviceToHost));
    
    CUDA_CALL(hipFree(expPt_d));
    CUDA_CALL(hipFree(pt_d));
    CUDA_CALL(hipFree(elem_d));
    CUDA_CALL(hipFree(p_d));
    CUDA_CALL(hipFree(pExp_d));
    
    return EXIT_SUCCESS;
}

int field_extrapolation_single_pt(const float wavNum, const vec3f* expPt, const int numExpPt, 
        const tri_elem* elem, const int numElem, const vec3f* pt, const int numPt, 
        const hipFloatComplex* p, const float strength, const vec3f src, hipFloatComplex *pExp)
{
    /*Extrapolation of an acoustic field from surface pressure and a single point source
     wavNum: wave number
     expPt: pointer for extrapolation points
     elem: mesh elements
     pt: nod and chief points
     p: surface pressure
     strength: intensity of a source
     src: location of the point source
     pExp: pressure at extrapolation points*/
    int width = 16, numBlock = (numExpPt+width-1)/width;
    
    // allocate memory on GPU and copy data to GPU memory
    vec3f *expPt_d, *pt_d;
    tri_elem *elem_d;
    hipFloatComplex *p_d, *pExp_d;
    
    CUDA_CALL(hipMalloc(&expPt_d,numExpPt*sizeof(vec3f)));
    CUDA_CALL(hipMemcpy(expPt_d,expPt,numExpPt*sizeof(vec3f),hipMemcpyHostToDevice));
    
    CUDA_CALL(hipMalloc(&pt_d,numPt*sizeof(vec3f)));
    CUDA_CALL(hipMemcpy(pt_d,pt,numPt*sizeof(vec3f),hipMemcpyHostToDevice));
    
    CUDA_CALL(hipMalloc(&elem_d,numElem*sizeof(tri_elem)));
    CUDA_CALL(hipMemcpy(elem_d,elem,numElem*sizeof(tri_elem),hipMemcpyHostToDevice));
    
    CUDA_CALL(hipMalloc(&p_d,numPt*sizeof(hipFloatComplex)));
    CUDA_CALL(hipMemcpy(p_d,p,numPt*sizeof(hipFloatComplex),hipMemcpyHostToDevice));
    
    CUDA_CALL(hipMalloc(&pExp_d,numExpPt*sizeof(hipFloatComplex)));
    
    extrap_pt_sgl_src<<<numBlock,width>>>(wavNum,expPt_d,numExpPt,elem_d,numElem,pt_d,p_d,
            strength,src,pExp_d);
    
    CUDA_CALL(hipMemcpy(pExp,pExp_d,numExpPt*sizeof(hipFloatComplex),hipMemcpyDeviceToHost));
    
    CUDA_CALL(hipFree(expPt_d));
    CUDA_CALL(hipFree(pt_d));
    CUDA_CALL(hipFree(elem_d));
    CUDA_CALL(hipFree(p_d));
    CUDA_CALL(hipFree(pExp_d));
    
    return EXIT_SUCCESS;
}

int field_extrapolation_single_mp(const float wavNum, const vec3f* expPt, const int numExpPt, 
        const tri_elem* elem, const int numElem, const vec3f* pt, const int numPt, 
        const hipFloatComplex* p, const float strength, const vec3f src, hipFloatComplex *pExp)
{
    int width = 16, numBlock = (numExpPt+width-1)/width;
    
    // allocate memory on GPU and copy data to GPU memory
    vec3f *expPt_d, *pt_d;
    tri_elem *elem_d;
    hipFloatComplex *p_d, *pExp_d;
    
    CUDA_CALL(hipMalloc(&expPt_d,numExpPt*sizeof(vec3f)));
    CUDA_CALL(hipMemcpy(expPt_d,expPt,numExpPt*sizeof(vec3f),hipMemcpyHostToDevice));
    
    CUDA_CALL(hipMalloc(&pt_d,numPt*sizeof(vec3f)));
    CUDA_CALL(hipMemcpy(pt_d,pt,numPt*sizeof(vec3f),hipMemcpyHostToDevice));
    
    CUDA_CALL(hipMalloc(&elem_d,numElem*sizeof(tri_elem)));
    CUDA_CALL(hipMemcpy(elem_d,elem,numElem*sizeof(tri_elem),hipMemcpyHostToDevice));
    
    CUDA_CALL(hipMalloc(&p_d,numPt*sizeof(hipFloatComplex)));
    CUDA_CALL(hipMemcpy(p_d,p,numPt*sizeof(hipFloatComplex),hipMemcpyHostToDevice));
    
    CUDA_CALL(hipMalloc(&pExp_d,numExpPt*sizeof(hipFloatComplex)));
    
    extrap_mp_sgl_src<<<numBlock,width>>>(wavNum,expPt_d,numExpPt,elem_d,numElem,pt_d,p_d,
            strength,src,pExp_d);
    
    CUDA_CALL(hipMemcpy(pExp,pExp_d,numExpPt*sizeof(hipFloatComplex),hipMemcpyDeviceToHost));
    
    CUDA_CALL(hipFree(expPt_d));
    CUDA_CALL(hipFree(pt_d));
    CUDA_CALL(hipFree(elem_d));
    CUDA_CALL(hipFree(p_d));
    CUDA_CALL(hipFree(pExp_d));
    
    return EXIT_SUCCESS;
}

vec3f rectCoordDbl2rectCoordFlt(const vec3d t)
{
    vec3f result;
    for(int i=0;i<3;i++) {
        result.coords[i] = t.coords[i];
    }
    return result;
}

void rectCoordDblArr2rectCoordFltArr(const vec3d* dArr, 
        const int num, vec3f* fArr)
{
    for(int i=0;i<num;i++) {
        fArr[i] = rectCoordDbl2rectCoordFlt(dArr[i]);
    }
}

gsl_complex rigid_sphere_plane(const double wavNum, const double strength, const double a, 
        const double r, const double theta)
{
    gsl_complex result = gsl_complex_rect(0,0), temp_c;
    const int numTrunc = 70;
    for(int n=0;n<numTrunc;n++)
    {
        temp_c = gsl_complex_div(gsl_complex_rect(jprime(n,wavNum*a),0),hprime(n,wavNum*a));
        temp_c = gsl_complex_mul(temp_c,gsl_sf_bessel_hl(n,wavNum*r));
        temp_c = gsl_complex_sub(gsl_complex_rect(gsl_sf_bessel_jl(n,wavNum*r),0),temp_c);
        temp_c = gsl_complex_mul(gsl_complex_pow_real(gsl_complex_rect(0,1),n),temp_c);
        temp_c = gsl_complex_mul_real(temp_c,2*n+1);
        temp_c = gsl_complex_mul_real(temp_c,gsl_sf_legendre_Pl(n,cos(theta)));
        result = gsl_complex_add(result,temp_c);
    }
    result = gsl_complex_mul_real(result,strength);
    return result;
}

gsl_complex rigid_sphere_point(const double wavNum, const double strength, const double rs, 
        const double a, const vec3d y)
{
    const int truncNum = 100;
    const vec3d src = {0,0,rs};
    vec3d temp_cart_coord = vecSub(y,src);
    sph3d temp_sph_coord = vec2sph(temp_cart_coord);
    double R = temp_sph_coord.coords[0];
    temp_sph_coord = vec2sph(y);
    double r = temp_sph_coord.coords[0];
    double theta = temp_sph_coord.coords[1];
    gsl_complex result = gsl_complex_rect(strength*cos(wavNum*R)/(4*PI*R),strength*sin(wavNum*R)/(4*PI*R));
    for(int n=0;n<truncNum;n++) {
        gsl_complex temp[2];
        double t = (n+0.5)*jprime(n,wavNum*a)*wavNum*strength/(2*PI)*gsl_sf_legendre_Pl(n,cos(theta));
        temp[0] = gsl_complex_rect(0,t);
        temp[1] = gsl_complex_mul(gsl_sf_bessel_hl(n,wavNum*rs),gsl_sf_bessel_hl(n,wavNum*r));
        temp[0] = gsl_complex_mul(temp[0],temp[1]);
        temp[0] = gsl_complex_div(temp[0],hprime(n,wavNum*a));
        result = gsl_complex_sub(result,temp[0]);
    }
    return result;
}

gsl_complex rigid_sphere_monopole(const double wavNum, const double strength, const double rs, 
        const double a, const vec3d y)
{
    const int truncNum = 100;
    const vec3d src = {0,0,rs};
    vec3d temp_cart_coord = vecSub(y,src);
    sph3d temp_sph_coord = vec2sph(temp_cart_coord);
    double R = temp_sph_coord.coords[0];
    temp_sph_coord = vec2sph(y);
    double r = temp_sph_coord.coords[0];
    double theta = temp_sph_coord.coords[1];
    gsl_complex result = gsl_complex_rect(cos(wavNum*R)/(4*PI*R),sin(wavNum*R)/(4*PI*R));
    result = gsl_complex_mul(result,gsl_complex_rect(0,-RHO_AIR*SPEED_SOUND*wavNum*strength));
    for(int n=0;n<truncNum;n++) {
        gsl_complex temp[2];
        double t = RHO_AIR*SPEED_SOUND*strength*pow(wavNum,2)/(2*PI)*(n+0.5)*jprime(n,wavNum*a)*gsl_sf_legendre_Pl(n,cos(theta));
        temp[0] = gsl_complex_rect(t,0);
        temp[1] = gsl_complex_mul(gsl_sf_bessel_hl(n,wavNum*rs),gsl_sf_bessel_hl(n,wavNum*r));
        temp[0] = gsl_complex_mul(temp[0],temp[1]);
        temp[0] = gsl_complex_div(temp[0],hprime(n,wavNum*a));
        result = gsl_complex_sub(result,temp[0]);
    }
    return result;
}

int CheckNanInMat(const hipFloatComplex* mat, const int m, const int n, const int ld)
{
    for(int i=0;i<m;i++) {
        for(int j=0;j<n;j++) {
            if(isnan(hipCrealf(mat[IDXC0(i,j,ld)])) || isnan(hipCimagf(mat[IDXC0(i,j,ld)]))) {
                printf("Found nan in matrix, %dth row, %dth column.\n",i,j);
                return EXIT_FAILURE;
            }
        }
    }
    return EXIT_SUCCESS;
}

int GenerateFieldUsingBEM(const vec3f* nod, const int numNod, const tri_elem* elem, const int numElem,
        const float wavNum, const char* src_type, const vec3f* src_loc, const float* mag, 
        const int numSrc, const vec3f* pt_extrap, const int numExtrap, hipFloatComplex* prsr)
{
    /*generate acoustic fields using BEM
     nod: nodes on the mesh
     numNod: number of nodes on the mesh
     elem: elements on  the mesh
     numElem: number of elements on the mesh
     wavNum: the wave number
     src_type: the type of the sources
     src_loc: locations of sources
     mag: magnitudes of sources
     extrap_pt: extrapolation points
     numExtrap: number of extrapolation points
     prsr: acoustic fields, of length numSrc*numExtrap*/
    if(strcmp(src_type,"point")!=0 && strcmp(src_type,"monopole")!=0) {
        printf("No valid source type provided!\n");
        return EXIT_FAILURE;
    }
    
    //generate chief points
    vec3f *chief = (vec3f*)malloc(NUMCHIEF*sizeof(vec3f));
    HOST_CALL(genCHIEF(nod,numNod,elem,numElem,chief,NUMCHIEF));
    //printf("Generated chief.\n");
    
    tri_elem *elem_d;
    CUDA_CALL(hipMalloc(&elem_d,numElem*sizeof(tri_elem)));
    CUDA_CALL(hipMemcpy(elem_d,elem,numElem*sizeof(tri_elem),hipMemcpyHostToDevice));
    
    vec3f *pt_d;
    CUDA_CALL(hipMalloc(&pt_d,(numNod+NUMCHIEF)*sizeof(vec3f)));
    CUDA_CALL(hipMemcpy(pt_d,nod,numNod*sizeof(vec3f),hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpy(pt_d+numNod,chief,NUMCHIEF*sizeof(vec3f),hipMemcpyHostToDevice));
    
    hipFloatComplex *A = (hipFloatComplex*)malloc((numNod+NUMCHIEF)*numNod*sizeof(hipFloatComplex));
    if(A==NULL) {
        printf("Allocating A failed.\n");
        return EXIT_FAILURE;
    }
    memset(A,0,(numNod+NUMCHIEF)*numNod*sizeof(hipFloatComplex));
    for(int i=0;i<numNod;i++) 
    {
        A[IDXC0(i,i,numNod+NUMCHIEF)] = make_hipFloatComplex(1,0);
    }
    
    hipFloatComplex *B = (hipFloatComplex*)malloc((numNod+NUMCHIEF)*numSrc*sizeof(hipFloatComplex));
    if(B==NULL) {
        printf("Allocating B failed.\n");
        return EXIT_FAILURE;
    }
    memset(B,0,(numNod+NUMCHIEF)*numSrc*sizeof(hipFloatComplex));
    
    if(strcmp(src_type,"point")==0) { //point source used
        for(int i=0;i<numNod+NUMCHIEF;i++) 
        {
            for(int j=0;j<numSrc;j++) 
            {
                if(i<numNod)
                    B[IDXC0(i,j,numNod+NUMCHIEF)] = ptSrc(wavNum,mag[j],src_loc[j],nod[i]);
                else
                    B[IDXC0(i,j,numNod+NUMCHIEF)] = ptSrc(wavNum,mag[j],src_loc[j],chief[i-numNod]);
            }
        }
    }
    else { //monopole source used
        for(int i=0;i<numNod+NUMCHIEF;i++) 
        {
            for(int j=0;j<numSrc;j++) 
            {
                if(i<numNod)
                    B[IDXC0(i,j,numNod+NUMCHIEF)] = mpSrc(wavNum,mag[j],src_loc[j],nod[i]);
                else
                    B[IDXC0(i,j,numNod+NUMCHIEF)] = mpSrc(wavNum,mag[j],src_loc[j],chief[i-numNod]);
            }
        }
    }
    
    free(chief); //chief no longer needed
    
    // copy A and B to device memory
    hipFloatComplex *A_d, *B_d;
    CUDA_CALL(hipMalloc(&A_d,(numNod+NUMCHIEF)*numNod*sizeof(hipFloatComplex)));
    CUDA_CALL(hipMemcpy(A_d,A,(numNod+NUMCHIEF)*numNod*sizeof(hipFloatComplex),hipMemcpyHostToDevice));
    
    CUDA_CALL(hipMalloc(&B_d,(numNod+NUMCHIEF)*numSrc*sizeof(hipFloatComplex)));
    CUDA_CALL(hipMemcpy(B_d,B,(numNod+NUMCHIEF)*numSrc*sizeof(hipFloatComplex),hipMemcpyHostToDevice));
    
    /* A and B no longer needed on host */
    if(A!=NULL) {
        free(A);
    }
    if(B!=NULL) {
        free(B);
    }
    
    // x dimension represents points and y dimension represents elements
    int numBlock_pt, width_pt = 16, numBlock_el, width_el = 16;
    numBlock_pt = (numNod+NUMCHIEF+width_pt-1)/width_pt;
    numBlock_el = (numElem+width_el-1)/width_el;
    dim3 gridLayout, blockLayout;
    gridLayout.x = numBlock_pt;
    gridLayout.y = numBlock_el;
    
    blockLayout.x = width_pt;
    blockLayout.y = width_el;
    
    /*generate a system*/
    atomicPtsElems_nsgl<<<gridLayout,blockLayout>>>(wavNum,pt_d,numNod,0,numNod+NUMCHIEF-1,
            elem_d,numElem,A_d,numNod+NUMCHIEF,B_d,numSrc,numNod+NUMCHIEF);
    atomicPtsElems_sgl<<<numBlock_el,width_el>>>(wavNum,pt_d,elem_d,numElem,A_d,numNod+NUMCHIEF,
            B_d,numSrc,numNod+NUMCHIEF);
    
    /*solve the system using cusolver*/
    // create a handle for cusolver
    hipsolverHandle_t cusolverH = NULL;
    CUSOLVER_CALL(hipsolverDnCreate(&cusolverH));
    
    // A = QR
    int lwork;
    CUSOLVER_CALL(hipsolverDnCgeqrf_bufferSize(cusolverH,numNod+NUMCHIEF,numNod,A_d
            ,numNod+NUMCHIEF,&lwork));
    
    hipFloatComplex *workspace_d;
    CUDA_CALL(hipMalloc(&workspace_d,lwork*sizeof(hipFloatComplex)));
    hipFloatComplex *tau_d;
    CUDA_CALL(hipMalloc(&tau_d,(numNod+NUMCHIEF)*sizeof(hipFloatComplex)));
    int *deviceInfo_d, deviceInfo = 1;
    CUDA_CALL(hipMalloc(&deviceInfo_d,sizeof(int)));
    
    
    CUSOLVER_CALL(hipsolverDnCgeqrf(cusolverH,numNod+NUMCHIEF,numNod,A_d,numNod+NUMCHIEF,
            tau_d,workspace_d,lwork,deviceInfo_d));
    CUDA_CALL(hipMemcpy(&deviceInfo,deviceInfo_d,sizeof(int),hipMemcpyDeviceToHost));
    if(deviceInfo!=0) {
        printf("QR decomposition failed.\n");
        return EXIT_FAILURE;
    }
    
    //B = (Q^H)*B
    CUSOLVER_CALL(hipsolverDnCunmqr(cusolverH,HIPBLAS_SIDE_LEFT,HIPBLAS_OP_C,numNod+NUMCHIEF,numSrc,
            numNod,A_d,numNod+NUMCHIEF,tau_d,B_d,numNod+NUMCHIEF,workspace_d,lwork,deviceInfo_d));
    CUDA_CALL(hipMemcpy(&deviceInfo,deviceInfo_d,sizeof(int),hipMemcpyDeviceToHost));
    if(deviceInfo!=0) {
        printf("QR decomposition failed.\n");
        return EXIT_FAILURE;
    }
    
    //Solve Rx = B
    hipFloatComplex alpha = make_hipFloatComplex(1,0);
    hipblasHandle_t cublasH;
    CUBLAS_CALL(hipblasCreate(&cublasH));
    CUBLAS_CALL(hipblasCtrsm(cublasH,HIPBLAS_SIDE_LEFT,HIPBLAS_FILL_MODE_UPPER,
            HIPBLAS_OP_N,HIPBLAS_DIAG_NON_UNIT,numNod,numSrc,&alpha,A_d,numNod+NUMCHIEF,B_d,numNod+NUMCHIEF));
    
    //release memory
    CUDA_CALL(hipFree(A_d)); // A_d no longer needed in extrapolation
    //CUDA_CALL(hipFree(B_d));
    CUDA_CALL(hipFree(tau_d));
    CUDA_CALL(hipFree(workspace_d));
    CUDA_CALL(hipFree(deviceInfo_d));
    CUBLAS_CALL(hipblasDestroy(cublasH));
    CUSOLVER_CALL(hipsolverDnDestroy(cusolverH));
    //CUDA_CALL(hipFree(elem_d));
    //CUDA_CALL(hipFree(pt_d));
    //CUDA_CALL(hipDeviceSynchronize());
    //printf("Computed surface pressure.\n");
    
    
    /* start extrapolation */
    //printf("Number of exrapolation points: %d\n",numExtrap);
    int numExtrapGroup = (numExtrap+NUM_EXTRAP_PER_LAUNCH-1)/NUM_EXTRAP_PER_LAUNCH;
    int crrNumExtrap, width_extrap = 32, numBlock_extrap;
    
    // copy extrapolation points to GPU
    vec3f *pt_extrap_d;
    CUDA_CALL(hipMalloc(&pt_extrap_d,numExtrap*sizeof(vec3f)));
    CUDA_CALL(hipMemcpy(pt_extrap_d,pt_extrap,numExtrap*sizeof(vec3f),hipMemcpyHostToDevice));
    
    // allocate memory for pressure array
    hipFloatComplex *prsr_d;
    CUDA_CALL(hipMalloc(&prsr_d,NUM_EXTRAP_PER_LAUNCH*sizeof(hipFloatComplex)));
    
    for(int i=0;i<numSrc;i++) {
        //printf("source: \n");
        //printVec(src_loc+i,1);
        //printf("magnitude: %f\n",mag[i]);
        for(int j=0;j<numExtrapGroup;j++) {
            if(j<numExtrapGroup-1) {
                crrNumExtrap = NUM_EXTRAP_PER_LAUNCH;
            }
            else {
                crrNumExtrap = numExtrap-j*NUM_EXTRAP_PER_LAUNCH;
            }
            //printf("%dth source, %d points\n",i,crrNumExtrap);
            numBlock_extrap = (crrNumExtrap+width_extrap-1)/width_extrap;
            if(strcmp(src_type,"point")==0) {
                extrap_pt_sgl_src<<<numBlock_extrap,width_extrap>>>(wavNum,&pt_extrap_d[j*NUM_EXTRAP_PER_LAUNCH],
                        crrNumExtrap,elem_d,numElem,pt_d,&B_d[IDXC0(0,i,numNod+NUMCHIEF)],
                        mag[i],src_loc[i],prsr_d);
            }
            else {
                extrap_mp_sgl_src<<<numBlock_extrap,width_extrap>>>(wavNum,&pt_extrap_d[j*NUM_EXTRAP_PER_LAUNCH],
                        crrNumExtrap,elem_d,numElem,pt_d,&B_d[IDXC0(0,i,numNod+NUMCHIEF)],
                        mag[i],src_loc[i],prsr_d);
            }
            //CUDA_CALL(hipDeviceSynchronize());
            CUDA_CALL(hipMemcpy(&prsr[i*numExtrap+j*NUM_EXTRAP_PER_LAUNCH],prsr_d,crrNumExtrap*sizeof(hipFloatComplex),
                    hipMemcpyDeviceToHost));
            
            //CUDA_CALL(hipDeviceSynchronize());
        }
    }
    
    CUDA_CALL(hipFree(pt_d));
    CUDA_CALL(hipFree(elem_d));
    CUDA_CALL(hipFree(prsr_d));
    CUDA_CALL(hipFree(pt_extrap_d));
    CUDA_CALL(hipFree(B_d));
    //printf("Completed extrapolation.\n");
    return EXIT_SUCCESS;
}

int GenerateFieldCheapExtrap(const vec3f* nod, const int numNod, const tri_elem* elem, const int numElem,
        const float wavNum, const char* src_type, const vec3f* src_loc, const float* mag, 
        const int numSrc, const vec3f* pt_extrap, const int numExtrap, hipFloatComplex* prsr)
{
    /*generate acoustic fields using BEM
     nod: nodes on the mesh
     numNod: number of nodes on the mesh
     elem: elements on  the mesh
     numElem: number of elements on the mesh
     wavNum: the wave number
     src_type: the type of the sources
     src_loc: locations of sources
     mag: magnitudes of sources
     extrap_pt: extrapolation points
     numExtrap: number of extrapolation points
     prsr: acoustic fields, of length numSrc*numExtrap*/
    if(strcmp(src_type,"point")!=0 && strcmp(src_type,"monopole")!=0) {
        printf("No valid source type provided!\n");
        return EXIT_FAILURE;
    }
    
    //generate chief points
    vec3f *chief = (vec3f*)malloc(NUMCHIEF*sizeof(vec3f));
    HOST_CALL(genCHIEF(nod,numNod,elem,numElem,chief,NUMCHIEF));
    //printf("Generated chief.\n");
    
    tri_elem *elem_d;
    CUDA_CALL(hipMalloc(&elem_d,numElem*sizeof(tri_elem)));
    CUDA_CALL(hipMemcpy(elem_d,elem,numElem*sizeof(tri_elem),hipMemcpyHostToDevice));
    
    vec3f *pt_d;
    CUDA_CALL(hipMalloc(&pt_d,(numNod+NUMCHIEF)*sizeof(vec3f)));
    CUDA_CALL(hipMemcpy(pt_d,nod,numNod*sizeof(vec3f),hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpy(pt_d+numNod,chief,NUMCHIEF*sizeof(vec3f),hipMemcpyHostToDevice));
    
    hipFloatComplex *A = (hipFloatComplex*)malloc((numNod+NUMCHIEF)*numNod*sizeof(hipFloatComplex));
    if(A==NULL) {
        printf("Allocating A failed.\n");
        return EXIT_FAILURE;
    }
    memset(A,0,(numNod+NUMCHIEF)*numNod*sizeof(hipFloatComplex));
    for(int i=0;i<numNod;i++) 
    {
        A[IDXC0(i,i,numNod+NUMCHIEF)] = make_hipFloatComplex(1,0);
    }
    
    hipFloatComplex *B = (hipFloatComplex*)malloc((numNod+NUMCHIEF)*numSrc*sizeof(hipFloatComplex));
    if(B==NULL) {
        printf("Allocating B failed.\n");
        return EXIT_FAILURE;
    }
    memset(B,0,(numNod+NUMCHIEF)*numSrc*sizeof(hipFloatComplex));
    
    if(strcmp(src_type,"point")==0) { //point source used
        for(int i=0;i<numNod+NUMCHIEF;i++) 
        {
            for(int j=0;j<numSrc;j++) 
            {
                if(i<numNod)
                    B[IDXC0(i,j,numNod+NUMCHIEF)] = ptSrc(wavNum,mag[j],src_loc[j],nod[i]);
                else
                    B[IDXC0(i,j,numNod+NUMCHIEF)] = ptSrc(wavNum,mag[j],src_loc[j],chief[i-numNod]);
            }
        }
    }
    else { //monopole source used
        for(int i=0;i<numNod+NUMCHIEF;i++) 
        {
            for(int j=0;j<numSrc;j++) 
            {
                if(i<numNod)
                    B[IDXC0(i,j,numNod+NUMCHIEF)] = mpSrc(wavNum,mag[j],src_loc[j],nod[i]);
                else
                    B[IDXC0(i,j,numNod+NUMCHIEF)] = mpSrc(wavNum,mag[j],src_loc[j],chief[i-numNod]);
            }
        }
    }
    
    free(chief); //chief no longer needed
    
    // copy A and B to device memory
    hipFloatComplex *A_d, *B_d;
    CUDA_CALL(hipMalloc(&A_d,(numNod+NUMCHIEF)*numNod*sizeof(hipFloatComplex)));
    CUDA_CALL(hipMemcpy(A_d,A,(numNod+NUMCHIEF)*numNod*sizeof(hipFloatComplex),hipMemcpyHostToDevice));
    
    CUDA_CALL(hipMalloc(&B_d,(numNod+NUMCHIEF)*numSrc*sizeof(hipFloatComplex)));
    CUDA_CALL(hipMemcpy(B_d,B,(numNod+NUMCHIEF)*numSrc*sizeof(hipFloatComplex),hipMemcpyHostToDevice));
    
    /* A and B no longer needed on host */
    if(A!=NULL) {
        free(A);
    }
    if(B!=NULL) {
        free(B);
    }
    
    // x dimension represents points and y dimension represents elements
    int numBlock_pt, width_pt = 16, numBlock_el, width_el = 16;
    numBlock_pt = (numNod+NUMCHIEF+width_pt-1)/width_pt;
    numBlock_el = (numElem+width_el-1)/width_el;
    dim3 gridLayout, blockLayout;
    gridLayout.x = numBlock_pt;
    gridLayout.y = numBlock_el;
    
    blockLayout.x = width_pt;
    blockLayout.y = width_el;
    
    /*generate a system*/
    atomicPtsElems_nsgl<<<gridLayout,blockLayout>>>(wavNum,pt_d,numNod,0,numNod+NUMCHIEF-1,
            elem_d,numElem,A_d,numNod+NUMCHIEF,B_d,numSrc,numNod+NUMCHIEF);
    atomicPtsElems_sgl<<<numBlock_el,width_el>>>(wavNum,pt_d,elem_d,numElem,A_d,numNod+NUMCHIEF,
            B_d,numSrc,numNod+NUMCHIEF);
    
    /*solve the system using cusolver*/
    // create a handle for cusolver
    hipsolverHandle_t cusolverH = NULL;
    CUSOLVER_CALL(hipsolverDnCreate(&cusolverH));
    
    // A = QR
    int lwork;
    CUSOLVER_CALL(hipsolverDnCgeqrf_bufferSize(cusolverH,numNod+NUMCHIEF,numNod,A_d
            ,numNod+NUMCHIEF,&lwork));
    
    hipFloatComplex *workspace_d;
    CUDA_CALL(hipMalloc(&workspace_d,lwork*sizeof(hipFloatComplex)));
    hipFloatComplex *tau_d;
    CUDA_CALL(hipMalloc(&tau_d,(numNod+NUMCHIEF)*sizeof(hipFloatComplex)));
    int *deviceInfo_d, deviceInfo = 1;
    CUDA_CALL(hipMalloc(&deviceInfo_d,sizeof(int)));
    
    
    CUSOLVER_CALL(hipsolverDnCgeqrf(cusolverH,numNod+NUMCHIEF,numNod,A_d,numNod+NUMCHIEF,
            tau_d,workspace_d,lwork,deviceInfo_d));
    CUDA_CALL(hipMemcpy(&deviceInfo,deviceInfo_d,sizeof(int),hipMemcpyDeviceToHost));
    if(deviceInfo!=0) {
        printf("QR decomposition failed.\n");
        return EXIT_FAILURE;
    }
    
    //B = (Q^H)*B
    CUSOLVER_CALL(hipsolverDnCunmqr(cusolverH,HIPBLAS_SIDE_LEFT,HIPBLAS_OP_C,numNod+NUMCHIEF,numSrc,
            numNod,A_d,numNod+NUMCHIEF,tau_d,B_d,numNod+NUMCHIEF,workspace_d,lwork,deviceInfo_d));
    CUDA_CALL(hipMemcpy(&deviceInfo,deviceInfo_d,sizeof(int),hipMemcpyDeviceToHost));
    if(deviceInfo!=0) {
        printf("QR decomposition failed.\n");
        return EXIT_FAILURE;
    }
    
    //Solve Rx = B
    hipFloatComplex alpha = make_hipFloatComplex(1,0);
    hipblasHandle_t cublasH;
    CUBLAS_CALL(hipblasCreate(&cublasH));
    CUBLAS_CALL(hipblasCtrsm(cublasH,HIPBLAS_SIDE_LEFT,HIPBLAS_FILL_MODE_UPPER,
            HIPBLAS_OP_N,HIPBLAS_DIAG_NON_UNIT,numNod,numSrc,&alpha,A_d,numNod+NUMCHIEF,B_d,numNod+NUMCHIEF));
    
    //release memory
    CUDA_CALL(hipFree(A_d)); // A_d no longer needed in extrapolation
    //CUDA_CALL(hipFree(B_d));
    CUDA_CALL(hipFree(tau_d));
    CUDA_CALL(hipFree(workspace_d));
    CUDA_CALL(hipFree(deviceInfo_d));
    CUBLAS_CALL(hipblasDestroy(cublasH));
    CUSOLVER_CALL(hipsolverDnDestroy(cusolverH));
    //CUDA_CALL(hipFree(elem_d));
    //CUDA_CALL(hipFree(pt_d));
    //CUDA_CALL(hipDeviceSynchronize());
    //printf("Computed surface pressure.\n");
    
    
    /* start extrapolation */
    
    // initialization of pressure array
    for(int i=0;i<numSrc;i++) {
        for(int j=0;j<numExtrap;j++) {
            int idx = i*numExtrap+j;
            if(strcmp(src_type,"point")==0) {
                prsr[idx] = ptSrc(wavNum,mag[i],src_loc[i],pt_extrap[j]);
            }
            else {
                prsr[idx] = mpSrc(wavNum,mag[i],src_loc[i],pt_extrap[j]);
            }
        }
    }
    hipFloatComplex *prsr_d;
    CUDA_CALL(hipMalloc(&prsr_d,numSrc*numExtrap*sizeof(hipFloatComplex)));
    CUDA_CALL(hipMemcpy(prsr_d,prsr,numSrc*numExtrap*sizeof(hipFloatComplex),hipMemcpyHostToDevice));
    
    vec3f *pt_extrap_d;
    CUDA_CALL(hipMalloc(&pt_extrap_d,numExtrap*sizeof(vec3f)));
    CUDA_CALL(hipMemcpy(pt_extrap_d,pt_extrap,numExtrap*sizeof(vec3f),hipMemcpyHostToDevice));
    
    int width_src = 16, numBlock_src = (numSrc+width_src-1)/width_src;
    int width_extrap = 16, numBlock_extrap = (numExtrap+width_extrap-1)/width_extrap;
    gridLayout.x = numBlock_extrap;
    gridLayout.y = numBlock_src;
    
    blockLayout.x = width_extrap;
    blockLayout.y = width_src;
    
    for(int i=0;i<numElem;i++) {
        UpdateExrapSglElem<<<gridLayout,blockLayout>>>(wavNum,elem[i],pt_d,pt_extrap_d,
                numExtrap,numSrc,B_d,numNod+NUMCHIEF,prsr_d);
        CUDA_CALL(hipDeviceSynchronize());
    }
    
    CUDA_CALL(hipMemcpy(prsr,prsr_d,numSrc*numExtrap*sizeof(hipFloatComplex),hipMemcpyDeviceToHost));
    
    
    CUDA_CALL(hipFree(pt_d));
    CUDA_CALL(hipFree(elem_d));
    CUDA_CALL(hipFree(prsr_d));
    CUDA_CALL(hipFree(pt_extrap_d));
    CUDA_CALL(hipFree(B_d));
    //printf("Completed extrapolation.\n");
    return EXIT_SUCCESS;
}

int GenerateVoxelField(const char* file_path, const float wavNum, const char* src_type, 
        const vec3f* src_loc, const float* mag, const int numSrc, const aarect3d rect, 
        const double len, const char* vox_grid_path, const char* field_grid_path)
{
    int numNod, numElem;
    findNum(file_path,&numNod,&numElem);
    vec3d *nod_d = (vec3d*)malloc(numNod*sizeof(vec3d));
    tri_elem *elem = (tri_elem*)malloc(numElem*sizeof(tri_elem));
    readOBJ(file_path,nod_d,elem);
    
    vec3f *nod_f = (vec3f*)malloc(numNod*sizeof(vec3f));
    vecd2f(nod_d,numNod,nod_f);
    
    int grid_size[3];
    for(int i=0;i<3;i++) {
        grid_size[i] = floor(rect.len[i]/len);
    }
    
    vec3f *pt_extrap = (vec3f*)malloc(grid_size[0]*grid_size[1]*grid_size[2]*sizeof(vec3f));
    vec3f cnr;
    for(int i=0;i<3;i++) {
        cnr.coords[i] = rect.cnr.coords[i];
    }
    // get the center of each voxel as the evaluation point
    for(int z=0;z<grid_size[2];z++) {
        for(int y=0;y<grid_size[1];y++) {
            for(int x=0;x<grid_size[0];x++) {
                int idx = z*grid_size[0]*grid_size[1]+y*grid_size[0]+x;
                pt_extrap[idx].coords[0] = cnr.coords[0]+x*len+len/2;
                pt_extrap[idx].coords[1] = cnr.coords[1]+y*len+len/2;
                pt_extrap[idx].coords[2] = cnr.coords[2]+z*len+len/2;
            }
        }
    }
    HOST_CALL(RectSpaceToOccGridOnGPU(rect,len,nod_d,elem,numElem,vox_grid_path));
    hipFloatComplex *prsr = (hipFloatComplex*)malloc(grid_size[0]*grid_size[1]*grid_size[2]
            *numSrc*sizeof(hipFloatComplex));
    printf("Voxelization completed.\n");
    HOST_CALL(GenerateFieldUsingBEM(nod_f,numNod,elem,numElem,wavNum,src_type,
            src_loc,mag,numSrc,pt_extrap,grid_size[0]*grid_size[1]*grid_size[2],prsr));
    
    for(int i=0;i<numSrc;i++) {
        char temp[4];
        char *result = (char*)malloc((strlen(field_grid_path)+5)*sizeof(char));
        strcpy(result,field_grid_path);
        sprintf(temp,"%d",i);
        strcat(result,temp);
        //printf("result: %s\n",result);
        HOST_CALL(write_field(&prsr[i*grid_size[0]*grid_size[1]*grid_size[2]],grid_size,result));
        free(result);
    }
    
    
    free(nod_f);
    free(nod_d);
    free(elem);
    free(prsr);
    free(pt_extrap);
    
    return EXIT_SUCCESS;
}

int GenLoudnessFieldUsingBEM(const vec3f* nod, const int numNod, const tri_elem* elem, const int numElem,
        const float band[2], const char* src_type, const vec3f* src_loc, const float* mag, 
        const int numSrc, const vec3f* pt_extrap, const int numExtrap, float* loudness)
{
    /*generate loudness fields of multiple sources using the boundary element method
     nod: nodes on the surface of an object
     numNod: number of nodes on the surface of an object
     elem: elements
     numElem: number of elements
     band: lower and upper bounds of angular frequencies
     src_type: type of sound sources
     src_loc: locations of sources
     mag: magnitudes of sources
     numSrc: number of sources
     pt_extrap: extrapolation points
     numExtrap: number of extrapolation points
     loudness: the loudness field*/
    
    // allocate memory for content of each frequency
    hipFloatComplex *prsr = (hipFloatComplex*)malloc(numSrc*numExtrap*sizeof(hipFloatComplex));
    // set loudness field to 0
    memset(loudness,0,numSrc*numExtrap*sizeof(float));
    
    for(int i=0;i<INTORDER;i++) {
        float omega = (band[1]-band[0])/2*intpt[i]+(band[0]+band[1])/2;
        float wavNum = omega/SPEED_SOUND;
        //HOST_CALL(GenerateFieldUsingBEM(nod,numNod,elem,numElem,wavNum,src_type,
        //        src_loc,mag,numSrc,pt_extrap,numExtrap,prsr));
        HOST_CALL(GenerateFieldCheapExtrap(nod,numNod,elem,numElem,wavNum,src_type,
                src_loc,mag,numSrc,pt_extrap,numExtrap,prsr));
        //CUDA_CALL(hipDeviceSynchronize());
        for(int j=0;j<numSrc*numExtrap;j++) {
            loudness[j] += 0.5*intwgt[i]*powf(hipCabsf(prsr[j])/REF_SOUND_PRESSURE,2);
        }
        CUDA_CALL(hipDeviceSynchronize());
    }
    for(int i=0;i<numSrc*numExtrap;i++) {
        loudness[i] = 10*logf(loudness[i])/logf(10);
    }
    free(prsr);
    return EXIT_SUCCESS;
}

int WriteLoudnessGeometry(const char* file_path, const float band[2], const char* src_type, 
        const float* mag, const vec3f* src_loc, const int numSrc, const aarect3d rect, 
        const double len, const char* vox_grid_path, const char* field_grid_path)
{
    /*write the loudness fields and their corresponding occupancy grid of object
     file_path: path of objects
     band: freqeuncy bands in radians
     src_type: the type of sources, "point" or "monopole"
     mag: magnitudes of sources
     vox_grid_path: path of the occupancy grid file
     field_grid_path: path of files of the fields*/
    int numNod, numElem;
    findNum(file_path,&numNod,&numElem);
    vec3d *nod_d = (vec3d*)malloc(numNod*sizeof(vec3d));
    tri_elem *elem = (tri_elem*)malloc(numElem*sizeof(tri_elem));
    readOBJ(file_path,nod_d,elem);
    
    vec3f *nod_f = (vec3f*)malloc(numNod*sizeof(vec3f));
    vecd2f(nod_d,numNod,nod_f);
    
    int grid_size[3];
    for(int i=0;i<3;i++) {
        grid_size[i] = floor(rect.len[i]/len);
    }
    
    vec3f *pt_extrap = (vec3f*)malloc(grid_size[0]*grid_size[1]*grid_size[2]*sizeof(vec3f));
    vec3f cnr;
    for(int i=0;i<3;i++) {
        cnr.coords[i] = rect.cnr.coords[i];
    }
    // get the center of each voxel as the evaluation point
    for(int z=0;z<grid_size[2];z++) {
        for(int y=0;y<grid_size[1];y++) {
            for(int x=0;x<grid_size[0];x++) {
                int idx = z*grid_size[0]*grid_size[1]+y*grid_size[0]+x;
                pt_extrap[idx].coords[0] = cnr.coords[0]+x*len+len/2;
                pt_extrap[idx].coords[1] = cnr.coords[1]+y*len+len/2;
                pt_extrap[idx].coords[2] = cnr.coords[2]+z*len+len/2;
            }
        }
    }
    HOST_CALL(RectSpaceToOccGridOnGPU(rect,len,nod_d,elem,numElem,vox_grid_path));
    
    float *loudness = (float*)malloc(numSrc*grid_size[0]*grid_size[1]*grid_size[2]*sizeof(float));
    if(loudness==NULL) {
        printf("Allocation of loudness failed.\n");
        return EXIT_FAILURE;
    }
    HOST_CALL(GenLoudnessFieldUsingBEM(nod_f,numNod,elem,numElem,band,src_type,src_loc,
            mag,numSrc,pt_extrap,grid_size[0]*grid_size[1]*grid_size[2],loudness));
    //printMat(loudness,1,numSrc*grid_size[0]*grid_size[1]*grid_size[2],1);
    for(int i=0;i<numSrc;i++) {
        char temp[4];
        char *result = (char*)malloc((strlen(field_grid_path)+5)*sizeof(char));
        strcpy(result,field_grid_path);
        sprintf(temp,"_%d",i);
        strcat(result,temp);
        //printf("result: %s\n",result);
        HOST_CALL(write_float_grid(&loudness[i*grid_size[0]*grid_size[1]*grid_size[2]],grid_size,result));
        free(result);
    }
    
    free(loudness);
    free(pt_extrap);
    free(nod_d);
    free(nod_f);
    free(elem);
    return EXIT_SUCCESS;
}

int WriteZSliceVoxLoudness(const char* file_path, const float band[2], const char* src_type, 
        const float* mag, const vec3f* src_loc, const int numSrc, const double zCoord, 
        const double len, const aarect2d rect_2d, const char* vox_grid_path, 
        const char* field_grid_path)
{
    // create a rectangle volume
    aarect3d rect_3d;
    rect_3d.cnr.coords[0] = rect_2d.cnr.coords[0];
    rect_3d.cnr.coords[1] = rect_2d.cnr.coords[1];
    rect_3d.cnr.coords[2] = zCoord-len/2-EPS;
    rect_3d.len[0] = rect_2d.len[0];
    rect_3d.len[1] = rect_2d.len[1];
    rect_3d.len[2] = len+2*EPS;
    
    //print(&rect_3d,1);
    HOST_CALL(WriteLoudnessGeometry(file_path,band,src_type,mag,src_loc,numSrc,
            rect_3d,len,vox_grid_path,field_grid_path));
    return EXIT_SUCCESS;
}