#include "hip/hip_runtime.h"
/*
 * To change this license header, choose License Headers in Project Properties.
 * To change this template file, choose Tools | Templates
 * and open the template in the editor.
 */
#include "numerical.h"
#include "mesh.h"

//air density and speed of sound
__constant__ float density = 1.2041;

__constant__ float speed = 343.21;

//Integral points and weights
__constant__ float INTPT[INTORDER]; 

__constant__ float INTWGT[INTORDER];

int genGaussParams(const int n, float *pt, float *wgt) 
{
    int i, j;
    double t;
    gsl_vector *v = gsl_vector_alloc(n);
    for(i=0;i<n-1;i++) {
        gsl_vector_set(v,i,sqrt(pow(2*(i+1),2)-1));
    }
    for(i=0;i<n-1;i++) {
        t = gsl_vector_get(v,i);
        gsl_vector_set(v,i,(i+1)/t);
    }
    gsl_matrix *A = gsl_matrix_alloc(n,n);
    gsl_matrix *B = gsl_matrix_alloc(n,n);
    for(i=0;i<n;i++) {
        for(j=0;j<n;j++) {
            gsl_matrix_set(A,i,j,0);
            if(i==j) {
                gsl_matrix_set(B,i,j,1);
            } else {
                gsl_matrix_set(B,i,j,0);
            }
        }
    }
    for(i=0;i<n-1;i++) {
        t = gsl_vector_get(v,i);
        gsl_matrix_set(A,i+1,i,t);
        gsl_matrix_set(A,i,i+1,t);
    }
    gsl_eigen_symmv_workspace * wsp = gsl_eigen_symmv_alloc(n);
    HOST_CALL(gsl_eigen_symmv(A,v,B,wsp));
    for(i=0;i<n;i++) {
        pt[i] = gsl_vector_get(v,i);
        t = gsl_matrix_get(B,0,i);
        wgt[i] = 2*pow(t,2);
    }
    gsl_vector_free(v);
    gsl_matrix_free(A);
    gsl_matrix_free(B);
    return EXIT_SUCCESS;
}

int gaussPtsToDevice(const float *evalPt, const float *wgt) 
{
    CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(INTPT),evalPt,INTORDER*sizeof(float),0,hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(INTWGT),wgt,INTORDER*sizeof(float),0,hipMemcpyHostToDevice));
    return EXIT_SUCCESS;
}

void printFltMat(const float *A, const int numRow, const int numCol, const int lda) 
{
    for(int i=0;i<numRow;i++) {
        for(int j=0;j<numCol;j++) {
            printf("%f ",A[IDXC0(i,j,lda)]);
        }
        printf("\n");
    }
}

void printCuFloatComplexMat(const hipFloatComplex *A, const int numRow, const int numCol, 
        const int lda)
{
    for(int i=0;i<numRow;i++) {
        for(int j=0;j<numCol;j++) {
            printf("(%f,%f) ",hipCrealf(A[IDXC0(i,j,lda)]),hipCimagf(A[IDXC0(i,j,lda)]));
        }
        printf("\n");
    }
}

cart_coord_float cartCoordDouble2cartCoordFloat(const cart_coord_double t)
{
    cart_coord_float result;
    for(int i=0;i<3;i++) {
        result.coords[i] = t.coords[i];
    }
    return result;
}

cart_coord_double cartCoordFloat2cartCoordDouble(const cart_coord_float t)
{
    cart_coord_double result;
    for(int i=0;i<3;i++) {
        result.coords[i] = t.coords[i];
    }
    return result;
}

__host__ __device__ float dotProd(const cart_coord_float u, const cart_coord_float v) {
    return u.coords[0]*v.coords[0]+u.coords[1]*v.coords[1]+u.coords[2]*v.coords[2];
}

__host__ __device__ cart_coord_float crossProd(const cart_coord_float u, const cart_coord_float v) {
    cart_coord_float r;
    r.coords[0] = (u.coords[1])*(v.coords[2])-(u.coords[2])*(v.coords[1]);
    r.coords[1] = (u.coords[2])*(v.coords[0])-(u.coords[0])*(v.coords[2]);
    r.coords[2] = (u.coords[0])*(v.coords[1])-(u.coords[1])*(v.coords[0]);
    return r;
}

__host__ __device__ cart_coord_float cartCoordAdd(const cart_coord_float u, const cart_coord_float v)
{
    cart_coord_float result;
    for(int i=0;i<3;i++) {
        result.coords[i] = u.coords[i]+v.coords[i];
    }
    return result;
}

__host__ __device__ cart_coord_float cartCoordSub(const cart_coord_float u, const cart_coord_float v)
{
    cart_coord_float result;
    for(int i=0;i<3;i++) {
        result.coords[i] = u.coords[i]-v.coords[i];
    }
    return result;
}

__host__ __device__ cart_coord_float scalarProd(const float lambda, const cart_coord_float v)
{
    cart_coord_float result;
    for(int i=0;i<3;i++) {
        result.coords[i] = lambda*v.coords[i];
    }
    return result;
}

__host__ __device__ cart_coord_double cartCoordAdd(const cart_coord_double u, const cart_coord_double v)
{
    cart_coord_double result;
    for(int i=0;i<3;i++) {
        result.coords[i] = u.coords[i]+v.coords[i];
    }
    return result;
}

__host__ __device__ cart_coord_double cartCoordSub(const cart_coord_double u, const cart_coord_double v)
{
    cart_coord_double result;
    for(int i=0;i<3;i++) {
        result.coords[i] = u.coords[i]-v.coords[i];
    }
    return result;
}

__host__ __device__ cart_coord_double scalarProd(const double lambda, const cart_coord_double v)
{
    cart_coord_double result;
    for(int i=0;i<3;i++) {
        result.coords[i] = lambda*v.coords[i];
    }
    return result;
}

__host__ __device__ cart_coord_double triCentroid(cart_coord_double nod[3])
{
    cart_coord_double ctr_23 = scalarProd(0.5,cartCoordAdd(nod[1],nod[2]));
    cart_coord_double centroid = cartCoordAdd(nod[0],scalarProd(2.0/3.0,cartCoordSub(ctr_23,nod[0])));
    return centroid;
}

__host__ __device__ bool ray_intersect_triangle(const cart_coord_float O, const cart_coord_float dir, 
        const cart_coord_float nod[3])
{
    /*vert0 is chosen as reference point*/
    cart_coord_float E1, E2;
    E1 = cartCoordSub(nod[1],nod[0]);
    E2 = cartCoordSub(nod[2],nod[0]);
    /*cross product of dir and v0 to v1*/
    cart_coord_float P = crossProd(dir,E2);
    float det = dotProd(P,E1);
    if(abs(det)<EPS) {
        return false;
    }
    /*Computation of parameter u*/
    cart_coord_float T = cartCoordSub(O,nod[0]);
    float u = 1.0f/det*dotProd(P,T);
    if(u<0 || u>1) {
        return false;
    }
    /*Computation of parameter v*/
    cart_coord_float Q = crossProd(T,E1);
    float v = 1.0f/det*dotProd(Q,dir);
    if(v<0 || u+v>1) {
        return false;
    }
    /*Computation of parameter t*/
    float t = 1.0f/det*dotProd(Q,E2);
    if(t<EPS) {
        return false;
    }
    return true;
}

__global__ void rayTrisInt(const cart_coord_float pt_s, const cart_coord_float dir, const cart_coord_float *nod, 
        const tri_elem *elem, const int numElem, bool *flag)
{
    // decides if a point pnt is in a closed surface elem
    int idx = blockDim.x*blockIdx.x+threadIdx.x;
    if(idx<numElem) {
        cart_coord_float pt[3];
        for(int i=0;i<3;i++) {
            pt[i].coords[0] = nod[elem[idx].nodes[i]].coords[0];
            pt[i].coords[1] = nod[elem[idx].nodes[i]].coords[1];
            pt[i].coords[2] = nod[elem[idx].nodes[i]].coords[2];
        }
        flag[idx] = ray_intersect_triangle(pt_s,dir,pt);
    }
}

__global__ void distPntPnts(const cart_coord_float pt, const cart_coord_float *nod, const int numNod, float *dist) {
    int idx = blockDim.x*blockIdx.x + threadIdx.x;
    if(idx < numNod) {
        dist[idx] = __fsqrt_rn((pt.coords[0]-nod[idx].coords[0])*(pt.coords[0]-nod[idx].coords[0])
                +(pt.coords[1]-nod[idx].coords[1])*(pt.coords[1]-nod[idx].coords[1])
                +(pt.coords[2]-nod[idx].coords[2])*(pt.coords[2]-nod[idx].coords[2]));
    }
}

__host__ __device__ float convRand(const float lb, const float ub, const float randNumber) {
    float result = (ub-lb)*randNumber+lb;
    return result;
}

bool inBdry(const bool *flag, const int numFlag) {
    int sum = 0;;
    for(int i=0;i<numFlag;i++) {
        if(flag[i]) {
            sum++;
        }
    }
    if(sum%2==0) {
        return false;
    } else {
        return true;
    }
}

int genCHIEF(const cart_coord_float *pt, const int numPt, const tri_elem *elem, const int numElem, 
        cart_coord_float *pCHIEF, const int numCHIEF) {
    int i, cnt;
    float threshold_inner = 0.0000001;
    float *dist_h = (float*)malloc(numPt*sizeof(float));
    float minDist; //minimum distance between the chief point to all surface nodes
    float *dist_d;
    CUDA_CALL(hipMalloc((void**)&dist_d, numPt*sizeof(float)));
    cart_coord_float dir; 
    
    //transfer the point cloud to GPU
    cart_coord_float *pt_d;
    CUDA_CALL(hipMalloc((void**)&pt_d,numPt*sizeof(cart_coord_float))); //point cloud allocated on device
    CUDA_CALL(hipMemcpy(pt_d,pt,numPt*sizeof(cart_coord_float),hipMemcpyHostToDevice)); //point cloud copied to device
    
    //transfer the element cloud to GPU
    tri_elem *elem_d;
    CUDA_CALL(hipMalloc((void**)&elem_d,numElem*sizeof(tri_elem))); //elements allcoated on device
    CUDA_CALL(hipMemcpy(elem_d,elem,numElem*sizeof(tri_elem),hipMemcpyHostToDevice)); //elements copied to device
    
    //create a flag array on CPU and on GPU
    bool *flag_h = (bool*)malloc(numElem*sizeof(bool));
    bool *flag_d;
    CUDA_CALL(hipMalloc((void**)&flag_d,numElem*sizeof(bool))); //memory for flags allocated on device

    unsigned long long seed = 0;
    int blockWidth = 32;
    int gridWidth;
    float xrand, yrand, zrand, unifRandNum[3];
    cart_coord_float chief;
    
    //Find the bounding box
    float xb[2], yb[2], zb[2];
    findBB(pt,numPt,0,xb,yb,zb);
    
    //create a handle to hiprand
    hiprandGenerator_t gen;
    CURAND_CALL(hiprandCreateGeneratorHost(&gen,HIPRAND_RNG_PSEUDO_DEFAULT)); //construct generator
    cnt = 0; // initialize count for number of points generated
    while(cnt<numCHIEF) {
        do
        {
            //set seed
            CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(gen,seed++));
            CURAND_CALL(hiprandGenerateUniform(gen,unifRandNum,3)); //generate a uniformly distributed random number
            //generate the direction
            for(i=0;i<3;i++) {
                dir.coords[i] = unifRandNum[i];
            }
            //Convert the rand numbers into a point in the bounding box
            xrand = convRand(xb[0],xb[1],unifRandNum[0]);
            yrand = convRand(yb[0],yb[1],unifRandNum[1]);
            zrand = convRand(zb[0],zb[1],unifRandNum[2]);
            chief.coords[0] = xrand;
            chief.coords[1] = yrand;
            chief.coords[2] = zrand;
            //(&chief,1);
            gridWidth = (numElem+blockWidth-1)/blockWidth;
            rayTrisInt<<<gridWidth,blockWidth>>>(chief,dir,pt_d,elem_d,numElem,flag_d);
            gridWidth = (numPt+blockWidth-1)/blockWidth;
            distPntPnts<<<gridWidth,blockWidth>>>(chief,pt_d,numPt,dist_d);
            CUDA_CALL(hipMemcpy(dist_h,dist_d,numPt*sizeof(float),hipMemcpyDeviceToHost));
            //printFltMat(dist_h,1,numPt,1);
            CUDA_CALL(hipMemcpy(flag_h,flag_d,numElem*sizeof(bool),hipMemcpyDeviceToHost));
            minDist = dist_h[0];
            for(i=1;i<numPt;i++) {
                if(dist_h[i]<minDist) {
                    minDist = dist_h[i];
                }
            }
            //printf("The minimum distance is %f, threshold is %f\n",dist_min,threshold_inner);
            //printf("inSurf: %d\n", inSurf(flags_h, numElem));
        } while (!inBdry(flag_h,numElem) || minDist<threshold_inner);
        pCHIEF[cnt] = chief;
        cnt++;
    }
    CURAND_CALL(hiprandDestroyGenerator(gen));
    free(flag_h);
    free(dist_h);
    CUDA_CALL(hipFree(pt_d));
    CUDA_CALL(hipFree(elem_d));
    CUDA_CALL(hipFree(flag_d));
    CUDA_CALL(hipFree(dist_d));
    return EXIT_SUCCESS;
}

inline __device__ void crossNorm(const cart_coord_float a, const cart_coord_float b, cart_coord_float *norm, float *length) 
{
    cart_coord_float c;
    c.coords[0] = a.coords[1]*b.coords[2]-a.coords[2]*b.coords[1];
    c.coords[1] = a.coords[2]*b.coords[0]-a.coords[0]*b.coords[2];
    c.coords[2] = a.coords[0]*b.coords[1]-a.coords[1]*b.coords[0];

    *length = __fsqrt_rn((c.coords[0]*c.coords[0])+(c.coords[1]*c.coords[1])+(c.coords[2]*c.coords[2]));

    norm->coords[0] = c.coords[0] / *length;
    norm->coords[1] = c.coords[1] / *length;
    norm->coords[2] = c.coords[2] / *length;
}

__device__ void g_h_c_nsgl(const float k, const cart_coord_float x, const cart_coord_float p[3], 
        hipFloatComplex gCoeff[3], hipFloatComplex hCoeff[3], float *cCoeff) {
    //Initalization of g, h and c
    //printf("(%f,%f,%f)\n",p[0].coords[0],p[0].coords[1],p[0].coords[2]);
    for(int i=0;i<3;i++) {
        gCoeff[i] = make_hipFloatComplex(0,0);
        hCoeff[i] = make_hipFloatComplex(0,0);
    }
    *cCoeff = 0;
    
    //Local variables
    float eta1, eta2, wn, wm, xi1, xi2, xi3, rho, theta, vertCrossProd, temp, 
            temp_gh[3], omega = k*speed, pPsiLpn2, radius, prpn2;
    cart_coord_float y, normal, rVec;
    hipFloatComplex Psi, pPsipn2;
    crossNorm(
    {
        p[0].coords[0]-p[2].coords[0],p[0].coords[1]-p[2].coords[1],p[0].coords[2]-p[2].coords[2]
    },
    {
        p[1].coords[0]-p[2].coords[0],p[1].coords[1]-p[2].coords[1],p[1].coords[2]-p[2].coords[2]
    },&normal,&vertCrossProd);
    vertCrossProd = vertCrossProd*0.25f;
    //printf("%f\n",normal.coords[0]);
    const float prodRhoOmega = density*omega;
    const float fourPI = 4.0f*PI;
    const float recipFourPI = 1.0f/fourPI;
    //printf("%f\n",k);
    for(int n=0;n<INTORDER;n++) {
        eta2 = INTPT[n];
        wn = INTWGT[n];
        theta = 0.5f+0.5f*eta2;
        for(int m=0;m<INTORDER;m++) {
            eta1 = INTPT[m];
            wm = INTWGT[m];
            rho = 0.5f+0.5f*eta1;
            temp = wn*wm*rho*vertCrossProd;
            
            xi1 = rho*(1-theta);
            xi2 = rho-xi1;
            xi3 = 1-xi1-xi2;
            //printf("xi1 = %f, xi2 = %f\n",xi1,xi2);
            y= {
                p[0].coords[0]*xi1+p[1].coords[0]*xi2+p[2].coords[0]*xi3, 
                p[0].coords[1]*xi1+p[1].coords[1]*xi2+p[2].coords[1]*xi3, 
                p[0].coords[2]*xi1+p[1].coords[2]*xi2+p[2].coords[2]*xi3
            };
            //printf("x: (%f,%f,%f), y: (%f,%f,%f)\n",x.coords[0],x.coords[1],x.coords[2],
            //        y.coords[0],y.coords[1],y.coords[2]);
            rVec = cartCoordSub(y,x);
            radius = __fsqrt_rn(rVec.coords[0]*rVec.coords[0]+rVec.coords[1]*rVec.coords[1]
                    +rVec.coords[2]*rVec.coords[2]);
            //printf("radius = %f\n",radius);
            prpn2 = ((y.coords[0]-x.coords[0])*normal.coords[0]+(y.coords[1]-x.coords[1])*normal.coords[1]
                    +(y.coords[2]-x.coords[2])*normal.coords[2])/radius;
            //printf("prpn2=%f\n",prpn2);
            pPsiLpn2 = -recipFourPI/(radius*radius)*prpn2;
            //printf("%f\n",pPsiLpn2);
            Psi = make_hipFloatComplex(__cosf(-k*radius)/(fourPI*radius),__sinf(-k*radius)/(fourPI*radius));
            pPsipn2 = hipCmulf(Psi,make_hipFloatComplex(-1.0f/radius,-k));
            pPsipn2 = make_hipFloatComplex(prpn2*hipCrealf(pPsipn2),prpn2*hipCimagf(pPsipn2));
            temp_gh[0] = temp*xi1;
            temp_gh[1] = temp*xi2;
            temp_gh[2] = temp*xi3;
            
            gCoeff[0] = hipCaddf(gCoeff[0],make_hipFloatComplex(temp_gh[0]*hipCrealf(Psi),temp_gh[0]*hipCimagf(Psi)));
            gCoeff[1] = hipCaddf(gCoeff[1],make_hipFloatComplex(temp_gh[1]*hipCrealf(Psi),temp_gh[1]*hipCimagf(Psi)));
            gCoeff[2] = hipCaddf(gCoeff[2],make_hipFloatComplex(temp_gh[2]*hipCrealf(Psi),temp_gh[2]*hipCimagf(Psi)));
            
            hCoeff[0] = hipCaddf(hCoeff[0],make_hipFloatComplex(temp_gh[0]*hipCrealf(pPsipn2),temp_gh[0]*hipCimagf(pPsipn2)));
            hCoeff[1] = hipCaddf(hCoeff[1],make_hipFloatComplex(temp_gh[1]*hipCrealf(pPsipn2),temp_gh[1]*hipCimagf(pPsipn2)));
            hCoeff[2] = hipCaddf(hCoeff[2],make_hipFloatComplex(temp_gh[2]*hipCrealf(pPsipn2),temp_gh[2]*hipCimagf(pPsipn2)));
            
            *cCoeff += temp*pPsiLpn2;
        }
    }
    gCoeff[0] = make_hipFloatComplex(-prodRhoOmega*hipCimagf(gCoeff[0]),prodRhoOmega*hipCrealf(gCoeff[0]));
    gCoeff[1] = make_hipFloatComplex(-prodRhoOmega*hipCimagf(gCoeff[1]),prodRhoOmega*hipCrealf(gCoeff[1]));
    gCoeff[2] = make_hipFloatComplex(-prodRhoOmega*hipCimagf(gCoeff[2]),prodRhoOmega*hipCrealf(gCoeff[2]));
}

__device__ void g_h_c_sgl(const float k, const cart_coord_float x_sgl1, const cart_coord_float x_sgl2, 
        const cart_coord_float x_sgl3, const cart_coord_float p[3], 
        hipFloatComplex gCoeff_sgl1[3], hipFloatComplex hCoeff_sgl1[3], float *cCoeff_sgl1,
        hipFloatComplex gCoeff_sgl2[3], hipFloatComplex hCoeff_sgl2[3], float *cCoeff_sgl2,
        hipFloatComplex gCoeff_sgl3[3], hipFloatComplex hCoeff_sgl3[3], float *cCoeff_sgl3) 
{
    //Initalization of g, h and c
    for(int i=0;i<3;i++) {
        gCoeff_sgl1[i] = make_hipFloatComplex(0,0);
        hCoeff_sgl1[i] = make_hipFloatComplex(0,0);
        gCoeff_sgl2[i] = make_hipFloatComplex(0,0);
        hCoeff_sgl2[i] = make_hipFloatComplex(0,0);
        gCoeff_sgl3[i] = make_hipFloatComplex(0,0);
        hCoeff_sgl3[i] = make_hipFloatComplex(0,0);
    }
    *cCoeff_sgl1 = 0;
    *cCoeff_sgl2 = 0;
    *cCoeff_sgl3 = 0;
    
    //Local variables
    float eta1, eta2, wn, wm, xi1_sgl1, xi2_sgl1, xi3_sgl1, xi1_sgl2, xi2_sgl2, xi3_sgl2,
            xi1_sgl3, xi2_sgl3, xi3_sgl3, rho, theta, vertCrossProd, temp, 
            temp_gh[3], omega = k*speed, pPsiLpn2, radius, prpn2;
    cart_coord_float y_sgl1, y_sgl2, y_sgl3, normal, rVec;
    hipFloatComplex Psi, pPsipn2;
    crossNorm(
    {
        p[0].coords[0]-p[2].coords[0],p[0].coords[1]-p[2].coords[1],p[0].coords[2]-p[2].coords[2]
    },
    {
        p[1].coords[0]-p[2].coords[0],p[1].coords[1]-p[2].coords[1],p[1].coords[2]-p[2].coords[2]
    },&normal,&vertCrossProd);
    vertCrossProd = vertCrossProd*0.25f;
    //printf("vert: %f\n",vertCrossProd);
    
    //printf("normal=(%f,%f,%f)\n",normal.coords[0],normal.coords[1],normal.coords[2]);
    const float prodRhoOmega = density*omega;
    const float fourPI = 4.0f*PI;
    const float recipFourPI = 1.0/fourPI;
    //printf("density*omega = %f\n",prodRhoOmega);
    //Compute integrals for g, h and c
    for(int n=0;n<INTORDER;n++) {
        eta2 = INTPT[n];
        wn = INTWGT[n];
        theta = 0.5f+0.5f*eta2;
        for(int m=0;m<INTORDER;m++) {
            eta1 = INTPT[m];
            wm = INTWGT[m];
            rho = 0.5f+0.5f*eta1;
            temp = wn*wm*rho*vertCrossProd;
            
            xi1_sgl3 = rho*(1-theta);
            xi2_sgl3 = rho-xi1_sgl3; //rho*theta
            xi3_sgl3 = 1-xi1_sgl3-xi2_sgl3;
            
            xi1_sgl1 = 1-rho;
            xi2_sgl1 = rho-xi2_sgl3; //rho-rho*theta
            xi3_sgl1 = 1-xi1_sgl1-xi2_sgl1;
            
            xi1_sgl2 = xi2_sgl3; //rho*theta
            xi2_sgl2 = 1-rho;
            xi3_sgl2 = 1-xi1_sgl2-xi2_sgl2;
            
            
            
            //printf("xi1 = %f, xi2 = %f\n",xi1,xi2);
            y_sgl1= {
                p[0].coords[0]*xi1_sgl1+p[1].coords[0]*xi2_sgl1+p[2].coords[0]*xi3_sgl1, 
                p[0].coords[1]*xi1_sgl1+p[1].coords[1]*xi2_sgl1+p[2].coords[1]*xi3_sgl1, 
                p[0].coords[2]*xi1_sgl1+p[1].coords[2]*xi2_sgl1+p[2].coords[2]*xi3_sgl1
            };
            y_sgl2= {
                p[0].coords[0]*xi1_sgl2+p[1].coords[0]*xi2_sgl2+p[2].coords[0]*xi3_sgl2, 
                p[0].coords[1]*xi1_sgl2+p[1].coords[1]*xi2_sgl2+p[2].coords[1]*xi3_sgl2, 
                p[0].coords[2]*xi1_sgl2+p[1].coords[2]*xi2_sgl2+p[2].coords[2]*xi3_sgl2
            };
            y_sgl3= {
                p[0].coords[0]*xi1_sgl3+p[1].coords[0]*xi2_sgl3+p[2].coords[0]*xi3_sgl3, 
                p[0].coords[1]*xi1_sgl3+p[1].coords[1]*xi2_sgl3+p[2].coords[1]*xi3_sgl3, 
                p[0].coords[2]*xi1_sgl3+p[1].coords[2]*xi2_sgl3+p[2].coords[2]*xi3_sgl3
            };
            
            //update coefficients with singularity on node 1
            rVec = cartCoordSub(y_sgl1,x_sgl1);
            radius = sqrtf(rVec.coords[0]*rVec.coords[0]+rVec.coords[1]*rVec.coords[1]+rVec.coords[2]*rVec.coords[2]);
            //printf("radius = %f\n",radius);
            prpn2 = ((y_sgl1.coords[0]-x_sgl1.coords[0])*normal.coords[0]+(y_sgl1.coords[1]-x_sgl1.coords[1])*normal.coords[1]
                    +(y_sgl1.coords[2]-x_sgl1.coords[2])*normal.coords[2])/radius;
            //printf("prpn2=%f\n",prpn2);
            pPsiLpn2 = -recipFourPI/(radius*radius)*prpn2;
            //printf("%f\n",pPsiLpn2);
            Psi = make_hipFloatComplex(__cosf(-k*radius)/(fourPI*radius),__sinf(-k*radius)/(fourPI*radius));
            pPsipn2 = hipCmulf(Psi,make_hipFloatComplex(-1.0f/radius,-k));
            pPsipn2 = make_hipFloatComplex(prpn2*hipCrealf(pPsipn2),prpn2*hipCimagf(pPsipn2));
            temp_gh[0] = temp*xi1_sgl1;
            temp_gh[1] = temp*xi2_sgl1;
            temp_gh[2] = temp*xi3_sgl1;
            
            gCoeff_sgl1[0] = hipCaddf(gCoeff_sgl1[0],make_hipFloatComplex(temp_gh[0]*hipCrealf(Psi),temp_gh[0]*hipCimagf(Psi)));
            gCoeff_sgl1[1] = hipCaddf(gCoeff_sgl1[1],make_hipFloatComplex(temp_gh[1]*hipCrealf(Psi),temp_gh[1]*hipCimagf(Psi)));
            gCoeff_sgl1[2] = hipCaddf(gCoeff_sgl1[2],make_hipFloatComplex(temp_gh[2]*hipCrealf(Psi),temp_gh[2]*hipCimagf(Psi)));
            
            hCoeff_sgl1[0] = hipCaddf(hCoeff_sgl1[0],make_hipFloatComplex(temp_gh[0]*hipCrealf(pPsipn2),temp_gh[0]*hipCimagf(pPsipn2)));
            hCoeff_sgl1[1] = hipCaddf(hCoeff_sgl1[1],make_hipFloatComplex(temp_gh[1]*hipCrealf(pPsipn2),temp_gh[1]*hipCimagf(pPsipn2)));
            hCoeff_sgl1[2] = hipCaddf(hCoeff_sgl1[2],make_hipFloatComplex(temp_gh[2]*hipCrealf(pPsipn2),temp_gh[2]*hipCimagf(pPsipn2)));
            
            *cCoeff_sgl1 += temp*pPsiLpn2;
            
            //update coefficients with singularity on node 2
            rVec = cartCoordSub(y_sgl2,x_sgl2);
            radius = sqrtf(rVec.coords[0]*rVec.coords[0]+rVec.coords[1]*rVec.coords[1]+rVec.coords[2]*rVec.coords[2]);
            //printf("radius = %f\n",radius);
            prpn2 = ((y_sgl1.coords[0]-x_sgl1.coords[0])*normal.coords[0]+(y_sgl1.coords[1]-x_sgl1.coords[1])*normal.coords[1]
                    +(y_sgl1.coords[2]-x_sgl1.coords[2])*normal.coords[2])/radius;
            //printf("prpn2=%f\n",prpn2);
            pPsiLpn2 = -recipFourPI/(radius*radius)*prpn2;
            //printf("%f\n",pPsiLpn2);
            Psi = make_hipFloatComplex(__cosf(-k*radius)/(fourPI*radius),__sinf(-k*radius)/(fourPI*radius));
            pPsipn2 = hipCmulf(Psi,make_hipFloatComplex(-1.0f/radius,-k));
            pPsipn2 = make_hipFloatComplex(prpn2*hipCrealf(pPsipn2),prpn2*hipCimagf(pPsipn2));
            temp_gh[0] = temp*xi1_sgl2;
            temp_gh[1] = temp*xi2_sgl2;
            temp_gh[2] = temp*xi3_sgl2;
            
            gCoeff_sgl2[0] = hipCaddf(gCoeff_sgl2[0],make_hipFloatComplex(temp_gh[0]*hipCrealf(Psi),temp_gh[0]*hipCimagf(Psi)));
            gCoeff_sgl2[1] = hipCaddf(gCoeff_sgl2[1],make_hipFloatComplex(temp_gh[1]*hipCrealf(Psi),temp_gh[1]*hipCimagf(Psi)));
            gCoeff_sgl2[2] = hipCaddf(gCoeff_sgl2[2],make_hipFloatComplex(temp_gh[2]*hipCrealf(Psi),temp_gh[2]*hipCimagf(Psi)));
            
            hCoeff_sgl2[0] = hipCaddf(hCoeff_sgl2[0],make_hipFloatComplex(temp_gh[0]*hipCrealf(pPsipn2),temp_gh[0]*hipCimagf(pPsipn2)));
            hCoeff_sgl2[1] = hipCaddf(hCoeff_sgl2[1],make_hipFloatComplex(temp_gh[1]*hipCrealf(pPsipn2),temp_gh[1]*hipCimagf(pPsipn2)));
            hCoeff_sgl2[2] = hipCaddf(hCoeff_sgl2[2],make_hipFloatComplex(temp_gh[2]*hipCrealf(pPsipn2),temp_gh[2]*hipCimagf(pPsipn2)));
            
            *cCoeff_sgl2 += temp*pPsiLpn2;
            
            //update coefficients with singularity on node 3
            rVec = cartCoordSub(y_sgl3,x_sgl3);
            radius = sqrtf(rVec.coords[0]*rVec.coords[0]+rVec.coords[1]*rVec.coords[1]+rVec.coords[2]*rVec.coords[2]);
            //printf("radius = %f\n",radius);
            prpn2 = ((y_sgl1.coords[0]-x_sgl1.coords[0])*normal.coords[0]+(y_sgl1.coords[1]-x_sgl1.coords[1])*normal.coords[1]
                    +(y_sgl1.coords[2]-x_sgl1.coords[2])*normal.coords[2])/radius;
            //printf("prpn2=%f\n",prpn2);
            pPsiLpn2 = -recipFourPI/(radius*radius)*prpn2;
            //printf("%f\n",pPsiLpn2);
            Psi = make_hipFloatComplex(__cosf(-k*radius)/(fourPI*radius),__sinf(-k*radius)/(fourPI*radius));
            pPsipn2 = hipCmulf(Psi,make_hipFloatComplex(-1.0f/radius,-k));
            pPsipn2 = make_hipFloatComplex(prpn2*hipCrealf(pPsipn2),prpn2*hipCimagf(pPsipn2));
            temp_gh[0] = temp*xi1_sgl3;
            temp_gh[1] = temp*xi2_sgl3;
            temp_gh[2] = temp*xi3_sgl3;
            
            gCoeff_sgl3[0] = hipCaddf(gCoeff_sgl3[0],make_hipFloatComplex(temp_gh[0]*hipCrealf(Psi),temp_gh[0]*hipCimagf(Psi)));
            gCoeff_sgl3[1] = hipCaddf(gCoeff_sgl3[1],make_hipFloatComplex(temp_gh[1]*hipCrealf(Psi),temp_gh[1]*hipCimagf(Psi)));
            gCoeff_sgl3[2] = hipCaddf(gCoeff_sgl3[2],make_hipFloatComplex(temp_gh[2]*hipCrealf(Psi),temp_gh[2]*hipCimagf(Psi)));
            
            hCoeff_sgl3[0] = hipCaddf(hCoeff_sgl3[0],make_hipFloatComplex(temp_gh[0]*hipCrealf(pPsipn2),temp_gh[0]*hipCimagf(pPsipn2)));
            hCoeff_sgl3[1] = hipCaddf(hCoeff_sgl3[1],make_hipFloatComplex(temp_gh[1]*hipCrealf(pPsipn2),temp_gh[1]*hipCimagf(pPsipn2)));
            hCoeff_sgl3[2] = hipCaddf(hCoeff_sgl3[2],make_hipFloatComplex(temp_gh[2]*hipCrealf(pPsipn2),temp_gh[2]*hipCimagf(pPsipn2)));
            
            *cCoeff_sgl3 += temp*pPsiLpn2;
        }
    }
    gCoeff_sgl1[0] = make_hipFloatComplex(-prodRhoOmega*hipCimagf(gCoeff_sgl1[0]),prodRhoOmega*hipCrealf(gCoeff_sgl1[0]));
    gCoeff_sgl1[1] = make_hipFloatComplex(-prodRhoOmega*hipCimagf(gCoeff_sgl1[1]),prodRhoOmega*hipCrealf(gCoeff_sgl1[1]));
    gCoeff_sgl1[2] = make_hipFloatComplex(-prodRhoOmega*hipCimagf(gCoeff_sgl1[2]),prodRhoOmega*hipCrealf(gCoeff_sgl1[2]));
    
    gCoeff_sgl2[0] = make_hipFloatComplex(-prodRhoOmega*hipCimagf(gCoeff_sgl2[0]),prodRhoOmega*hipCrealf(gCoeff_sgl2[0]));
    gCoeff_sgl2[1] = make_hipFloatComplex(-prodRhoOmega*hipCimagf(gCoeff_sgl2[1]),prodRhoOmega*hipCrealf(gCoeff_sgl2[1]));
    gCoeff_sgl2[2] = make_hipFloatComplex(-prodRhoOmega*hipCimagf(gCoeff_sgl2[2]),prodRhoOmega*hipCrealf(gCoeff_sgl2[2]));
    
    gCoeff_sgl3[0] = make_hipFloatComplex(-prodRhoOmega*hipCimagf(gCoeff_sgl3[0]),prodRhoOmega*hipCrealf(gCoeff_sgl3[0]));
    gCoeff_sgl3[1] = make_hipFloatComplex(-prodRhoOmega*hipCimagf(gCoeff_sgl3[1]),prodRhoOmega*hipCrealf(gCoeff_sgl3[1]));
    gCoeff_sgl3[2] = make_hipFloatComplex(-prodRhoOmega*hipCimagf(gCoeff_sgl3[2]),prodRhoOmega*hipCrealf(gCoeff_sgl3[2]));
}

__host__ __device__ hipFloatComplex ptSrc(const float k, const float amp, const cart_coord_float srcLoc, const cart_coord_float evalLoc)
{
    float fourPI = 4.0f*PI;
    cart_coord_float rVec = cartCoordSub(evalLoc,srcLoc);
    float radius = sqrtf(rVec.coords[0]*rVec.coords[0]+rVec.coords[1]*rVec.coords[1]+rVec.coords[2]*rVec.coords[2]);
    return make_hipFloatComplex(amp*cosf(-k*radius)/(fourPI*radius),amp*sinf(-k*radius)/(fourPI*radius));
}

__host__ __device__ hipFloatComplex dirSrc(const float k, const float strength, const cart_coord_float dir, const cart_coord_float evalLoc)
{
    float theta = -k*dotProd(dir,evalLoc);
    return make_hipFloatComplex(strength*cosf(theta),strength*sinf(theta));
}

// compute non-singular relationship between points and elements
__global__ void atomicPtsElems_nsgl(const float k, const cart_coord_float *pt, const int numNod, 
        const int idxPntStart, const int idxPntEnd, const tri_elem *elem, const int numElem, 
        hipFloatComplex *A, const int lda, hipFloatComplex *B, const int numSrc, const int ldb) {
    int xIdx = blockIdx.x*blockDim.x+threadIdx.x; //Index for points
    int yIdx = blockIdx.y*blockDim.y+threadIdx.y; //Index for elements
    //The thread with indices xIdx and yIdx process the point xIdx and elem yIdx
    if(xIdx>=idxPntStart && xIdx<=idxPntEnd && yIdx<numElem && xIdx!=elem[yIdx].nodes[0] 
            && xIdx!=elem[yIdx].nodes[1] && xIdx!=elem[yIdx].nodes[2]) {
        int i, j;
        hipFloatComplex hCoeff[3], gCoeff[3], bc, pCoeffs[3], temp;
        float cCoeff;
        cart_coord_float triNod[3];
        triNod[0] = pt[elem[yIdx].nodes[0]];
        triNod[1] = pt[elem[yIdx].nodes[1]];
        triNod[2] = pt[elem[yIdx].nodes[2]];
        g_h_c_nsgl(k,pt[xIdx],triNod,gCoeff,hCoeff,&cCoeff);
        
        //Update the A matrix
        bc = hipCdivf(elem[yIdx].bc[0],elem[yIdx].bc[1]);
        for(i=0;i<3;i++) {
            pCoeffs[i] = hipCsubf(hCoeff[i],hipCmulf(bc,gCoeff[i]));
        }
        
        for(i=0;i<3;i++) {
            //atomicFloatComplexAdd(&A[IDXC0(xIdx,elem[yIdx].nodes[i],lda)],pCoeffs[i]);
            atomicAdd(&A[IDXC0(xIdx,elem[yIdx].nodes[i],lda)].x,hipCrealf(pCoeffs[i]));
            atomicAdd(&A[IDXC0(xIdx,elem[yIdx].nodes[i],lda)].y,hipCimagf(pCoeffs[i]));
        }
        
        //Update from C coefficients
        if(xIdx<numNod) {
            //atomicFloatComplexSub(&A[IDXC0(xIdx,xIdx,lda)],make_hipFloatComplex(cCoeff,0));
            atomicAdd(&A[IDXC0(xIdx,xIdx,lda)].x,-cCoeff);
        }
        
        //Update the B matrix
        bc = hipCdivf(elem[yIdx].bc[2],elem[yIdx].bc[1]);
        //printf("bc: \n");
        //printComplexMatrix(&bc,1,1,1);
        for(i=0;i<numSrc;i++) {
            for(j=0;j<3;j++) {
                //atomicFloatComplexSub(&B[IDXC0(xIdx,i,ldb)],hipCmulf(bc,gCoeff[j]));
                temp = hipCmulf(bc,gCoeff[j]);
                atomicAdd(&B[IDXC0(xIdx,i,ldb)].x,-hipCrealf(temp));
                atomicAdd(&B[IDXC0(xIdx,i,ldb)].y,-hipCimagf(temp));
            }
        }
    }
}

__global__ void atomicPtsElems_sgl(const float k, const cart_coord_float *pt, const tri_elem *elem, 
        const int numElem, hipFloatComplex *A, const int lda, hipFloatComplex *B, 
        const int numSrc, const int ldb) {
    int idx = blockIdx.x*blockDim.x+threadIdx.x;
    if(idx < numElem) {
        int i, j;
        hipFloatComplex hCoeff_sgl1[3], hCoeff_sgl2[3], hCoeff_sgl3[3], 
                gCoeff_sgl1[3], gCoeff_sgl2[3], gCoeff_sgl3[3], pCoeffs_sgl1[3], 
                pCoeffs_sgl2[3], pCoeffs_sgl3[3], bc, temp;
        float cCoeff_sgl1, cCoeff_sgl2, cCoeff_sgl3;
        
        cart_coord_float nod[3];
        for(i=0;i<3;i++) {
            nod[i] = pt[elem[idx].nodes[i]];
        }
        // Compute h and g coefficients
        g_h_c_sgl(k,pt[elem[idx].nodes[0]],pt[elem[idx].nodes[1]],pt[elem[idx].nodes[2]],
                nod,gCoeff_sgl1,hCoeff_sgl1,&cCoeff_sgl1,gCoeff_sgl2,hCoeff_sgl2,&cCoeff_sgl2,
                gCoeff_sgl3,hCoeff_sgl3,&cCoeff_sgl3);
        
        //Compute p coefficients
        bc = hipCdivf(elem[idx].bc[0],elem[idx].bc[1]);
        for(j=0;j<3;j++) {
            pCoeffs_sgl1[j] = hipCsubf(hCoeff_sgl1[j],hipCmulf(bc,gCoeff_sgl1[j]));
            pCoeffs_sgl2[j] = hipCsubf(hCoeff_sgl2[j],hipCmulf(bc,gCoeff_sgl2[j]));
            pCoeffs_sgl3[j] = hipCsubf(hCoeff_sgl3[j],hipCmulf(bc,gCoeff_sgl3[j]));
        }
        
        //Update matrix A using pCoeffs
        for(j=0;j<3;j++) {
            //atomicFloatComplexAdd(&A[IDXC0(elem[idx].nodes[0],elem[idx].nodes[j],lda)],
            //        pCoeffs_sgl1[j]);
            atomicAdd(&A[IDXC0(elem[idx].nodes[0],elem[idx].nodes[j],lda)].x,
                    hipCrealf(pCoeffs_sgl1[j]));
            atomicAdd(&A[IDXC0(elem[idx].nodes[0],elem[idx].nodes[j],lda)].y,
                    hipCimagf(pCoeffs_sgl1[j]));
            //atomicFloatComplexAdd(&A[IDXC0(elem[idx].nodes[1],elem[idx].nodes[j],lda)],
            //        pCoeffs_sgl2[j]);
            atomicAdd(&A[IDXC0(elem[idx].nodes[1],elem[idx].nodes[j],lda)].x,
                    hipCrealf(pCoeffs_sgl2[j]));
            atomicAdd(&A[IDXC0(elem[idx].nodes[1],elem[idx].nodes[j],lda)].y,
                    hipCimagf(pCoeffs_sgl2[j]));
            //atomicFloatComplexAdd(&A[IDXC0(elem[idx].nodes[2],elem[idx].nodes[j],lda)],
            //        pCoeffs_sgl3[j]);
            atomicAdd(&A[IDXC0(elem[idx].nodes[2],elem[idx].nodes[j],lda)].x,
                    hipCrealf(pCoeffs_sgl3[j]));
            atomicAdd(&A[IDXC0(elem[idx].nodes[2],elem[idx].nodes[j],lda)].y,
                    hipCimagf(pCoeffs_sgl3[j]));
        }
        
        //atomicFloatComplexSub(&A[IDXC0(elem[idx].nodes[0],elem[idx].nodes[0],lda)],
        //        make_hipFloatComplex(cCoeff_sgl1,0));
        atomicAdd(&A[IDXC0(elem[idx].nodes[0],elem[idx].nodes[0],lda)].x,
                -cCoeff_sgl1);
        //atomicFloatComplexSub(&A[IDXC0(elem[idx].nodes[1],elem[idx].nodes[1],lda)],
        //        make_hipFloatComplex(cCoeff_sgl2,0));
        atomicAdd(&A[IDXC0(elem[idx].nodes[1],elem[idx].nodes[1],lda)].x,
                -cCoeff_sgl2);
        //atomicFloatComplexSub(&A[IDXC0(elem[idx].nodes[2],elem[idx].nodes[2],lda)],
        //        make_hipFloatComplex(cCoeff_sgl3,0));
        atomicAdd(&A[IDXC0(elem[idx].nodes[2],elem[idx].nodes[2],lda)].x,
                -cCoeff_sgl3);
        
        //Update matrix B using g Coefficients
        bc = hipCdivf(elem[idx].bc[2],elem[idx].bc[1]);
        for(i=0;i<numSrc;i++) {
            for(j=0;j<3;j++) {
                //atomicFloatComplexSub(&B[IDXC0(elem[idx].nodes[0],i,ldb)],
                //        hipCmulf(bc,gCoeff_sgl1[j]));
                temp = hipCmulf(bc,gCoeff_sgl1[j]);
                atomicAdd(&B[IDXC0(elem[idx].nodes[0],i,ldb)].x,-hipCrealf(temp));
                atomicAdd(&B[IDXC0(elem[idx].nodes[0],i,ldb)].y,-hipCimagf(temp));
                //atomicFloatComplexSub(&B[IDXC0(elem[idx].nodes[1],i,ldb)],
                //        hipCmulf(bc,gCoeff_sgl2[j]));
                temp = hipCmulf(bc,gCoeff_sgl2[j]);
                atomicAdd(&B[IDXC0(elem[idx].nodes[1],i,ldb)].x,-hipCrealf(temp));
                atomicAdd(&B[IDXC0(elem[idx].nodes[1],i,ldb)].y,-hipCimagf(temp));
                //atomicFloatComplexSub(&B[IDXC0(elem[idx].nodes[2],i,ldb)],
                //        hipCmulf(bc,gCoeff_sgl3[j]));
                temp = hipCmulf(bc,gCoeff_sgl3[j]);
                atomicAdd(&B[IDXC0(elem[idx].nodes[2],i,ldb)].x,-hipCrealf(temp));
                atomicAdd(&B[IDXC0(elem[idx].nodes[2],i,ldb)].y,-hipCimagf(temp));
            }
        }
    }
}

int atomicGenSystem(const float k, const tri_elem *elem, const int numElem, 
        const cart_coord_float *nod, const int numNod, const cart_coord_float *chief, const int numCHIEF, 
        const cart_coord_float *src, const int numSrc, hipFloatComplex *A, const int lda, 
        hipFloatComplex *B, const int ldb) {
    int i, j;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    //Move elements to GPU
    tri_elem *elem_d;
    CUDA_CALL(hipMalloc(&elem_d,numElem*sizeof(tri_elem)));
    CUDA_CALL(hipMemcpy(elem_d,elem,numElem*sizeof(tri_elem),hipMemcpyHostToDevice));
    
    //Move points to GPU
    cart_coord_float *pt_h = (cart_coord_float*)malloc((numNod+numCHIEF)*sizeof(cart_coord_float));
    for(i=0;i<numNod;i++) {
        pt_h[i] = nod[i];
    }
    for(i=0;i<numCHIEF;i++) {
        pt_h[numNod+i] = chief[i];
    }
    
    cart_coord_float *pt_d;
    CUDA_CALL(hipMalloc(&pt_d,(numNod+numCHIEF)*sizeof(cart_coord_float)));
    CUDA_CALL(hipMemcpy(pt_d,pt_h,(numNod+numCHIEF)*sizeof(cart_coord_float),hipMemcpyHostToDevice));
    
    //Initialization of A
    for(i=0;i<numNod+numCHIEF;i++) {
        for(j=0;j<numNod;j++) {
            if(i==j) {
                A[IDXC0(i,j,lda)] = make_hipFloatComplex(1,0);
            } else {
                A[IDXC0(i,j,lda)] = make_hipFloatComplex(0,0);
            }
        }
    }
    
    //Initialization of B
    for(i=0;i<numNod+numCHIEF;i++) {
        for(j=0;j<numSrc;j++) {
            B[IDXC0(i,j,ldb)] = ptSrc(k,STRENGTH,src[j],pt_h[i]);
        }
    }
    
    hipFloatComplex *A_d, *B_d;
    CUDA_CALL(hipMalloc(&A_d,(numNod+numCHIEF)*numNod*sizeof(hipFloatComplex)));
    CUDA_CALL(hipMemcpy(A_d,A,(numNod+numCHIEF)*numNod*sizeof(hipFloatComplex),hipMemcpyHostToDevice));
    
    CUDA_CALL(hipMalloc(&B_d,(numNod+numCHIEF)*numSrc*sizeof(hipFloatComplex)));
    CUDA_CALL(hipMemcpy(B_d,B,(numNod+numCHIEF)*numSrc*sizeof(hipFloatComplex),hipMemcpyHostToDevice));
    
    int xNumBlocks, xWidth = 16, yNumBlocks, yWidth = 16;
    xNumBlocks = (numNod+numCHIEF+xWidth-1)/xWidth;
    yNumBlocks = (numElem+yWidth-1)/yWidth;
    dim3 gridLayout, blockLayout;
    gridLayout.x = xNumBlocks;
    gridLayout.y = yNumBlocks;
    
    blockLayout.x = xWidth;
    blockLayout.y = yWidth;
    
    hipEventRecord(start);
    atomicPtsElems_nsgl<<<gridLayout,blockLayout>>>(k,pt_d,numNod,0,numNod+numCHIEF-1,
            elem_d,numElem,A_d,lda,B_d,numSrc,ldb);
    
    //CUDA_CALL(hipMemcpy(A,A_d,(numNod+numCHIEF)*numNod*sizeof(hipFloatComplex),hipMemcpyDeviceToHost));
    //printCuFloatComplexMat(A,numNod+numCHIEF,numNod,numNod+numCHIEF);
    atomicPtsElems_sgl<<<yNumBlocks,yWidth>>>(k,pt_d,elem_d,numElem,A_d,lda,B_d,numSrc,ldb);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds,start,stop);
    printf("Elapsed system generation time: %f milliseconds.\n",milliseconds);
    CUDA_CALL(hipMemcpy(A,A_d,(numNod+numCHIEF)*numNod*sizeof(hipFloatComplex),hipMemcpyDeviceToHost));
    CUDA_CALL(hipMemcpy(B,B_d,(numNod+numCHIEF)*numSrc*sizeof(hipFloatComplex),hipMemcpyDeviceToHost));
    
    
    CUDA_CALL(hipFree(A_d));
    CUDA_CALL(hipFree(B_d));
    CUDA_CALL(hipFree(elem_d));
    CUDA_CALL(hipFree(pt_d));
    
    return EXIT_SUCCESS;
}

int qrSolver(const hipFloatComplex *A, const int mA, const int nA, const int ldA, 
        hipFloatComplex *B, const int nB, const int ldB) {
    hipEvent_t start, stop;
    CUDA_CALL(hipEventCreate(&start));
    CUDA_CALL(hipEventCreate(&stop));
    hipsolverHandle_t cusolverH = NULL;
    CUSOLVER_CALL(hipsolverDnCreate(&cusolverH));
    
    
    hipFloatComplex *A_d;
    CUDA_CALL(hipMalloc(&A_d,ldA*nA*sizeof(hipFloatComplex)));
    CUDA_CALL(hipMemcpy(A_d,A,ldA*nA*sizeof(hipFloatComplex),hipMemcpyHostToDevice));
    
    hipFloatComplex *B_d;
    CUDA_CALL(hipMalloc(&B_d,ldB*nB*sizeof(hipFloatComplex)));
    CUDA_CALL(hipMemcpy(B_d,B,ldB*nB*sizeof(hipFloatComplex),hipMemcpyHostToDevice));
    
    //A = QR
    int lwork;
    CUSOLVER_CALL(hipsolverDnCgeqrf_bufferSize(cusolverH,mA,nA,A_d,ldA,&lwork));
    
    hipFloatComplex *workspace_d;
    CUDA_CALL(hipMalloc(&workspace_d,lwork*sizeof(hipFloatComplex)));
    hipFloatComplex *tau_d;
    CUDA_CALL(hipMalloc(&tau_d,max(mA,nA)*sizeof(hipFloatComplex)));
    int *deviceInfo_d, deviceInfo;
    CUDA_CALL(hipMalloc(&deviceInfo_d,sizeof(int)));
    
    CUDA_CALL(hipEventRecord(start));
    CUSOLVER_CALL(hipsolverDnCgeqrf(cusolverH,mA,nA,A_d,ldA,tau_d,workspace_d,lwork,
            deviceInfo_d));
    CUDA_CALL(hipMemcpy(&deviceInfo,deviceInfo_d,sizeof(int),hipMemcpyDeviceToHost));
    
    //B = (Q^H)*B
    CUSOLVER_CALL(hipsolverDnCunmqr(cusolverH,HIPBLAS_SIDE_LEFT,HIPBLAS_OP_C,mA,nB,
            nA,A_d,ldA,tau_d,B_d,ldB,workspace_d,lwork,deviceInfo_d));
    CUDA_CALL(hipMemcpy(&deviceInfo,deviceInfo_d,sizeof(int),hipMemcpyDeviceToHost));
    
    //Solve Rx = B
    hipFloatComplex alpha = make_hipFloatComplex(1,0);
    hipblasHandle_t cublasH;
    CUBLAS_CALL(hipblasCreate(&cublasH));
    CUBLAS_CALL(hipblasCtrsm(cublasH,HIPBLAS_SIDE_LEFT,HIPBLAS_FILL_MODE_UPPER,
            HIPBLAS_OP_N,HIPBLAS_DIAG_NON_UNIT,nA,nB,&alpha,A_d,ldA,B_d,ldB));
    CUDA_CALL(hipEventRecord(stop));
    
    CUDA_CALL(hipMemcpy(B,B_d,ldB*nB*sizeof(hipFloatComplex),hipMemcpyDeviceToHost));
    CUDA_CALL(hipEventSynchronize(stop));
    
    float milliseconds = 0;
    CUDA_CALL(hipEventElapsedTime(&milliseconds,start,stop));
    printf("Elapsed system solving time: %f milliseconds.\n",milliseconds);
    CUDA_CALL(hipEventDestroy(start));
    CUDA_CALL(hipEventDestroy(stop));
    CUDA_CALL(hipFree(A_d));
    CUDA_CALL(hipFree(B_d));
    CUDA_CALL(hipFree(tau_d));
    CUDA_CALL(hipFree(workspace_d));
    CUDA_CALL(hipFree(deviceInfo_d));
    CUBLAS_CALL(hipblasDestroy(cublasH));
    CUSOLVER_CALL(hipsolverDnDestroy(cusolverH));
    
    return EXIT_SUCCESS;
}

int bemSolver_pt(const float k, const tri_elem *elem, const int numElem, 
        const cart_coord_float *nod, const int numNod, const cart_coord_float *chief, const int numCHIEF, 
        const cart_coord_float *src, const int numSrc, hipFloatComplex *B, const int ldb)
{
    int i, j;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    //Move elements to GPU
    tri_elem *elem_d;
    CUDA_CALL(hipMalloc(&elem_d,numElem*sizeof(tri_elem)));
    CUDA_CALL(hipMemcpy(elem_d,elem,numElem*sizeof(tri_elem),hipMemcpyHostToDevice));
    
    //Move points to GPU
    // cart_coord_float *pt_h = (cart_coord_float*)malloc((numNod+numCHIEF)*sizeof(cart_coord_float));
    // for(i=0;i<numNod;i++) {
    //     pt_h[i] = nod[i];
    // }
    // for(i=0;i<numCHIEF;i++) {
    //     pt_h[numNod+i] = chief[i];
    // }
    
    cart_coord_float *pt_d;
    CUDA_CALL(hipMalloc(&pt_d, (numNod + numCHIEF) * sizeof(cart_coord_float)));
    CUDA_CALL(hipMemcpy(pt_d, nod, numNod * sizeof(cart_coord_float),hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpy(pt_d + numNod, chief, numCHIEF * sizeof(cart_coord_float),hipMemcpyHostToDevice));
    
    CUDA_CALL(hipEventRecord(start));
    //Generate the system
    hipFloatComplex *A = (hipFloatComplex*)malloc((numNod+numCHIEF)*numNod*sizeof(hipFloatComplex));
    
    memset(A,0,(numNod+numCHIEF)*numNod*sizeof(hipFloatComplex));
    memset(B,0,(numNod+numCHIEF)*numSrc*sizeof(hipFloatComplex));

    for(i=0;i<numNod;i++) 
    {
        A[IDXC0(i,i,numNod+numCHIEF)] = make_hipFloatComplex(1,0);
    }
    
    //Initialization of B
    for(i=0;i<numNod+numCHIEF;i++) 
    {
        for(j=0;j<numSrc;j++) 
        {
            if(i < numNod)
                B[IDXC0(i,j,ldb)] = ptSrc(k,STRENGTH,src[j],nod[i]);
            else
                B[IDXC0(i,j,ldb)] = ptSrc(k,STRENGTH,src[j],chief[i - numNod]);
        }
    }
    
    hipFloatComplex *A_d, *B_d;
    CUDA_CALL(hipMalloc(&A_d,(numNod+numCHIEF)*numNod*sizeof(hipFloatComplex)));
    CUDA_CALL(hipMemcpy(A_d,A,(numNod+numCHIEF)*numNod*sizeof(hipFloatComplex),hipMemcpyHostToDevice));
    
    CUDA_CALL(hipMalloc(&B_d,(numNod+numCHIEF)*numSrc*sizeof(hipFloatComplex)));
    CUDA_CALL(hipMemcpy(B_d,B,(numNod+numCHIEF)*numSrc*sizeof(hipFloatComplex),hipMemcpyHostToDevice));
    
    int xNumBlocks, xWidth = 16, yNumBlocks, yWidth = 16;
    xNumBlocks = (numNod+numCHIEF+xWidth-1)/xWidth;
    yNumBlocks = (numElem+yWidth-1)/yWidth;
    dim3 gridLayout, blockLayout;
    gridLayout.x = xNumBlocks;
    gridLayout.y = yNumBlocks;
    
    blockLayout.x = xWidth;
    blockLayout.y = yWidth;
    
    atomicPtsElems_nsgl<<<gridLayout,blockLayout>>>(k,pt_d,numNod,0,numNod+numCHIEF-1,
            elem_d,numElem,A_d,numNod+numCHIEF,B_d,numSrc,ldb);
    atomicPtsElems_sgl<<<yNumBlocks,yWidth>>>(k,pt_d,elem_d,numElem,A_d,numNod+numCHIEF,
            B_d,numSrc,ldb);
    
    //Solving the system
    hipsolverHandle_t cusolverH = NULL;
    CUSOLVER_CALL(hipsolverDnCreate(&cusolverH));
    
    //A = QR
    int lwork;
    CUSOLVER_CALL(hipsolverDnCgeqrf_bufferSize(cusolverH,numNod+numCHIEF,numNod,A_d
            ,numNod+numCHIEF,&lwork));
    
    hipFloatComplex *workspace_d;
    CUDA_CALL(hipMalloc(&workspace_d,lwork*sizeof(hipFloatComplex)));
    hipFloatComplex *tau_d;
    CUDA_CALL(hipMalloc(&tau_d,(numNod+numCHIEF)*sizeof(hipFloatComplex)));
    int *deviceInfo_d, deviceInfo;
    CUDA_CALL(hipMalloc(&deviceInfo_d,sizeof(int)));
    
    
    CUSOLVER_CALL(hipsolverDnCgeqrf(cusolverH,numNod+numCHIEF,numNod,A_d,numNod+numCHIEF,
            tau_d,workspace_d,lwork,deviceInfo_d));
    CUDA_CALL(hipMemcpy(&deviceInfo,deviceInfo_d,sizeof(int),hipMemcpyDeviceToHost));
    
    //B = (Q^H)*B
    CUSOLVER_CALL(hipsolverDnCunmqr(cusolverH,HIPBLAS_SIDE_LEFT,HIPBLAS_OP_C,numNod+numCHIEF,numSrc,
            numNod,A_d,numNod+numCHIEF,tau_d,B_d,ldb,workspace_d,lwork,deviceInfo_d));
    CUDA_CALL(hipMemcpy(&deviceInfo,deviceInfo_d,sizeof(int),hipMemcpyDeviceToHost));
    
    //Solve Rx = B
    hipFloatComplex alpha = make_hipFloatComplex(1,0);
    hipblasHandle_t cublasH;
    CUBLAS_CALL(hipblasCreate(&cublasH));
    CUBLAS_CALL(hipblasCtrsm(cublasH,HIPBLAS_SIDE_LEFT,HIPBLAS_FILL_MODE_UPPER,
            HIPBLAS_OP_N,HIPBLAS_DIAG_NON_UNIT,numNod,numSrc,&alpha,A_d,numNod+numCHIEF,B_d,ldb));
    CUDA_CALL(hipEventRecord(stop));
    CUDA_CALL(hipEventSynchronize(stop));
    CUDA_CALL(hipMemcpy(B,B_d,ldb*numSrc*sizeof(hipFloatComplex),hipMemcpyDeviceToHost));
    
    float milliseconds = 0;
    CUDA_CALL(hipEventElapsedTime(&milliseconds,start,stop));
    printf("Elapsed system solving time: %f milliseconds.\n",milliseconds);
    
    //release memory
    CUDA_CALL(hipEventDestroy(start));
    CUDA_CALL(hipEventDestroy(stop));
    CUDA_CALL(hipFree(A_d));
    CUDA_CALL(hipFree(B_d));
    CUDA_CALL(hipFree(tau_d));
    CUDA_CALL(hipFree(workspace_d));
    CUDA_CALL(hipFree(deviceInfo_d));
    CUBLAS_CALL(hipblasDestroy(cublasH));
    CUSOLVER_CALL(hipsolverDnDestroy(cusolverH));
    CUDA_CALL(hipFree(elem_d));
    CUDA_CALL(hipFree(pt_d));
    free(A);
    return EXIT_SUCCESS;
}

int bemSolver_dir(const float k, const tri_elem *elem, const int numElem, 
        const cart_coord_float *nod, const int numNod, const cart_coord_float *chief, const int numCHIEF, 
        const cart_coord_float *dir, const int numSrc, hipFloatComplex *B, const int ldb)
{
    int i, j;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    //Move elements to GPU
    tri_elem *elem_d;
    CUDA_CALL(hipMalloc(&elem_d,numElem*sizeof(tri_elem)));
    CUDA_CALL(hipMemcpy(elem_d,elem,numElem*sizeof(tri_elem),hipMemcpyHostToDevice));
    
    //Move points to GPU
    // cart_coord_float *pt_h = (cart_coord_float*)malloc((numNod+numCHIEF)*sizeof(cart_coord_float));
    // for(i=0;i<numNod;i++) {
    //     pt_h[i] = nod[i];
    // }
    // for(i=0;i<numCHIEF;i++) {
    //     pt_h[numNod+i] = chief[i];
    // }
    
    cart_coord_float *pt_d;
    CUDA_CALL(hipMalloc(&pt_d,(numNod+numCHIEF)*sizeof(cart_coord_float)));
    CUDA_CALL(hipMemcpy(pt_d,nod,numNod*sizeof(cart_coord_float),hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpy(pt_d+numNod,chief,numCHIEF*sizeof(cart_coord_float),hipMemcpyHostToDevice));
    
    CUDA_CALL(hipEventRecord(start));
    //Generate the system
    hipFloatComplex *A = (hipFloatComplex*)malloc((numNod+numCHIEF)*numNod*sizeof(hipFloatComplex));
    memset(A,0,(numNod+numCHIEF)*numNod*sizeof(hipFloatComplex));

    for(i=0;i<numNod;i++) 
    {
        A[IDXC0(i,i,numNod+numCHIEF)] = make_hipFloatComplex(1,0);
    }
    
    //Initialization of B
    for(i=0;i<numNod+numCHIEF;i++) 
    {
        for(j=0;j<numSrc;j++) 
        {
            if(i < numNod)
                //B[IDXC0(i,j,ldb)] = ptSrc(k,STRENGTH,src[j],nod[i]);
                B[IDXC0(i,j,ldb)] = dirSrc(k,STRENGTH,dir[j],nod[i]);
            else
                //B[IDXC0(i,j,ldb)] = ptSrc(k,STRENGTH,src[j],chief[i - numNod]);
                B[IDXC0(i,j,ldb)] = dirSrc(k,STRENGTH,dir[j],chief[i-numNod]);
        }
    }
    
    hipFloatComplex *A_d, *B_d;
    CUDA_CALL(hipMalloc(&A_d,(numNod+numCHIEF)*numNod*sizeof(hipFloatComplex)));
    CUDA_CALL(hipMemcpy(A_d,A,(numNod+numCHIEF)*numNod*sizeof(hipFloatComplex),hipMemcpyHostToDevice));
    
    CUDA_CALL(hipMalloc(&B_d,(numNod+numCHIEF)*numSrc*sizeof(hipFloatComplex)));
    CUDA_CALL(hipMemcpy(B_d,B,(numNod+numCHIEF)*numSrc*sizeof(hipFloatComplex),hipMemcpyHostToDevice));
    
    int xNumBlocks, xWidth = 16, yNumBlocks, yWidth = 16;
    xNumBlocks = (numNod+numCHIEF+xWidth-1)/xWidth;
    yNumBlocks = (numElem+yWidth-1)/yWidth;
    dim3 gridLayout, blockLayout;
    gridLayout.x = xNumBlocks;
    gridLayout.y = yNumBlocks;
    
    blockLayout.x = xWidth;
    blockLayout.y = yWidth;
    
    atomicPtsElems_nsgl<<<gridLayout,blockLayout>>>(k,pt_d,numNod,0,numNod+numCHIEF-1,
            elem_d,numElem,A_d,numNod+numCHIEF,B_d,numSrc,ldb);
    atomicPtsElems_sgl<<<yNumBlocks,yWidth>>>(k,pt_d,elem_d,numElem,A_d,numNod+numCHIEF,
            B_d,numSrc,ldb);
    
    //Solving the system
    hipsolverHandle_t cusolverH = NULL;
    CUSOLVER_CALL(hipsolverDnCreate(&cusolverH));
    
    //A = QR
    int lwork;
    CUSOLVER_CALL(hipsolverDnCgeqrf_bufferSize(cusolverH,numNod+numCHIEF,numNod,A_d
            ,numNod+numCHIEF,&lwork));
    
    hipFloatComplex *workspace_d;
    CUDA_CALL(hipMalloc(&workspace_d,lwork*sizeof(hipFloatComplex)));
    hipFloatComplex *tau_d;
    CUDA_CALL(hipMalloc(&tau_d,(numNod+numCHIEF)*sizeof(hipFloatComplex)));
    int *deviceInfo_d, deviceInfo;
    CUDA_CALL(hipMalloc(&deviceInfo_d,sizeof(int)));
    
    
    CUSOLVER_CALL(hipsolverDnCgeqrf(cusolverH,numNod+numCHIEF,numNod,A_d,numNod+numCHIEF,
            tau_d,workspace_d,lwork,deviceInfo_d));
    CUDA_CALL(hipMemcpy(&deviceInfo,deviceInfo_d,sizeof(int),hipMemcpyDeviceToHost));
    
    //B = (Q^H)*B
    CUSOLVER_CALL(hipsolverDnCunmqr(cusolverH,HIPBLAS_SIDE_LEFT,HIPBLAS_OP_C,numNod+numCHIEF,numSrc,
            numNod,A_d,numNod+numCHIEF,tau_d,B_d,ldb,workspace_d,lwork,deviceInfo_d));
    CUDA_CALL(hipMemcpy(&deviceInfo,deviceInfo_d,sizeof(int),hipMemcpyDeviceToHost));
    
    //Solve Rx = B
    hipFloatComplex alpha = make_hipFloatComplex(1,0);
    hipblasHandle_t cublasH;
    CUBLAS_CALL(hipblasCreate(&cublasH));
    CUBLAS_CALL(hipblasCtrsm(cublasH,HIPBLAS_SIDE_LEFT,HIPBLAS_FILL_MODE_UPPER,
            HIPBLAS_OP_N,HIPBLAS_DIAG_NON_UNIT,numNod,numSrc,&alpha,A_d,numNod+numCHIEF,B_d,ldb));
    CUDA_CALL(hipEventRecord(stop));
    CUDA_CALL(hipEventSynchronize(stop));
    CUDA_CALL(hipMemcpy(B,B_d,ldb*numSrc*sizeof(hipFloatComplex),hipMemcpyDeviceToHost));
    
    float milliseconds = 0;
    CUDA_CALL(hipEventElapsedTime(&milliseconds,start,stop));
    printf("Elapsed system solving time: %f milliseconds.\n",milliseconds);
    
    //release memory
    CUDA_CALL(hipEventDestroy(start));
    CUDA_CALL(hipEventDestroy(stop));
    CUDA_CALL(hipFree(A_d));
    CUDA_CALL(hipFree(B_d));
    CUDA_CALL(hipFree(tau_d));
    CUDA_CALL(hipFree(workspace_d));
    CUDA_CALL(hipFree(deviceInfo_d));
    CUBLAS_CALL(hipblasDestroy(cublasH));
    CUSOLVER_CALL(hipsolverDnDestroy(cusolverH));
    CUDA_CALL(hipFree(elem_d));
    CUDA_CALL(hipFree(pt_d));
    free(A);
    return EXIT_SUCCESS;
}

__host__ gsl_complex gsl_sf_bessel_hl(const int l, const double s)
{
    double x = gsl_sf_bessel_jl(l,s);
    double y = gsl_sf_bessel_yl(l,s);
    gsl_complex z = gsl_complex_rect(x,y);
    return z;
}

double jprime(const int n, const double r)
{
    double result;
    if(n == 0) {
        result = -gsl_sf_bessel_jl(1,r);
    } else {
        result = gsl_sf_bessel_jl(n-1,r)-(n+1)*gsl_sf_bessel_jl(n,r)/r;
    }
    return result;
}

gsl_complex hprime(const int n, const double r)
{
    gsl_complex result;
    if(n == 0) {
        result = gsl_complex_negative(gsl_sf_bessel_hl(1,r));
    } else {
        result = gsl_complex_sub(gsl_sf_bessel_hl(n-1,r),gsl_complex_mul_real(gsl_sf_bessel_hl(n,r),(n+1)/r));
    }
    return result;
}

__host__ __device__ sph_coord_float cart2sph(const cart_coord_float s)
{
    sph_coord_float temp;
    temp.coords[0] = sqrtf(powf(s.coords[0],2)+powf(s.coords[1],2)+powf(s.coords[2],2));
    temp.coords[1] = acosf(s.coords[2]/(temp.coords[0]));
    temp.coords[2] = atan2f(s.coords[1],s.coords[0]);
    return temp;
}

__host__ __device__ cart_coord_float sph2cart(const sph_coord_float s)
{
    float r = s.coords[0], theta = s.coords[1], phi = s.coords[2];
    float x = r*sinf(theta)*cosf(phi), y = r*sinf(theta)*sinf(phi), z = r*cosf(theta);
    cart_coord_float result;
    result.coords[0] = x;
    result.coords[1] = y;
    result.coords[2] = z;
    return result;
}

__host__ __device__ sph_coord_double cart2sph(const cart_coord_double s)
{
    sph_coord_double temp;
    temp.coords[0] = sqrt(pow(s.coords[0],2)+pow(s.coords[1],2)+pow(s.coords[2],2));
    temp.coords[1] = acos(s.coords[2]/(temp.coords[0]));
    temp.coords[2] = atan2(s.coords[1],s.coords[0]);
    return temp;
}

__host__ __device__ cart_coord_double sph2cart(const sph_coord_double s)
{
    double r = s.coords[0], theta = s.coords[1], phi = s.coords[2];
    double x = r*sin(theta)*cos(phi), y = r*sin(theta)*sin(phi), z = r*cos(theta);
    cart_coord_double result;
    result.coords[0] = x;
    result.coords[1] = y;
    result.coords[2] = z;
    return result;
}

void rigidSpherePlaneMultipleEval(const cart_coord_float *pt, const int numPt, 
        const double a, const double wavNum, const double strength)
{
    gsl_complex *p = (gsl_complex*)malloc(numPt*sizeof(gsl_complex));
    sph_coord_float tempCoord;
    gsl_complex result;
    //double temp;
    //const int truncNum = 30;
    for(int i=0;i<numPt;i++)
    {
        tempCoord = cart2sph(pt[i]);
        result = rigid_sphere_plane(wavNum,strength,a,tempCoord.coords[0],tempCoord.coords[1]);
        p[i] = result;
        printf("(%.8f,%.8f)\n",GSL_REAL(p[i]),GSL_IMAG(p[i]));
    }
    free(p);
}

gsl_complex rigid_sphere_plane(const double wavNum, const double strength, const double a, 
        const double r, const double theta)
{
    gsl_complex result = gsl_complex_rect(0,0), temp_c;
    const int numTrunc = 70;
    for(int n=0;n<numTrunc;n++)
    {
        temp_c = gsl_complex_div(gsl_complex_rect(jprime(n,wavNum*a),0),hprime(n,wavNum*a));
        temp_c = gsl_complex_mul(temp_c,gsl_sf_bessel_hl(n,wavNum*r));
        temp_c = gsl_complex_sub(gsl_complex_rect(gsl_sf_bessel_jl(n,wavNum*r),0),temp_c);
        temp_c = gsl_complex_mul(gsl_complex_pow_real(gsl_complex_rect(0,1),n),temp_c);
        temp_c = gsl_complex_mul_real(temp_c,2*n+1);
        temp_c = gsl_complex_mul_real(temp_c,gsl_sf_legendre_Pl(n,cos(theta)));
        result = gsl_complex_add(result,temp_c);
    }
    result = gsl_complex_mul_real(result,strength);
    return result;
}

gsl_complex rigid_sphere_point(const double wavNum, const double strength, const double rs, 
        const double a, const cart_coord_double y)
{
    const int truncNum = 100;
    const cart_coord_double src = {0,0,rs};
    cart_coord_double temp_cart_coord = cartCoordSub(y,src);
    sph_coord_double temp_sph_coord = cart2sph(temp_cart_coord);
    double R = temp_sph_coord.coords[0];
    temp_sph_coord = cart2sph(y);
    double r = temp_sph_coord.coords[0];
    double theta = temp_sph_coord.coords[1];
    gsl_complex result = gsl_complex_rect(strength*cos(wavNum*R)/(4*PI*R),strength*sin(wavNum*R)/(4*PI*R));
    for(int n=0;n<truncNum;n++) {
        gsl_complex temp[2];
        double t = (n+0.5)*jprime(n,wavNum*a)*wavNum*strength/(2*PI)*gsl_sf_legendre_Pl(n,cos(theta));
        temp[0] = gsl_complex_rect(0,t);
        temp[1] = gsl_complex_mul(gsl_sf_bessel_hl(n,wavNum*rs),gsl_sf_bessel_hl(n,wavNum*r));
        temp[0] = gsl_complex_mul(temp[0],temp[1]);
        temp[0] = gsl_complex_div(temp[0],hprime(n,wavNum*a));
        result = gsl_complex_sub(result,temp[0]);
    }
    return result;
}

__device__ hipFloatComplex extrapolation_dir(const float wavNum, const cart_coord_float x, 
        const tri_elem* elem, const int numElem, const cart_coord_float* pt, 
        const hipFloatComplex* p, const float strength, const cart_coord_float dir)
{
    hipFloatComplex result = dirSrc(wavNum,strength,dir,x);
    hipFloatComplex temp;
    for(int i=0;i<numElem;i++) {
        cart_coord_float nod[3];
        for(int j=0;j<3;j++) {
            nod[j] = pt[elem[i].nodes[j]];
        }
        hipFloatComplex gCoeff[3], hCoeff[3]; 
        float cCoeff[3];
        g_h_c_nsgl(wavNum,x,nod,gCoeff,hCoeff,cCoeff);
        for(int j=0;j<3;j++) {
            temp = hipCdivf(elem[i].bc[2],elem[i].bc[1]);
            temp = hipCmulf(temp,gCoeff[j]);
            result = hipCsubf(result,temp);
            temp = hipCdivf(elem[i].bc[0],elem[i].bc[1]);
            temp = hipCmulf(temp,gCoeff[j]);
            temp = hipCsubf(hCoeff[j],temp);
            temp = hipCmulf(temp,p[elem[i].nodes[j]]);
            result = hipCsubf(result,temp);
        }
    }
    return result;
}

__global__ void extrapolation_dirs(const float wavNum, const cart_coord_float* expPt, const int numExpPt,
        const tri_elem* elem, const int numElem, const cart_coord_float* pt, const hipFloatComplex* p, 
        const float strength, const cart_coord_float dir, hipFloatComplex *p_exp)
{
    int idx = blockIdx.x*blockDim.x+threadIdx.x;
    if(idx < numExpPt) {
        p_exp[idx] = extrapolation_dir(wavNum,expPt[idx],elem,numElem,pt,p,strength,dir);
    }
}

int extrapolation_dirs_single_source(const float wavNum, const cart_coord_float* expPt, const int numExpPt, 
        const tri_elem* elem, const int numElem, const cart_coord_float* pt, const int numPt, 
        const hipFloatComplex* p, const float strength, const cart_coord_float dir, hipFloatComplex *pExp)
{
    int width = 16, numBlock = (numExpPt+width-1)/width;
    
    // allocate memory on GPU and copy data to GPU memory
    cart_coord_float *expPt_d, *pt_d;
    tri_elem *elem_d;
    hipFloatComplex *p_d, *pExp_d;
    
    CUDA_CALL(hipMalloc(&expPt_d,numExpPt*sizeof(cart_coord_float)));
    CUDA_CALL(hipMemcpy(expPt_d,expPt,numExpPt*sizeof(cart_coord_float),hipMemcpyHostToDevice));
    
    CUDA_CALL(hipMalloc(&pt_d,numPt*sizeof(cart_coord_float)));
    CUDA_CALL(hipMemcpy(pt_d,pt,numPt*sizeof(cart_coord_float),hipMemcpyHostToDevice));
    
    CUDA_CALL(hipMalloc(&elem_d,numElem*sizeof(tri_elem)));
    CUDA_CALL(hipMemcpy(elem_d,elem,numElem*sizeof(tri_elem),hipMemcpyHostToDevice));
    
    CUDA_CALL(hipMalloc(&p_d,numPt*sizeof(hipFloatComplex)));
    CUDA_CALL(hipMemcpy(p_d,p,numPt*sizeof(hipFloatComplex),hipMemcpyHostToDevice));
    
    CUDA_CALL(hipMalloc(&pExp_d,numExpPt*sizeof(hipFloatComplex)));
    
    extrapolation_dirs<<<numBlock,width>>>(wavNum,expPt_d,numExpPt,elem_d,numElem,pt_d,p_d,
            strength,dir,pExp_d);
    
    CUDA_CALL(hipMemcpy(pExp,pExp_d,numExpPt*sizeof(hipFloatComplex),hipMemcpyDeviceToHost));
    
    CUDA_CALL(hipFree(expPt_d));
    CUDA_CALL(hipFree(pt_d));
    CUDA_CALL(hipFree(elem_d));
    CUDA_CALL(hipFree(p_d));
    CUDA_CALL(hipFree(pExp_d));
    
    return EXIT_SUCCESS;
}