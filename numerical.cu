#include "hip/hip_runtime.h"
/*
 * To change this license header, choose License Headers in Project Properties.
 * To change this template file, choose Tools | Templates
 * and open the template in the editor.
 */
#include "numerical.h"
#include "octree.h"
#include "mesh.h"
#include <hip/hip_math_constants.h>
#include <hipsolver.h>

//air density and speed of sound
__constant__ float density = 1.2041;

__constant__ float speed = 343.21;

//Integral points and weights
__constant__ float INTPT[INTORDER]; 

__constant__ float INTWGT[INTORDER];
/*
int genGaussParams(const int n, float* pt, float* wgt) 
{
    int i, j;
    double t;
    gsl_vector *v = gsl_vector_alloc(n);
    for(i=0;i<n-1;i++) {
        gsl_vector_set(v,i,sqrt(pow(2*(i+1),2)-1));
    }
    for(i=0;i<n-1;i++) {
        t = gsl_vector_get(v,i);
        gsl_vector_set(v,i,(i+1)/t);
    }
    gsl_matrix *A = gsl_matrix_alloc(n,n);
    gsl_matrix *B = gsl_matrix_alloc(n,n);
    for(i=0;i<n;i++) {
        for(j=0;j<n;j++) {
            gsl_matrix_set(A,i,j,0);
            if(i==j) {
                gsl_matrix_set(B,i,j,1);
            } else {
                gsl_matrix_set(B,i,j,0);
            }
        }
    }
    for(i=0;i<n-1;i++) {
        t = gsl_vector_get(v,i);
        gsl_matrix_set(A,i+1,i,t);
        gsl_matrix_set(A,i,i+1,t);
    }
    gsl_eigen_symmv_workspace * wsp = gsl_eigen_symmv_alloc(n);
    HOST_CALL(gsl_eigen_symmv(A,v,B,wsp));
    for(i=0;i<n;i++) {
        pt[i] = gsl_vector_get(v,i);
        t = gsl_matrix_get(B,0,i);
        wgt[i] = 2*pow(t,2);
    }
    gsl_vector_free(v);
    gsl_matrix_free(A);
    gsl_matrix_free(B);
    return EXIT_SUCCESS;
}
*/

int cuGenGaussParams(const int n, float* pt, float* wgt)
{
    hipsolverHandle_t handle;
    CUSOLVER_CALL(hipsolverDnCreate(&handle));
    
    // allocate memory for vector v of length n
    float *v = (float*)malloc(n*sizeof(float));
    
    // set the vector v
    for(int i=0;i<n-1;i++) {
        v[i] = sqrt(pow(2*(i+1),2)-1);
    }
    for(int i=0;i<n-1;i++) {
        float t = v[i];
        v[i] = (i+1)/t;
    }
    //printf("The vector v is set properly.\n");
    
    float *A = (float*)malloc(n*n*sizeof(float));
    memset(A,0,n*n*sizeof(float));
    
    // set up matrix A
    for(int i=0;i<n-1;i++) {
        float t = v[i];
        A[IDXC0(i+1,i,n)] = t;
        A[IDXC0(i,i+1,n)] = t;
    }
    
    //printf("The matrix A is set properly.\n");
    
    float *A_d, *Lambda_d;
    CUDA_CALL(hipMalloc(&A_d,n*n*sizeof(float)));
    //printf("A_d allocated.\n");
    CUDA_CALL(hipMemcpy(A_d,A,n*n*sizeof(float),hipMemcpyHostToDevice));
    //printf("A copied to A_d.\n");
    CUDA_CALL(hipMalloc(&Lambda_d,n*sizeof(float)));
    //printf("Lambda_d allocated successfully.\n");
    
    int lwork;
    hipsolverEigMode_t jobz = HIPSOLVER_EIG_MODE_VECTOR; // compute eigenvalues and eigenvectors.
    hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;
    CUSOLVER_CALL(hipsolverDnSsyevd_bufferSize(handle,jobz,
            uplo,n,A_d,n,Lambda_d,&lwork));
    //printf("Buffer is set up.\n");
    float *work_d;
    CUDA_CALL(hipMalloc(&work_d,lwork*sizeof(float)));
    int *devInfo;
    CUDA_CALL(hipMalloc(&devInfo,sizeof(int)));
    CUSOLVER_CALL(hipsolverDnSsyevd(handle,jobz,uplo,n,A_d,n,Lambda_d,work_d,lwork,devInfo));
    //printf("Eigenvalues and eigenvectors found.\n");
    float *Lambda = (float*)malloc(n*sizeof(float));
    CUDA_CALL(hipMemcpy(A,A_d,n*n*sizeof(float),hipMemcpyDeviceToHost));
    CUDA_CALL(hipMemcpy(Lambda,Lambda_d,n*sizeof(float),hipMemcpyDeviceToHost));
    
    memcpy(pt,Lambda,n*sizeof(float));
    for(int i=0;i<n;i++) {
        float t = A[IDXC0(0,i,n)];
        wgt[i] = 2*pow(t,2);
    }
    
    if(A_d) {
        CUDA_CALL(hipFree(A_d));
    }
    if(Lambda_d) {
        CUDA_CALL(hipFree(Lambda_d));
    }
    if(work_d) {
        CUDA_CALL(hipFree(work_d));
    }
    if(devInfo) {
        CUDA_CALL(hipFree(devInfo));
    }
    if(handle) {
        CUSOLVER_CALL(hipsolverDnDestroy(handle));
    }
    
    free(v);
    free(Lambda);
    free(A);
    
    
    return EXIT_SUCCESS;
}

int gaussPtsToDevice(const float *evalPt, const float *wgt) 
{
    CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(INTPT),evalPt,INTORDER*sizeof(float),0,hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(INTWGT),wgt,INTORDER*sizeof(float),0,hipMemcpyHostToDevice));
    return EXIT_SUCCESS;
}

void print_float_mat(const float *A, const int numRow, const int numCol, const int lda) 
{
    for(int i=0;i<numRow;i++) {
        for(int j=0;j<numCol;j++) {
            printf("%f ",A[IDXC0(i,j,lda)]);
        }
        printf("\n");
    }
}

void print_cuFloatComplex_mat(const hipFloatComplex *A, const int numRow, const int numCol, 
        const int lda)
{
    for(int i=0;i<numRow;i++) {
        for(int j=0;j<numCol;j++) {
            printf("(%f,%f) ",hipCrealf(A[IDXC0(i,j,lda)]),hipCimagf(A[IDXC0(i,j,lda)]));
        }
        printf("\n");
    }
}

__host__ __device__ void printVec(const vec3f* pt, const int numPt)
{
    for(int i=0;i<numPt;i++) {
        printf("(%f,%f,%f), ",pt[i].coords[0],pt[i].coords[1],pt[i].coords[2]);
    }
    printf("\n");
}

__host__ __device__ void printVec(const vec3d* pt, const int numPt)
{
    for(int i=0;i<numPt;i++) {
        printf("(%f,%f,%f), ",pt[i].coords[0],pt[i].coords[1],pt[i].coords[2]);
    }
    printf("\n");
}

__host__ __device__ float vecDotMul(const vec3f u, const vec3f v)
{
    return u.coords[0]*v.coords[0]+u.coords[1]*v.coords[1]+u.coords[2]*v.coords[2];
}

__host__ __device__ double vecDotMul(const vec3d u, const vec3d v)
{
    return u.coords[0]*v.coords[0]+u.coords[1]*v.coords[1]+u.coords[2]*v.coords[2];
}

__host__ __device__ float vecDotMul(const vec2f u, const vec2f v)
{
    return u.coords[0]*v.coords[0]+u.coords[1]+v.coords[1];
}

__host__ __device__ double vecDotMul(const vec2d u, const vec2d v)
{
    return u.coords[0]*v.coords[0]+u.coords[1]+v.coords[1];
}

__host__ __device__ float vecNorm(const vec3f v)
{
    return sqrtf(vecDotMul(v,v));
}

__host__ __device__ double vecNorm(const vec3d v)
{
    return sqrt(vecDotMul(v,v));
}

__host__ __device__ vec3f vecCrossMul(const vec3f a, const vec3f b)
{
    vec3f temp;
    temp.coords[0] = a.coords[1]*b.coords[2]-a.coords[2]*b.coords[1];
    temp.coords[1] = -(a.coords[0]*b.coords[2]-a.coords[2]*b.coords[0]);
    temp.coords[2] = a.coords[0]*b.coords[1]-a.coords[1]*b.coords[0];
    return temp;
}

__host__ __device__ vec3d vecCrossMul(const vec3d a, const vec3d b)
{
    vec3d temp;
    temp.coords[0] = a.coords[1]*b.coords[2]-a.coords[2]*b.coords[1];
    temp.coords[1] = -(a.coords[0]*b.coords[2]-a.coords[2]*b.coords[0]);
    temp.coords[2] = a.coords[0]*b.coords[1]-a.coords[1]*b.coords[0];
    return temp;
}

__host__ __device__ vec3d nrmlzVec(const vec3d v)
{
    double nrm = sqrt(vecDotMul(v,v));
    return scaVecMul(1.0/nrm,v);
}

__host__ __device__ vec3f nrmlzVec(const vec3f v)
{
    float nrm = sqrt(vecDotMul(v,v));
    return scaVecMul(1.0/nrm,v);
}

__host__ __device__ int vecEqual(const vec3f v1, const vec3f v2)
{
    vec3f v = vecSub(v1,v2);
    if(vecNorm(v) < EPS) {
        return 1;
    } else {
        return 0;
    }
}

__host__ __device__ int vecEqual(const vec3d v1, const vec3d v2)
{
    vec3d v = vecSub(v1,v2);
    if(vecNorm(v) < EPS) {
        return 1;
    } else {
        return 0;
    }
}

__host__ __device__ vec3f scaVecMul(const float lambda, const vec3f v)
{
    vec3f result;
    for(int i=0;i<3;i++) {
        result.coords[i] = lambda*v.coords[i];
    }
    return result;
}

__host__ __device__ vec3d scaVecMul(const double lambda, const vec3d v)
{
    vec3d result;
    for(int i=0;i<3;i++) {
        result.coords[i] = lambda*v.coords[i];
    }
    return result;
}

__host__ __device__ vec2d scaVecMul(const double lambda, const vec2d v)
{
    vec2d result;
    for(int i=0;i<2;i++) {
        result.coords[i] = lambda*v.coords[i];
    }
    return result;
}

__host__ __device__ vec2f scaVecMul(const float lambda, const vec2f v)
{
    vec2f result;
    for(int i=0;i<2;i++) {
        result.coords[i] = lambda*v.coords[i];
    }
    return result;
}

__host__ __device__ vec3f vecAdd(const vec3f u, const vec3f v)
{
    vec3f result;
    for(int i=0;i<3;i++) {
        result.coords[i] = u.coords[i]+v.coords[i];
    }
    return result;
}

__host__ __device__ vec3d vecAdd(const vec3d u, const vec3d v)
{
    vec3d result;
    for(int i=0;i<3;i++) {
        result.coords[i] = u.coords[i]+v.coords[i];
    }
    return result;
}

__host__ __device__ vec2d vecAdd(const vec2d u, const vec2d v)
{
    vec2d result;
    for(int i=0;i<2;i++) {
        result.coords[i] = u.coords[i]+v.coords[i];
    }
    return result;
}

__host__ __device__ vec2f vecAdd(const vec2f u, const vec2f v)
{
    vec2f result;
    for(int i=0;i<2;i++) {
        result.coords[i] = u.coords[i]+v.coords[i];
    }
    return result;
}

__host__ __device__ vec3d vecSub(const vec3d u, const vec3d v)
{
    vec3d result;
    for(int i=0;i<3;i++) {
        result.coords[i] = u.coords[i]-v.coords[i];
    }
    return result;
}

__host__ __device__ vec3f vecSub(const vec3f u, const vec3f v)
{
    vec3f result;
    for(int i=0;i<3;i++) {
        result.coords[i] = u.coords[i]-v.coords[i];
    }
    return result;
}

__host__ __device__ vec2d vecSub(const vec2d u, const vec2d v)
{
    vec2d result;
    for(int i=0;i<2;i++) {
        result.coords[i] = u.coords[i]-v.coords[i];
    }
    return result;
}

__host__ __device__ vec2f vecSub(const vec2f u, const vec2f v)
{
    vec2f result;
    for(int i=0;i<2;i++) {
        result.coords[i] = u.coords[i]-v.coords[i];
    }
    return result;
}

__host__ __device__ vec3d triCentroid(vec3d nod[3])
{
    vec3d ctr_23 = scaVecMul(0.5,vecAdd(nod[1],nod[2]));
    vec3d centroid = vecAdd(nod[0],scaVecMul(2.0/3.0,vecSub(ctr_23,nod[0])));
    return centroid;
}

__host__ __device__ bool ray_intersect_triangle(const vec3f O, const vec3f dir, 
        const vec3f nod[3])
{
    /*vert0 is chosen as reference point*/
    vec3f E1, E2;
    E1 = vecSub(nod[1],nod[0]);
    E2 = vecSub(nod[2],nod[0]);
    /*cross product of dir and v0 to v1*/
    vec3f P = vecCrossMul(dir,E2);
    float det = vecDotMul(P,E1);
    if(abs(det)<EPS) {
        return false;
    }
    /*Computation of parameter u*/
    vec3f T = vecSub(O,nod[0]);
    float u = 1.0f/det*vecDotMul(P,T);
    if(u<0 || u>1) {
        return false;
    }
    /*Computation of parameter v*/
    vec3f Q = vecCrossMul(T,E1);
    float v = 1.0f/det*vecDotMul(Q,dir);
    if(v<0 || u+v>1) {
        return false;
    }
    /*Computation of parameter t*/
    float t = 1.0f/det*vecDotMul(Q,E2);
    if(t<EPS) {
        return false;
    }
    return true;
}

__global__ void rayTrisInt(const vec3f pt_s, const vec3f dir, const vec3f *nod, 
        const tri_elem *elem, const int numElem, bool *flag)
{
    // decides if a point pnt is in a closed surface elem
    int idx = blockDim.x*blockIdx.x+threadIdx.x;
    if(idx<numElem) {
        vec3f pt[3];
        for(int i=0;i<3;i++) {
            pt[i].coords[0] = nod[elem[idx].nod[i]].coords[0];
            pt[i].coords[1] = nod[elem[idx].nod[i]].coords[1];
            pt[i].coords[2] = nod[elem[idx].nod[i]].coords[2];
        }
        flag[idx] = ray_intersect_triangle(pt_s,dir,pt);
    }
}

__global__ void distPntPnts(const vec3f pt, const vec3f *nod, const int numNod, float *dist) {
    int idx = blockDim.x*blockIdx.x + threadIdx.x;
    if(idx < numNod) {
        dist[idx] = __fsqrt_rn((pt.coords[0]-nod[idx].coords[0])*(pt.coords[0]-nod[idx].coords[0])
                +(pt.coords[1]-nod[idx].coords[1])*(pt.coords[1]-nod[idx].coords[1])
                +(pt.coords[2]-nod[idx].coords[2])*(pt.coords[2]-nod[idx].coords[2]));
    }
}

__host__ __device__ float convRand(const float lb, const float ub, const float randNumber) {
    float result = (ub-lb)*randNumber+lb;
    return result;
}

bool inBdry(const bool *flag, const int numFlag) {
    int sum = 0;;
    for(int i=0;i<numFlag;i++) {
        if(flag[i]) {
            sum++;
        }
    }
    if(sum%2==0) {
        return false;
    } else {
        return true;
    }
}

int genCHIEF(const vec3f *pt, const int numPt, const tri_elem *elem, const int numElem, 
        vec3f *pCHIEF, const int numCHIEF) {
    int i, cnt;
    float threshold_inner = 0.0000001;
    float *dist_h = (float*)malloc(numPt*sizeof(float));
    float minDist; //minimum distance between the chief point to all surface nod
    float *dist_d;
    CUDA_CALL(hipMalloc((void**)&dist_d, numPt*sizeof(float)));
    vec3f dir; 
    
    //transfer the point cloud to GPU
    vec3f *pt_d;
    CUDA_CALL(hipMalloc((void**)&pt_d,numPt*sizeof(vec3f))); //point cloud allocated on device
    CUDA_CALL(hipMemcpy(pt_d,pt,numPt*sizeof(vec3f),hipMemcpyHostToDevice)); //point cloud copied to device
    
    //transfer the element cloud to GPU
    tri_elem *elem_d;
    CUDA_CALL(hipMalloc((void**)&elem_d,numElem*sizeof(tri_elem))); //elements allcoated on device
    CUDA_CALL(hipMemcpy(elem_d,elem,numElem*sizeof(tri_elem),hipMemcpyHostToDevice)); //elements copied to device
    
    //create a flag array on CPU and on GPU
    bool *flag_h = (bool*)malloc(numElem*sizeof(bool));
    bool *flag_d;
    CUDA_CALL(hipMalloc((void**)&flag_d,numElem*sizeof(bool))); //memory for flags allocated on device

    unsigned long long seed = 0;
    int blockWidth = 32;
    int gridWidth;
    float xrand, yrand, zrand, unifRandNum[3];
    vec3f chief;
    
    //Find the bounding box
    float xb[2], yb[2], zb[2];
    findBB(pt,numPt,0,xb,yb,zb);
    
    //create a handle to hiprand
    hiprandGenerator_t gen;
    CURAND_CALL(hiprandCreateGeneratorHost(&gen,HIPRAND_RNG_PSEUDO_DEFAULT)); //construct generator
    cnt = 0; // initialize count for number of points generated
    while(cnt<numCHIEF) {
        do
        {
            //set seed
            CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(gen,seed++));
            CURAND_CALL(hiprandGenerateUniform(gen,unifRandNum,3)); //generate a uniformly distributed random number
            //generate the direction
            for(i=0;i<3;i++) {
                dir.coords[i] = unifRandNum[i];
            }
            //Convert the rand numbers into a point in the bounding box
            xrand = convRand(xb[0],xb[1],unifRandNum[0]);
            yrand = convRand(yb[0],yb[1],unifRandNum[1]);
            zrand = convRand(zb[0],zb[1],unifRandNum[2]);
            chief.coords[0] = xrand;
            chief.coords[1] = yrand;
            chief.coords[2] = zrand;
            //(&chief,1);
            gridWidth = (numElem+blockWidth-1)/blockWidth;
            rayTrisInt<<<gridWidth,blockWidth>>>(chief,dir,pt_d,elem_d,numElem,flag_d);
            gridWidth = (numPt+blockWidth-1)/blockWidth;
            distPntPnts<<<gridWidth,blockWidth>>>(chief,pt_d,numPt,dist_d);
            CUDA_CALL(hipMemcpy(dist_h,dist_d,numPt*sizeof(float),hipMemcpyDeviceToHost));
            //printFltMat(dist_h,1,numPt,1);
            CUDA_CALL(hipMemcpy(flag_h,flag_d,numElem*sizeof(bool),hipMemcpyDeviceToHost));
            minDist = dist_h[0];
            for(i=1;i<numPt;i++) {
                if(dist_h[i]<minDist) {
                    minDist = dist_h[i];
                }
            }
            //printf("The minimum distance is %f, threshold is %f\n",dist_min,threshold_inner);
            //printf("inSurf: %d\n", inSurf(flags_h, numElem));
        } while (!inBdry(flag_h,numElem) || minDist<threshold_inner);
        pCHIEF[cnt] = chief;
        cnt++;
    }
    CURAND_CALL(hiprandDestroyGenerator(gen));
    free(flag_h);
    free(dist_h);
    CUDA_CALL(hipFree(pt_d));
    CUDA_CALL(hipFree(elem_d));
    CUDA_CALL(hipFree(flag_d));
    CUDA_CALL(hipFree(dist_d));
    return EXIT_SUCCESS;
}

inline __device__ void crossNorm(const vec3f a, const vec3f b, vec3f *norm, float *length) 
{
    vec3f c;
    c.coords[0] = a.coords[1]*b.coords[2]-a.coords[2]*b.coords[1];
    c.coords[1] = a.coords[2]*b.coords[0]-a.coords[0]*b.coords[2];
    c.coords[2] = a.coords[0]*b.coords[1]-a.coords[1]*b.coords[0];

    *length = __fsqrt_rn((c.coords[0]*c.coords[0])+(c.coords[1]*c.coords[1])+(c.coords[2]*c.coords[2]));

    norm->coords[0] = c.coords[0] / *length;
    norm->coords[1] = c.coords[1] / *length;
    norm->coords[2] = c.coords[2] / *length;
}

__device__ void g_h_c_nsgl(const float k, const vec3f x, const vec3f p[3], 
        hipFloatComplex gCoeff[3], hipFloatComplex hCoeff[3], float *cCoeff) {
    //Initalization of g, h and c
    //printf("(%f,%f,%f)\n",p[0].coords[0],p[0].coords[1],p[0].coords[2]);
    for(int i=0;i<3;i++) {
        gCoeff[i] = make_hipFloatComplex(0,0);
        hCoeff[i] = make_hipFloatComplex(0,0);
    }
    *cCoeff = 0;
    
    //Local variables
    float eta1, eta2, wn, wm, xi1, xi2, xi3, rho, theta, vertCrossProd, temp, 
            temp_gh[3], omega = k*speed, pPsiLpn2, radius, prpn2;
    vec3f y, normal, rVec;
    hipFloatComplex Psi, pPsipn2;
    crossNorm(
    {
        p[0].coords[0]-p[2].coords[0],p[0].coords[1]-p[2].coords[1],p[0].coords[2]-p[2].coords[2]
    },
    {
        p[1].coords[0]-p[2].coords[0],p[1].coords[1]-p[2].coords[1],p[1].coords[2]-p[2].coords[2]
    },&normal,&vertCrossProd);
    vertCrossProd = vertCrossProd*0.25f;
    //printf("%f\n",normal.coords[0]);
    const float prodRhoOmega = density*omega;
    const float fourPI = 4.0f*PI;
    const float recipFourPI = 1.0f/fourPI;
    //printf("%f\n",k);
    for(int n=0;n<INTORDER;n++) {
        eta2 = INTPT[n];
        wn = INTWGT[n];
        theta = 0.5f+0.5f*eta2;
        for(int m=0;m<INTORDER;m++) {
            eta1 = INTPT[m];
            wm = INTWGT[m];
            rho = 0.5f+0.5f*eta1;
            temp = wn*wm*rho*vertCrossProd;
            
            xi1 = rho*(1-theta);
            xi2 = rho-xi1;
            xi3 = 1-xi1-xi2;
            //printf("xi1 = %f, xi2 = %f\n",xi1,xi2);
            y= {
                p[0].coords[0]*xi1+p[1].coords[0]*xi2+p[2].coords[0]*xi3, 
                p[0].coords[1]*xi1+p[1].coords[1]*xi2+p[2].coords[1]*xi3, 
                p[0].coords[2]*xi1+p[1].coords[2]*xi2+p[2].coords[2]*xi3
            };
            //printf("x: (%f,%f,%f), y: (%f,%f,%f)\n",x.coords[0],x.coords[1],x.coords[2],
            //        y.coords[0],y.coords[1],y.coords[2]);
            rVec = vecSub(y,x);
            radius = __fsqrt_rn(rVec.coords[0]*rVec.coords[0]+rVec.coords[1]*rVec.coords[1]
                    +rVec.coords[2]*rVec.coords[2]);
            //printf("radius = %f\n",radius);
            prpn2 = ((y.coords[0]-x.coords[0])*normal.coords[0]+(y.coords[1]-x.coords[1])*normal.coords[1]
                    +(y.coords[2]-x.coords[2])*normal.coords[2])/radius;
            //printf("prpn2=%f\n",prpn2);
            pPsiLpn2 = -recipFourPI/(radius*radius)*prpn2;
            //printf("%f\n",pPsiLpn2);
            Psi = make_hipFloatComplex(__cosf(-k*radius)/(fourPI*radius),__sinf(-k*radius)/(fourPI*radius));
            pPsipn2 = hipCmulf(Psi,make_hipFloatComplex(-1.0f/radius,-k));
            pPsipn2 = make_hipFloatComplex(prpn2*hipCrealf(pPsipn2),prpn2*hipCimagf(pPsipn2));
            temp_gh[0] = temp*xi1;
            temp_gh[1] = temp*xi2;
            temp_gh[2] = temp*xi3;
            
            gCoeff[0] = hipCaddf(gCoeff[0],make_hipFloatComplex(temp_gh[0]*hipCrealf(Psi),temp_gh[0]*hipCimagf(Psi)));
            gCoeff[1] = hipCaddf(gCoeff[1],make_hipFloatComplex(temp_gh[1]*hipCrealf(Psi),temp_gh[1]*hipCimagf(Psi)));
            gCoeff[2] = hipCaddf(gCoeff[2],make_hipFloatComplex(temp_gh[2]*hipCrealf(Psi),temp_gh[2]*hipCimagf(Psi)));
            
            hCoeff[0] = hipCaddf(hCoeff[0],make_hipFloatComplex(temp_gh[0]*hipCrealf(pPsipn2),temp_gh[0]*hipCimagf(pPsipn2)));
            hCoeff[1] = hipCaddf(hCoeff[1],make_hipFloatComplex(temp_gh[1]*hipCrealf(pPsipn2),temp_gh[1]*hipCimagf(pPsipn2)));
            hCoeff[2] = hipCaddf(hCoeff[2],make_hipFloatComplex(temp_gh[2]*hipCrealf(pPsipn2),temp_gh[2]*hipCimagf(pPsipn2)));
            
            *cCoeff += temp*pPsiLpn2;
        }
    }
    gCoeff[0] = make_hipFloatComplex(-prodRhoOmega*hipCimagf(gCoeff[0]),prodRhoOmega*hipCrealf(gCoeff[0]));
    gCoeff[1] = make_hipFloatComplex(-prodRhoOmega*hipCimagf(gCoeff[1]),prodRhoOmega*hipCrealf(gCoeff[1]));
    gCoeff[2] = make_hipFloatComplex(-prodRhoOmega*hipCimagf(gCoeff[2]),prodRhoOmega*hipCrealf(gCoeff[2]));
}

__device__ void g_h_c_sgl(const float k, const vec3f x_sgl1, const vec3f x_sgl2, 
        const vec3f x_sgl3, const vec3f p[3], 
        hipFloatComplex gCoeff_sgl1[3], hipFloatComplex hCoeff_sgl1[3], float *cCoeff_sgl1,
        hipFloatComplex gCoeff_sgl2[3], hipFloatComplex hCoeff_sgl2[3], float *cCoeff_sgl2,
        hipFloatComplex gCoeff_sgl3[3], hipFloatComplex hCoeff_sgl3[3], float *cCoeff_sgl3) 
{
    //Initalization of g, h and c
    for(int i=0;i<3;i++) {
        gCoeff_sgl1[i] = make_hipFloatComplex(0,0);
        hCoeff_sgl1[i] = make_hipFloatComplex(0,0);
        gCoeff_sgl2[i] = make_hipFloatComplex(0,0);
        hCoeff_sgl2[i] = make_hipFloatComplex(0,0);
        gCoeff_sgl3[i] = make_hipFloatComplex(0,0);
        hCoeff_sgl3[i] = make_hipFloatComplex(0,0);
    }
    *cCoeff_sgl1 = 0;
    *cCoeff_sgl2 = 0;
    *cCoeff_sgl3 = 0;
    
    //Local variables
    float eta1, eta2, wn, wm, xi1_sgl1, xi2_sgl1, xi3_sgl1, xi1_sgl2, xi2_sgl2, xi3_sgl2,
            xi1_sgl3, xi2_sgl3, xi3_sgl3, rho, theta, vertCrossProd, temp, 
            temp_gh[3], omega = k*speed, pPsiLpn2, radius, prpn2;
    vec3f y_sgl1, y_sgl2, y_sgl3, normal, rVec;
    hipFloatComplex Psi, pPsipn2;
    crossNorm(
    {
        p[0].coords[0]-p[2].coords[0],p[0].coords[1]-p[2].coords[1],p[0].coords[2]-p[2].coords[2]
    },
    {
        p[1].coords[0]-p[2].coords[0],p[1].coords[1]-p[2].coords[1],p[1].coords[2]-p[2].coords[2]
    },&normal,&vertCrossProd);
    vertCrossProd = vertCrossProd*0.25f;
    //printf("vert: %f\n",vertCrossProd);
    
    //printf("normal=(%f,%f,%f)\n",normal.coords[0],normal.coords[1],normal.coords[2]);
    const float prodRhoOmega = density*omega;
    const float fourPI = 4.0f*PI;
    const float recipFourPI = 1.0/fourPI;
    //printf("density*omega = %f\n",prodRhoOmega);
    //Compute integrals for g, h and c
    for(int n=0;n<INTORDER;n++) {
        eta2 = INTPT[n];
        wn = INTWGT[n];
        theta = 0.5f+0.5f*eta2;
        for(int m=0;m<INTORDER;m++) {
            eta1 = INTPT[m];
            wm = INTWGT[m];
            rho = 0.5f+0.5f*eta1;
            temp = wn*wm*rho*vertCrossProd;
            
            xi1_sgl3 = rho*(1-theta);
            xi2_sgl3 = rho-xi1_sgl3; //rho*theta
            xi3_sgl3 = 1-xi1_sgl3-xi2_sgl3;
            
            xi1_sgl1 = 1-rho;
            xi2_sgl1 = rho-xi2_sgl3; //rho-rho*theta
            xi3_sgl1 = 1-xi1_sgl1-xi2_sgl1;
            
            xi1_sgl2 = xi2_sgl3; //rho*theta
            xi2_sgl2 = 1-rho;
            xi3_sgl2 = 1-xi1_sgl2-xi2_sgl2;
            
            
            
            //printf("xi1 = %f, xi2 = %f\n",xi1,xi2);
            y_sgl1= {
                p[0].coords[0]*xi1_sgl1+p[1].coords[0]*xi2_sgl1+p[2].coords[0]*xi3_sgl1, 
                p[0].coords[1]*xi1_sgl1+p[1].coords[1]*xi2_sgl1+p[2].coords[1]*xi3_sgl1, 
                p[0].coords[2]*xi1_sgl1+p[1].coords[2]*xi2_sgl1+p[2].coords[2]*xi3_sgl1
            };
            y_sgl2= {
                p[0].coords[0]*xi1_sgl2+p[1].coords[0]*xi2_sgl2+p[2].coords[0]*xi3_sgl2, 
                p[0].coords[1]*xi1_sgl2+p[1].coords[1]*xi2_sgl2+p[2].coords[1]*xi3_sgl2, 
                p[0].coords[2]*xi1_sgl2+p[1].coords[2]*xi2_sgl2+p[2].coords[2]*xi3_sgl2
            };
            y_sgl3= {
                p[0].coords[0]*xi1_sgl3+p[1].coords[0]*xi2_sgl3+p[2].coords[0]*xi3_sgl3, 
                p[0].coords[1]*xi1_sgl3+p[1].coords[1]*xi2_sgl3+p[2].coords[1]*xi3_sgl3, 
                p[0].coords[2]*xi1_sgl3+p[1].coords[2]*xi2_sgl3+p[2].coords[2]*xi3_sgl3
            };
            
            //update coefficients with singularity on node 1
            rVec = vecSub(y_sgl1,x_sgl1);
            radius = sqrtf(rVec.coords[0]*rVec.coords[0]+rVec.coords[1]*rVec.coords[1]+rVec.coords[2]*rVec.coords[2]);
            //printf("radius = %f\n",radius);
            prpn2 = ((y_sgl1.coords[0]-x_sgl1.coords[0])*normal.coords[0]+(y_sgl1.coords[1]-x_sgl1.coords[1])*normal.coords[1]
                    +(y_sgl1.coords[2]-x_sgl1.coords[2])*normal.coords[2])/radius;
            //printf("prpn2=%f\n",prpn2);
            pPsiLpn2 = -recipFourPI/(radius*radius)*prpn2;
            //printf("%f\n",pPsiLpn2);
            Psi = make_hipFloatComplex(__cosf(-k*radius)/(fourPI*radius),__sinf(-k*radius)/(fourPI*radius));
            pPsipn2 = hipCmulf(Psi,make_hipFloatComplex(-1.0f/radius,-k));
            pPsipn2 = make_hipFloatComplex(prpn2*hipCrealf(pPsipn2),prpn2*hipCimagf(pPsipn2));
            temp_gh[0] = temp*xi1_sgl1;
            temp_gh[1] = temp*xi2_sgl1;
            temp_gh[2] = temp*xi3_sgl1;
            
            gCoeff_sgl1[0] = hipCaddf(gCoeff_sgl1[0],make_hipFloatComplex(temp_gh[0]*hipCrealf(Psi),temp_gh[0]*hipCimagf(Psi)));
            gCoeff_sgl1[1] = hipCaddf(gCoeff_sgl1[1],make_hipFloatComplex(temp_gh[1]*hipCrealf(Psi),temp_gh[1]*hipCimagf(Psi)));
            gCoeff_sgl1[2] = hipCaddf(gCoeff_sgl1[2],make_hipFloatComplex(temp_gh[2]*hipCrealf(Psi),temp_gh[2]*hipCimagf(Psi)));
            
            hCoeff_sgl1[0] = hipCaddf(hCoeff_sgl1[0],make_hipFloatComplex(temp_gh[0]*hipCrealf(pPsipn2),temp_gh[0]*hipCimagf(pPsipn2)));
            hCoeff_sgl1[1] = hipCaddf(hCoeff_sgl1[1],make_hipFloatComplex(temp_gh[1]*hipCrealf(pPsipn2),temp_gh[1]*hipCimagf(pPsipn2)));
            hCoeff_sgl1[2] = hipCaddf(hCoeff_sgl1[2],make_hipFloatComplex(temp_gh[2]*hipCrealf(pPsipn2),temp_gh[2]*hipCimagf(pPsipn2)));
            
            *cCoeff_sgl1 += temp*pPsiLpn2;
            
            //update coefficients with singularity on node 2
            rVec = vecSub(y_sgl2,x_sgl2);
            radius = sqrtf(rVec.coords[0]*rVec.coords[0]+rVec.coords[1]*rVec.coords[1]+rVec.coords[2]*rVec.coords[2]);
            //printf("radius = %f\n",radius);
            prpn2 = ((y_sgl1.coords[0]-x_sgl1.coords[0])*normal.coords[0]+(y_sgl1.coords[1]-x_sgl1.coords[1])*normal.coords[1]
                    +(y_sgl1.coords[2]-x_sgl1.coords[2])*normal.coords[2])/radius;
            //printf("prpn2=%f\n",prpn2);
            pPsiLpn2 = -recipFourPI/(radius*radius)*prpn2;
            //printf("%f\n",pPsiLpn2);
            Psi = make_hipFloatComplex(__cosf(-k*radius)/(fourPI*radius),__sinf(-k*radius)/(fourPI*radius));
            pPsipn2 = hipCmulf(Psi,make_hipFloatComplex(-1.0f/radius,-k));
            pPsipn2 = make_hipFloatComplex(prpn2*hipCrealf(pPsipn2),prpn2*hipCimagf(pPsipn2));
            temp_gh[0] = temp*xi1_sgl2;
            temp_gh[1] = temp*xi2_sgl2;
            temp_gh[2] = temp*xi3_sgl2;
            
            gCoeff_sgl2[0] = hipCaddf(gCoeff_sgl2[0],make_hipFloatComplex(temp_gh[0]*hipCrealf(Psi),temp_gh[0]*hipCimagf(Psi)));
            gCoeff_sgl2[1] = hipCaddf(gCoeff_sgl2[1],make_hipFloatComplex(temp_gh[1]*hipCrealf(Psi),temp_gh[1]*hipCimagf(Psi)));
            gCoeff_sgl2[2] = hipCaddf(gCoeff_sgl2[2],make_hipFloatComplex(temp_gh[2]*hipCrealf(Psi),temp_gh[2]*hipCimagf(Psi)));
            
            hCoeff_sgl2[0] = hipCaddf(hCoeff_sgl2[0],make_hipFloatComplex(temp_gh[0]*hipCrealf(pPsipn2),temp_gh[0]*hipCimagf(pPsipn2)));
            hCoeff_sgl2[1] = hipCaddf(hCoeff_sgl2[1],make_hipFloatComplex(temp_gh[1]*hipCrealf(pPsipn2),temp_gh[1]*hipCimagf(pPsipn2)));
            hCoeff_sgl2[2] = hipCaddf(hCoeff_sgl2[2],make_hipFloatComplex(temp_gh[2]*hipCrealf(pPsipn2),temp_gh[2]*hipCimagf(pPsipn2)));
            
            *cCoeff_sgl2 += temp*pPsiLpn2;
            
            //update coefficients with singularity on node 3
            rVec = vecSub(y_sgl3,x_sgl3);
            radius = sqrtf(rVec.coords[0]*rVec.coords[0]+rVec.coords[1]*rVec.coords[1]+rVec.coords[2]*rVec.coords[2]);
            //printf("radius = %f\n",radius);
            prpn2 = ((y_sgl1.coords[0]-x_sgl1.coords[0])*normal.coords[0]+(y_sgl1.coords[1]-x_sgl1.coords[1])*normal.coords[1]
                    +(y_sgl1.coords[2]-x_sgl1.coords[2])*normal.coords[2])/radius;
            //printf("prpn2=%f\n",prpn2);
            pPsiLpn2 = -recipFourPI/(radius*radius)*prpn2;
            //printf("%f\n",pPsiLpn2);
            Psi = make_hipFloatComplex(__cosf(-k*radius)/(fourPI*radius),__sinf(-k*radius)/(fourPI*radius));
            pPsipn2 = hipCmulf(Psi,make_hipFloatComplex(-1.0f/radius,-k));
            pPsipn2 = make_hipFloatComplex(prpn2*hipCrealf(pPsipn2),prpn2*hipCimagf(pPsipn2));
            temp_gh[0] = temp*xi1_sgl3;
            temp_gh[1] = temp*xi2_sgl3;
            temp_gh[2] = temp*xi3_sgl3;
            
            gCoeff_sgl3[0] = hipCaddf(gCoeff_sgl3[0],make_hipFloatComplex(temp_gh[0]*hipCrealf(Psi),temp_gh[0]*hipCimagf(Psi)));
            gCoeff_sgl3[1] = hipCaddf(gCoeff_sgl3[1],make_hipFloatComplex(temp_gh[1]*hipCrealf(Psi),temp_gh[1]*hipCimagf(Psi)));
            gCoeff_sgl3[2] = hipCaddf(gCoeff_sgl3[2],make_hipFloatComplex(temp_gh[2]*hipCrealf(Psi),temp_gh[2]*hipCimagf(Psi)));
            
            hCoeff_sgl3[0] = hipCaddf(hCoeff_sgl3[0],make_hipFloatComplex(temp_gh[0]*hipCrealf(pPsipn2),temp_gh[0]*hipCimagf(pPsipn2)));
            hCoeff_sgl3[1] = hipCaddf(hCoeff_sgl3[1],make_hipFloatComplex(temp_gh[1]*hipCrealf(pPsipn2),temp_gh[1]*hipCimagf(pPsipn2)));
            hCoeff_sgl3[2] = hipCaddf(hCoeff_sgl3[2],make_hipFloatComplex(temp_gh[2]*hipCrealf(pPsipn2),temp_gh[2]*hipCimagf(pPsipn2)));
            
            *cCoeff_sgl3 += temp*pPsiLpn2;
        }
    }
    gCoeff_sgl1[0] = make_hipFloatComplex(-prodRhoOmega*hipCimagf(gCoeff_sgl1[0]),prodRhoOmega*hipCrealf(gCoeff_sgl1[0]));
    gCoeff_sgl1[1] = make_hipFloatComplex(-prodRhoOmega*hipCimagf(gCoeff_sgl1[1]),prodRhoOmega*hipCrealf(gCoeff_sgl1[1]));
    gCoeff_sgl1[2] = make_hipFloatComplex(-prodRhoOmega*hipCimagf(gCoeff_sgl1[2]),prodRhoOmega*hipCrealf(gCoeff_sgl1[2]));
    
    gCoeff_sgl2[0] = make_hipFloatComplex(-prodRhoOmega*hipCimagf(gCoeff_sgl2[0]),prodRhoOmega*hipCrealf(gCoeff_sgl2[0]));
    gCoeff_sgl2[1] = make_hipFloatComplex(-prodRhoOmega*hipCimagf(gCoeff_sgl2[1]),prodRhoOmega*hipCrealf(gCoeff_sgl2[1]));
    gCoeff_sgl2[2] = make_hipFloatComplex(-prodRhoOmega*hipCimagf(gCoeff_sgl2[2]),prodRhoOmega*hipCrealf(gCoeff_sgl2[2]));
    
    gCoeff_sgl3[0] = make_hipFloatComplex(-prodRhoOmega*hipCimagf(gCoeff_sgl3[0]),prodRhoOmega*hipCrealf(gCoeff_sgl3[0]));
    gCoeff_sgl3[1] = make_hipFloatComplex(-prodRhoOmega*hipCimagf(gCoeff_sgl3[1]),prodRhoOmega*hipCrealf(gCoeff_sgl3[1]));
    gCoeff_sgl3[2] = make_hipFloatComplex(-prodRhoOmega*hipCimagf(gCoeff_sgl3[2]),prodRhoOmega*hipCrealf(gCoeff_sgl3[2]));
}

__host__ __device__ hipFloatComplex ptSrc(const float k, const float amp, const vec3f srcLoc, const vec3f evalLoc)
{
    float fourPI = 4.0f*PI;
    vec3f rVec = vecSub(evalLoc,srcLoc);
    float radius = sqrtf(rVec.coords[0]*rVec.coords[0]+rVec.coords[1]*rVec.coords[1]+rVec.coords[2]*rVec.coords[2]);
    return make_hipFloatComplex(amp*cosf(-k*radius)/(fourPI*radius),amp*sinf(-k*radius)/(fourPI*radius));
}

__host__ __device__ hipFloatComplex mpSrc(const float k, const float qs, const vec3f src, const vec3f eval)
{
    vec3f vec = vecSub(eval,src);
    float radius = sqrtf(vec.coords[0]*vec.coords[0]+vec.coords[1]*vec.coords[1]+vec.coords[2]*vec.coords[2]);
    hipFloatComplex result = make_hipFloatComplex(0,RHO_AIR*SPEED_SOUND*k*qs/(4*PI));
    result = hipCmulf(result,make_hipFloatComplex(cos(-k*radius)/radius,sin(-k*radius)/radius));
    return result;
}

__host__ __device__ hipFloatComplex dirSrc(const float k, const float strength, const vec3f dir, const vec3f evalLoc)
{
    float theta = -k*vecDotMul(dir,evalLoc);
    return make_hipFloatComplex(strength*cosf(theta),strength*sinf(theta));
}

// compute non-singular relationship between points and elements
__global__ void atomicPtsElems_nsgl(const float k, const vec3f *pt, const int numNod, 
        const int idxPntStart, const int idxPntEnd, const tri_elem *elem, const int numElem, 
        hipFloatComplex *A, const int lda, hipFloatComplex *B, const int numSrc, const int ldb) {
    int xIdx = blockIdx.x*blockDim.x+threadIdx.x; //Index for points
    int yIdx = blockIdx.y*blockDim.y+threadIdx.y; //Index for elements
    //The thread with indices xIdx and yIdx process the point xIdx and elem yIdx
    if(xIdx>=idxPntStart && xIdx<=idxPntEnd && yIdx<numElem && xIdx!=elem[yIdx].nod[0] 
            && xIdx!=elem[yIdx].nod[1] && xIdx!=elem[yIdx].nod[2]) {
        int i, j;
        hipFloatComplex hCoeff[3], gCoeff[3], bc, pCoeffs[3], temp;
        float cCoeff;
        vec3f triNod[3];
        triNod[0] = pt[elem[yIdx].nod[0]];
        triNod[1] = pt[elem[yIdx].nod[1]];
        triNod[2] = pt[elem[yIdx].nod[2]];
        g_h_c_nsgl(k,pt[xIdx],triNod,gCoeff,hCoeff,&cCoeff);
        
        //Update the A matrix
        bc = hipCdivf(elem[yIdx].bc[0],elem[yIdx].bc[1]);
        for(i=0;i<3;i++) {
            pCoeffs[i] = hipCsubf(hCoeff[i],hipCmulf(bc,gCoeff[i]));
        }
        
        for(i=0;i<3;i++) {
            //atomicFloatComplexAdd(&A[IDXC0(xIdx,elem[yIdx].nod[i],lda)],pCoeffs[i]);
            atomicAdd(&A[IDXC0(xIdx,elem[yIdx].nod[i],lda)].x,hipCrealf(pCoeffs[i]));
            atomicAdd(&A[IDXC0(xIdx,elem[yIdx].nod[i],lda)].y,hipCimagf(pCoeffs[i]));
        }
        
        //Update from C coefficients
        if(xIdx<numNod) {
            //atomicFloatComplexSub(&A[IDXC0(xIdx,xIdx,lda)],make_hipFloatComplex(cCoeff,0));
            atomicAdd(&A[IDXC0(xIdx,xIdx,lda)].x,-cCoeff);
        }
        
        //Update the B matrix
        bc = hipCdivf(elem[yIdx].bc[2],elem[yIdx].bc[1]);
        //printf("bc: \n");
        //printComplexMatrix(&bc,1,1,1);
        for(i=0;i<numSrc;i++) {
            for(j=0;j<3;j++) {
                //atomicFloatComplexSub(&B[IDXC0(xIdx,i,ldb)],hipCmulf(bc,gCoeff[j]));
                temp = hipCmulf(bc,gCoeff[j]);
                atomicAdd(&B[IDXC0(xIdx,i,ldb)].x,-hipCrealf(temp));
                atomicAdd(&B[IDXC0(xIdx,i,ldb)].y,-hipCimagf(temp));
            }
        }
    }
}

__global__ void atomicPtsElems_sgl(const float k, const vec3f *pt, const tri_elem *elem, 
        const int numElem, hipFloatComplex *A, const int lda, hipFloatComplex *B, 
        const int numSrc, const int ldb) {
    int idx = blockIdx.x*blockDim.x+threadIdx.x;
    if(idx < numElem) {
        int i, j;
        hipFloatComplex hCoeff_sgl1[3], hCoeff_sgl2[3], hCoeff_sgl3[3], 
                gCoeff_sgl1[3], gCoeff_sgl2[3], gCoeff_sgl3[3], pCoeffs_sgl1[3], 
                pCoeffs_sgl2[3], pCoeffs_sgl3[3], bc, temp;
        float cCoeff_sgl1, cCoeff_sgl2, cCoeff_sgl3;
        
        vec3f nod[3];
        for(i=0;i<3;i++) {
            nod[i] = pt[elem[idx].nod[i]];
        }
        // Compute h and g coefficients
        g_h_c_sgl(k,pt[elem[idx].nod[0]],pt[elem[idx].nod[1]],pt[elem[idx].nod[2]],
                nod,gCoeff_sgl1,hCoeff_sgl1,&cCoeff_sgl1,gCoeff_sgl2,hCoeff_sgl2,&cCoeff_sgl2,
                gCoeff_sgl3,hCoeff_sgl3,&cCoeff_sgl3);
        
        //Compute p coefficients
        bc = hipCdivf(elem[idx].bc[0],elem[idx].bc[1]);
        for(j=0;j<3;j++) {
            pCoeffs_sgl1[j] = hipCsubf(hCoeff_sgl1[j],hipCmulf(bc,gCoeff_sgl1[j]));
            pCoeffs_sgl2[j] = hipCsubf(hCoeff_sgl2[j],hipCmulf(bc,gCoeff_sgl2[j]));
            pCoeffs_sgl3[j] = hipCsubf(hCoeff_sgl3[j],hipCmulf(bc,gCoeff_sgl3[j]));
        }
        
        //Update matrix A using pCoeffs
        for(j=0;j<3;j++) {
            //atomicFloatComplexAdd(&A[IDXC0(elem[idx].nod[0],elem[idx].nod[j],lda)],
            //        pCoeffs_sgl1[j]);
            atomicAdd(&A[IDXC0(elem[idx].nod[0],elem[idx].nod[j],lda)].x,
                    hipCrealf(pCoeffs_sgl1[j]));
            atomicAdd(&A[IDXC0(elem[idx].nod[0],elem[idx].nod[j],lda)].y,
                    hipCimagf(pCoeffs_sgl1[j]));
            //atomicFloatComplexAdd(&A[IDXC0(elem[idx].nod[1],elem[idx].nod[j],lda)],
            //        pCoeffs_sgl2[j]);
            atomicAdd(&A[IDXC0(elem[idx].nod[1],elem[idx].nod[j],lda)].x,
                    hipCrealf(pCoeffs_sgl2[j]));
            atomicAdd(&A[IDXC0(elem[idx].nod[1],elem[idx].nod[j],lda)].y,
                    hipCimagf(pCoeffs_sgl2[j]));
            //atomicFloatComplexAdd(&A[IDXC0(elem[idx].nod[2],elem[idx].nod[j],lda)],
            //        pCoeffs_sgl3[j]);
            atomicAdd(&A[IDXC0(elem[idx].nod[2],elem[idx].nod[j],lda)].x,
                    hipCrealf(pCoeffs_sgl3[j]));
            atomicAdd(&A[IDXC0(elem[idx].nod[2],elem[idx].nod[j],lda)].y,
                    hipCimagf(pCoeffs_sgl3[j]));
        }
        
        //atomicFloatComplexSub(&A[IDXC0(elem[idx].nod[0],elem[idx].nod[0],lda)],
        //        make_hipFloatComplex(cCoeff_sgl1,0));
        atomicAdd(&A[IDXC0(elem[idx].nod[0],elem[idx].nod[0],lda)].x,
                -cCoeff_sgl1);
        //atomicFloatComplexSub(&A[IDXC0(elem[idx].nod[1],elem[idx].nod[1],lda)],
        //        make_hipFloatComplex(cCoeff_sgl2,0));
        atomicAdd(&A[IDXC0(elem[idx].nod[1],elem[idx].nod[1],lda)].x,
                -cCoeff_sgl2);
        //atomicFloatComplexSub(&A[IDXC0(elem[idx].nod[2],elem[idx].nod[2],lda)],
        //        make_hipFloatComplex(cCoeff_sgl3,0));
        atomicAdd(&A[IDXC0(elem[idx].nod[2],elem[idx].nod[2],lda)].x,
                -cCoeff_sgl3);
        
        //Update matrix B using g Coefficients
        bc = hipCdivf(elem[idx].bc[2],elem[idx].bc[1]);
        for(i=0;i<numSrc;i++) {
            for(j=0;j<3;j++) {
                //atomicFloatComplexSub(&B[IDXC0(elem[idx].nod[0],i,ldb)],
                //        hipCmulf(bc,gCoeff_sgl1[j]));
                temp = hipCmulf(bc,gCoeff_sgl1[j]);
                atomicAdd(&B[IDXC0(elem[idx].nod[0],i,ldb)].x,-hipCrealf(temp));
                atomicAdd(&B[IDXC0(elem[idx].nod[0],i,ldb)].y,-hipCimagf(temp));
                //atomicFloatComplexSub(&B[IDXC0(elem[idx].nod[1],i,ldb)],
                //        hipCmulf(bc,gCoeff_sgl2[j]));
                temp = hipCmulf(bc,gCoeff_sgl2[j]);
                atomicAdd(&B[IDXC0(elem[idx].nod[1],i,ldb)].x,-hipCrealf(temp));
                atomicAdd(&B[IDXC0(elem[idx].nod[1],i,ldb)].y,-hipCimagf(temp));
                //atomicFloatComplexSub(&B[IDXC0(elem[idx].nod[2],i,ldb)],
                //        hipCmulf(bc,gCoeff_sgl3[j]));
                temp = hipCmulf(bc,gCoeff_sgl3[j]);
                atomicAdd(&B[IDXC0(elem[idx].nod[2],i,ldb)].x,-hipCrealf(temp));
                atomicAdd(&B[IDXC0(elem[idx].nod[2],i,ldb)].y,-hipCimagf(temp));
            }
        }
    }
}

int atomicGenSystem(const float k, const tri_elem *elem, const int numElem, 
        const vec3f *nod, const int numNod, const vec3f *chief, const int numCHIEF, 
        const vec3f *src, const int numSrc, hipFloatComplex *A, const int lda, 
        hipFloatComplex *B, const int ldb) {
    int i, j;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    //Move elements to GPU
    tri_elem *elem_d;
    CUDA_CALL(hipMalloc(&elem_d,numElem*sizeof(tri_elem)));
    CUDA_CALL(hipMemcpy(elem_d,elem,numElem*sizeof(tri_elem),hipMemcpyHostToDevice));
    
    //Move points to GPU
    vec3f *pt_h = (vec3f*)malloc((numNod+numCHIEF)*sizeof(vec3f));
    for(i=0;i<numNod;i++) {
        pt_h[i] = nod[i];
    }
    for(i=0;i<numCHIEF;i++) {
        pt_h[numNod+i] = chief[i];
    }
    
    vec3f *pt_d;
    CUDA_CALL(hipMalloc(&pt_d,(numNod+numCHIEF)*sizeof(vec3f)));
    CUDA_CALL(hipMemcpy(pt_d,pt_h,(numNod+numCHIEF)*sizeof(vec3f),hipMemcpyHostToDevice));
    
    //Initialization of A
    for(i=0;i<numNod+numCHIEF;i++) {
        for(j=0;j<numNod;j++) {
            if(i==j) {
                A[IDXC0(i,j,lda)] = make_hipFloatComplex(1,0);
            } else {
                A[IDXC0(i,j,lda)] = make_hipFloatComplex(0,0);
            }
        }
    }
    
    //Initialization of B
    for(i=0;i<numNod+numCHIEF;i++) {
        for(j=0;j<numSrc;j++) {
            B[IDXC0(i,j,ldb)] = ptSrc(k,STRENGTH,src[j],pt_h[i]);
        }
    }
    
    hipFloatComplex *A_d, *B_d;
    CUDA_CALL(hipMalloc(&A_d,(numNod+numCHIEF)*numNod*sizeof(hipFloatComplex)));
    CUDA_CALL(hipMemcpy(A_d,A,(numNod+numCHIEF)*numNod*sizeof(hipFloatComplex),hipMemcpyHostToDevice));
    
    CUDA_CALL(hipMalloc(&B_d,(numNod+numCHIEF)*numSrc*sizeof(hipFloatComplex)));
    CUDA_CALL(hipMemcpy(B_d,B,(numNod+numCHIEF)*numSrc*sizeof(hipFloatComplex),hipMemcpyHostToDevice));
    
    int xNumBlocks, xWidth = 16, yNumBlocks, yWidth = 16;
    xNumBlocks = (numNod+numCHIEF+xWidth-1)/xWidth;
    yNumBlocks = (numElem+yWidth-1)/yWidth;
    dim3 gridLayout, blockLayout;
    gridLayout.x = xNumBlocks;
    gridLayout.y = yNumBlocks;
    
    blockLayout.x = xWidth;
    blockLayout.y = yWidth;
    
    hipEventRecord(start);
    atomicPtsElems_nsgl<<<gridLayout,blockLayout>>>(k,pt_d,numNod,0,numNod+numCHIEF-1,
            elem_d,numElem,A_d,lda,B_d,numSrc,ldb);
    
    //CUDA_CALL(hipMemcpy(A,A_d,(numNod+numCHIEF)*numNod*sizeof(hipFloatComplex),hipMemcpyDeviceToHost));
    //printCuFloatComplexMat(A,numNod+numCHIEF,numNod,numNod+numCHIEF);
    atomicPtsElems_sgl<<<yNumBlocks,yWidth>>>(k,pt_d,elem_d,numElem,A_d,lda,B_d,numSrc,ldb);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds,start,stop);
    printf("Elapsed system generation time: %f milliseconds.\n",milliseconds);
    CUDA_CALL(hipMemcpy(A,A_d,(numNod+numCHIEF)*numNod*sizeof(hipFloatComplex),hipMemcpyDeviceToHost));
    CUDA_CALL(hipMemcpy(B,B_d,(numNod+numCHIEF)*numSrc*sizeof(hipFloatComplex),hipMemcpyDeviceToHost));
    
    
    CUDA_CALL(hipFree(A_d));
    CUDA_CALL(hipFree(B_d));
    CUDA_CALL(hipFree(elem_d));
    CUDA_CALL(hipFree(pt_d));
    
    return EXIT_SUCCESS;
}

int qrSolver(const hipFloatComplex *A, const int mA, const int nA, const int ldA, 
        hipFloatComplex *B, const int nB, const int ldB) {
    hipEvent_t start, stop;
    CUDA_CALL(hipEventCreate(&start));
    CUDA_CALL(hipEventCreate(&stop));
    hipsolverHandle_t cusolverH = NULL;
    CUSOLVER_CALL(hipsolverDnCreate(&cusolverH));
    
    
    hipFloatComplex *A_d;
    CUDA_CALL(hipMalloc(&A_d,ldA*nA*sizeof(hipFloatComplex)));
    CUDA_CALL(hipMemcpy(A_d,A,ldA*nA*sizeof(hipFloatComplex),hipMemcpyHostToDevice));
    
    hipFloatComplex *B_d;
    CUDA_CALL(hipMalloc(&B_d,ldB*nB*sizeof(hipFloatComplex)));
    CUDA_CALL(hipMemcpy(B_d,B,ldB*nB*sizeof(hipFloatComplex),hipMemcpyHostToDevice));
    
    //A = QR
    int lwork;
    CUSOLVER_CALL(hipsolverDnCgeqrf_bufferSize(cusolverH,mA,nA,A_d,ldA,&lwork));
    
    hipFloatComplex *workspace_d;
    CUDA_CALL(hipMalloc(&workspace_d,lwork*sizeof(hipFloatComplex)));
    hipFloatComplex *tau_d;
    CUDA_CALL(hipMalloc(&tau_d,max(mA,nA)*sizeof(hipFloatComplex)));
    int *deviceInfo_d, deviceInfo;
    CUDA_CALL(hipMalloc(&deviceInfo_d,sizeof(int)));
    
    CUDA_CALL(hipEventRecord(start));
    CUSOLVER_CALL(hipsolverDnCgeqrf(cusolverH,mA,nA,A_d,ldA,tau_d,workspace_d,lwork,
            deviceInfo_d));
    CUDA_CALL(hipMemcpy(&deviceInfo,deviceInfo_d,sizeof(int),hipMemcpyDeviceToHost));
    
    //B = (Q^H)*B
    CUSOLVER_CALL(hipsolverDnCunmqr(cusolverH,HIPBLAS_SIDE_LEFT,HIPBLAS_OP_C,mA,nB,
            nA,A_d,ldA,tau_d,B_d,ldB,workspace_d,lwork,deviceInfo_d));
    CUDA_CALL(hipMemcpy(&deviceInfo,deviceInfo_d,sizeof(int),hipMemcpyDeviceToHost));
    
    //Solve Rx = B
    hipFloatComplex alpha = make_hipFloatComplex(1,0);
    hipblasHandle_t cublasH;
    CUBLAS_CALL(hipblasCreate(&cublasH));
    CUBLAS_CALL(hipblasCtrsm(cublasH,HIPBLAS_SIDE_LEFT,HIPBLAS_FILL_MODE_UPPER,
            HIPBLAS_OP_N,HIPBLAS_DIAG_NON_UNIT,nA,nB,&alpha,A_d,ldA,B_d,ldB));
    CUDA_CALL(hipEventRecord(stop));
    
    CUDA_CALL(hipMemcpy(B,B_d,ldB*nB*sizeof(hipFloatComplex),hipMemcpyDeviceToHost));
    CUDA_CALL(hipEventSynchronize(stop));
    
    float milliseconds = 0;
    CUDA_CALL(hipEventElapsedTime(&milliseconds,start,stop));
    printf("Elapsed system solving time: %f milliseconds.\n",milliseconds);
    CUDA_CALL(hipEventDestroy(start));
    CUDA_CALL(hipEventDestroy(stop));
    CUDA_CALL(hipFree(A_d));
    CUDA_CALL(hipFree(B_d));
    CUDA_CALL(hipFree(tau_d));
    CUDA_CALL(hipFree(workspace_d));
    CUDA_CALL(hipFree(deviceInfo_d));
    CUBLAS_CALL(hipblasDestroy(cublasH));
    CUSOLVER_CALL(hipsolverDnDestroy(cusolverH));
    
    return EXIT_SUCCESS;
}

int bemSolver_pt(const float k, const tri_elem *elem, const int numElem, 
        const vec3f *nod, const int numNod, const vec3f *chief, const int numCHIEF, 
        const vec3f *src, const int numSrc, hipFloatComplex *B, const int ldb)
{
    int i, j;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    //Move elements to GPU
    tri_elem *elem_d;
    CUDA_CALL(hipMalloc(&elem_d,numElem*sizeof(tri_elem)));
    CUDA_CALL(hipMemcpy(elem_d,elem,numElem*sizeof(tri_elem),hipMemcpyHostToDevice));
    
    //Move points to GPU
    // vec3f *pt_h = (vec3f*)malloc((numNod+numCHIEF)*sizeof(vec3f));
    // for(i=0;i<numNod;i++) {
    //     pt_h[i] = nod[i];
    // }
    // for(i=0;i<numCHIEF;i++) {
    //     pt_h[numNod+i] = chief[i];
    // }
    
    vec3f *pt_d;
    CUDA_CALL(hipMalloc(&pt_d, (numNod + numCHIEF) * sizeof(vec3f)));
    CUDA_CALL(hipMemcpy(pt_d, nod, numNod * sizeof(vec3f),hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpy(pt_d + numNod, chief, numCHIEF * sizeof(vec3f),hipMemcpyHostToDevice));
    
    CUDA_CALL(hipEventRecord(start));
    //Generate the system
    hipFloatComplex *A = (hipFloatComplex*)malloc((numNod+numCHIEF)*numNod*sizeof(hipFloatComplex));
    
    memset(A,0,(numNod+numCHIEF)*numNod*sizeof(hipFloatComplex));

    for(i=0;i<numNod;i++) 
    {
        A[IDXC0(i,i,numNod+numCHIEF)] = make_hipFloatComplex(1,0);
    }
    
    //Initialization of B
    for(i=0;i<numNod+numCHIEF;i++) 
    {
        for(j=0;j<numSrc;j++) 
        {
            if(i < numNod)
                B[IDXC0(i,j,ldb)] = ptSrc(k,STRENGTH,src[j],nod[i]);
            else
                B[IDXC0(i,j,ldb)] = ptSrc(k,STRENGTH,src[j],chief[i - numNod]);
        }
    }
    
    hipFloatComplex *A_d, *B_d;
    CUDA_CALL(hipMalloc(&A_d,(numNod+numCHIEF)*numNod*sizeof(hipFloatComplex)));
    CUDA_CALL(hipMemcpy(A_d,A,(numNod+numCHIEF)*numNod*sizeof(hipFloatComplex),hipMemcpyHostToDevice));
    
    CUDA_CALL(hipMalloc(&B_d,(numNod+numCHIEF)*numSrc*sizeof(hipFloatComplex)));
    CUDA_CALL(hipMemcpy(B_d,B,(numNod+numCHIEF)*numSrc*sizeof(hipFloatComplex),hipMemcpyHostToDevice));
    
    int xNumBlocks, xWidth = 16, yNumBlocks, yWidth = 16;
    xNumBlocks = (numNod+numCHIEF+xWidth-1)/xWidth;
    yNumBlocks = (numElem+yWidth-1)/yWidth;
    dim3 gridLayout, blockLayout;
    gridLayout.x = xNumBlocks;
    gridLayout.y = yNumBlocks;
    
    blockLayout.x = xWidth;
    blockLayout.y = yWidth;
    
    atomicPtsElems_nsgl<<<gridLayout,blockLayout>>>(k,pt_d,numNod,0,numNod+numCHIEF-1,
            elem_d,numElem,A_d,numNod+numCHIEF,B_d,numSrc,ldb);
    atomicPtsElems_sgl<<<yNumBlocks,yWidth>>>(k,pt_d,elem_d,numElem,A_d,numNod+numCHIEF,
            B_d,numSrc,ldb);
    
    //Solving the system
    hipsolverHandle_t cusolverH = NULL;
    CUSOLVER_CALL(hipsolverDnCreate(&cusolverH));
    
    //A = QR
    int lwork;
    CUSOLVER_CALL(hipsolverDnCgeqrf_bufferSize(cusolverH,numNod+numCHIEF,numNod,A_d
            ,numNod+numCHIEF,&lwork));
    
    hipFloatComplex *workspace_d;
    CUDA_CALL(hipMalloc(&workspace_d,lwork*sizeof(hipFloatComplex)));
    hipFloatComplex *tau_d;
    CUDA_CALL(hipMalloc(&tau_d,(numNod+numCHIEF)*sizeof(hipFloatComplex)));
    int *deviceInfo_d, deviceInfo;
    CUDA_CALL(hipMalloc(&deviceInfo_d,sizeof(int)));
    
    
    CUSOLVER_CALL(hipsolverDnCgeqrf(cusolverH,numNod+numCHIEF,numNod,A_d,numNod+numCHIEF,
            tau_d,workspace_d,lwork,deviceInfo_d));
    CUDA_CALL(hipMemcpy(&deviceInfo,deviceInfo_d,sizeof(int),hipMemcpyDeviceToHost));
    
    //B = (Q^H)*B
    CUSOLVER_CALL(hipsolverDnCunmqr(cusolverH,HIPBLAS_SIDE_LEFT,HIPBLAS_OP_C,numNod+numCHIEF,numSrc,
            numNod,A_d,numNod+numCHIEF,tau_d,B_d,ldb,workspace_d,lwork,deviceInfo_d));
    CUDA_CALL(hipMemcpy(&deviceInfo,deviceInfo_d,sizeof(int),hipMemcpyDeviceToHost));
    
    //Solve Rx = B
    hipFloatComplex alpha = make_hipFloatComplex(1,0);
    hipblasHandle_t cublasH;
    CUBLAS_CALL(hipblasCreate(&cublasH));
    CUBLAS_CALL(hipblasCtrsm(cublasH,HIPBLAS_SIDE_LEFT,HIPBLAS_FILL_MODE_UPPER,
            HIPBLAS_OP_N,HIPBLAS_DIAG_NON_UNIT,numNod,numSrc,&alpha,A_d,numNod+numCHIEF,B_d,ldb));
    CUDA_CALL(hipEventRecord(stop));
    CUDA_CALL(hipEventSynchronize(stop));
    CUDA_CALL(hipMemcpy(B,B_d,ldb*numSrc*sizeof(hipFloatComplex),hipMemcpyDeviceToHost));
    
    float milliseconds = 0;
    CUDA_CALL(hipEventElapsedTime(&milliseconds,start,stop));
    printf("Elapsed system solving time: %f milliseconds.\n",milliseconds);
    
    //release memory
    CUDA_CALL(hipEventDestroy(start));
    CUDA_CALL(hipEventDestroy(stop));
    CUDA_CALL(hipFree(A_d));
    CUDA_CALL(hipFree(B_d));
    CUDA_CALL(hipFree(tau_d));
    CUDA_CALL(hipFree(workspace_d));
    CUDA_CALL(hipFree(deviceInfo_d));
    CUBLAS_CALL(hipblasDestroy(cublasH));
    CUSOLVER_CALL(hipsolverDnDestroy(cusolverH));
    CUDA_CALL(hipFree(elem_d));
    CUDA_CALL(hipFree(pt_d));
    free(A);
    return EXIT_SUCCESS;
}

int bemSolver_mp(const float k, const tri_elem *elem, const int numElem, 
        const vec3f *nod, const int numNod, const vec3f *chief, const int numCHIEF, 
        const vec3f *src, const int numSrc, hipFloatComplex *B, const int ldb)
{
    int i, j;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    //Move elements to GPU
    tri_elem *elem_d;
    CUDA_CALL(hipMalloc(&elem_d,numElem*sizeof(tri_elem)));
    CUDA_CALL(hipMemcpy(elem_d,elem,numElem*sizeof(tri_elem),hipMemcpyHostToDevice));
    
    //Move points to GPU
    // vec3f *pt_h = (vec3f*)malloc((numNod+numCHIEF)*sizeof(vec3f));
    // for(i=0;i<numNod;i++) {
    //     pt_h[i] = nod[i];
    // }
    // for(i=0;i<numCHIEF;i++) {
    //     pt_h[numNod+i] = chief[i];
    // }
    
    vec3f *pt_d;
    CUDA_CALL(hipMalloc(&pt_d, (numNod + numCHIEF) * sizeof(vec3f)));
    CUDA_CALL(hipMemcpy(pt_d, nod, numNod * sizeof(vec3f),hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpy(pt_d + numNod, chief, numCHIEF * sizeof(vec3f),hipMemcpyHostToDevice));
    
    CUDA_CALL(hipEventRecord(start));
    //Generate the system
    hipFloatComplex *A = (hipFloatComplex*)malloc((numNod+numCHIEF)*numNod*sizeof(hipFloatComplex));
    
    memset(A,0,(numNod+numCHIEF)*numNod*sizeof(hipFloatComplex));
    memset(B,0,(numNod+numCHIEF)*numSrc*sizeof(hipFloatComplex));

    for(i=0;i<numNod;i++) 
    {
        A[IDXC0(i,i,numNod+numCHIEF)] = make_hipFloatComplex(1,0);
    }
    
    //Initialization of B
    for(i=0;i<numNod+numCHIEF;i++) 
    {
        for(j=0;j<numSrc;j++) 
        {
            if(i < numNod)
                B[IDXC0(i,j,ldb)] = mpSrc(k,STRENGTH,src[j],nod[i]);
            else
                B[IDXC0(i,j,ldb)] = mpSrc(k,STRENGTH,src[j],chief[i-numNod]);
        }
    }
    
    hipFloatComplex *A_d, *B_d;
    CUDA_CALL(hipMalloc(&A_d,(numNod+numCHIEF)*numNod*sizeof(hipFloatComplex)));
    CUDA_CALL(hipMemcpy(A_d,A,(numNod+numCHIEF)*numNod*sizeof(hipFloatComplex),hipMemcpyHostToDevice));
    
    CUDA_CALL(hipMalloc(&B_d,(numNod+numCHIEF)*numSrc*sizeof(hipFloatComplex)));
    CUDA_CALL(hipMemcpy(B_d,B,(numNod+numCHIEF)*numSrc*sizeof(hipFloatComplex),hipMemcpyHostToDevice));
    
    int xNumBlocks, xWidth = 16, yNumBlocks, yWidth = 16;
    xNumBlocks = (numNod+numCHIEF+xWidth-1)/xWidth;
    yNumBlocks = (numElem+yWidth-1)/yWidth;
    dim3 gridLayout, blockLayout;
    gridLayout.x = xNumBlocks;
    gridLayout.y = yNumBlocks;
    
    blockLayout.x = xWidth;
    blockLayout.y = yWidth;
    
    atomicPtsElems_nsgl<<<gridLayout,blockLayout>>>(k,pt_d,numNod,0,numNod+numCHIEF-1,
            elem_d,numElem,A_d,numNod+numCHIEF,B_d,numSrc,ldb);
    atomicPtsElems_sgl<<<yNumBlocks,yWidth>>>(k,pt_d,elem_d,numElem,A_d,numNod+numCHIEF,
            B_d,numSrc,ldb);
    
    //Solving the system
    hipsolverHandle_t cusolverH = NULL;
    CUSOLVER_CALL(hipsolverDnCreate(&cusolverH));
    
    //A = QR
    int lwork;
    CUSOLVER_CALL(hipsolverDnCgeqrf_bufferSize(cusolverH,numNod+numCHIEF,numNod,A_d
            ,numNod+numCHIEF,&lwork));
    
    hipFloatComplex *workspace_d;
    CUDA_CALL(hipMalloc(&workspace_d,lwork*sizeof(hipFloatComplex)));
    hipFloatComplex *tau_d;
    CUDA_CALL(hipMalloc(&tau_d,(numNod+numCHIEF)*sizeof(hipFloatComplex)));
    int *deviceInfo_d, deviceInfo;
    CUDA_CALL(hipMalloc(&deviceInfo_d,sizeof(int)));
    
    
    CUSOLVER_CALL(hipsolverDnCgeqrf(cusolverH,numNod+numCHIEF,numNod,A_d,numNod+numCHIEF,
            tau_d,workspace_d,lwork,deviceInfo_d));
    CUDA_CALL(hipMemcpy(&deviceInfo,deviceInfo_d,sizeof(int),hipMemcpyDeviceToHost));
    
    //B = (Q^H)*B
    CUSOLVER_CALL(hipsolverDnCunmqr(cusolverH,HIPBLAS_SIDE_LEFT,HIPBLAS_OP_C,numNod+numCHIEF,numSrc,
            numNod,A_d,numNod+numCHIEF,tau_d,B_d,ldb,workspace_d,lwork,deviceInfo_d));
    CUDA_CALL(hipMemcpy(&deviceInfo,deviceInfo_d,sizeof(int),hipMemcpyDeviceToHost));
    
    //Solve Rx = B
    hipFloatComplex alpha = make_hipFloatComplex(1,0);
    hipblasHandle_t cublasH;
    CUBLAS_CALL(hipblasCreate(&cublasH));
    CUBLAS_CALL(hipblasCtrsm(cublasH,HIPBLAS_SIDE_LEFT,HIPBLAS_FILL_MODE_UPPER,
            HIPBLAS_OP_N,HIPBLAS_DIAG_NON_UNIT,numNod,numSrc,&alpha,A_d,numNod+numCHIEF,B_d,ldb));
    CUDA_CALL(hipEventRecord(stop));
    CUDA_CALL(hipEventSynchronize(stop));
    CUDA_CALL(hipMemcpy(B,B_d,ldb*numSrc*sizeof(hipFloatComplex),hipMemcpyDeviceToHost));
    
    float milliseconds = 0;
    CUDA_CALL(hipEventElapsedTime(&milliseconds,start,stop));
    printf("Elapsed system solving time: %f milliseconds.\n",milliseconds);
    
    //release memory
    CUDA_CALL(hipEventDestroy(start));
    CUDA_CALL(hipEventDestroy(stop));
    CUDA_CALL(hipFree(A_d));
    CUDA_CALL(hipFree(B_d));
    CUDA_CALL(hipFree(tau_d));
    CUDA_CALL(hipFree(workspace_d));
    CUDA_CALL(hipFree(deviceInfo_d));
    CUBLAS_CALL(hipblasDestroy(cublasH));
    CUSOLVER_CALL(hipsolverDnDestroy(cusolverH));
    CUDA_CALL(hipFree(elem_d));
    CUDA_CALL(hipFree(pt_d));
    free(A);
    return EXIT_SUCCESS;
}

int bemSolver_dir(const float k, const tri_elem *elem, const int numElem, 
        const vec3f *nod, const int numNod, const vec3f *chief, const int numCHIEF, 
        const vec3f *dir, const int numSrc, hipFloatComplex *B, const int ldb)
{
    int i, j;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    //Move elements to GPU
    tri_elem *elem_d;
    CUDA_CALL(hipMalloc(&elem_d,numElem*sizeof(tri_elem)));
    CUDA_CALL(hipMemcpy(elem_d,elem,numElem*sizeof(tri_elem),hipMemcpyHostToDevice));
    
    //Move points to GPU
    // vec3f *pt_h = (vec3f*)malloc((numNod+numCHIEF)*sizeof(vec3f));
    // for(i=0;i<numNod;i++) {
    //     pt_h[i] = nod[i];
    // }
    // for(i=0;i<numCHIEF;i++) {
    //     pt_h[numNod+i] = chief[i];
    // }
    
    vec3f *pt_d;
    CUDA_CALL(hipMalloc(&pt_d,(numNod+numCHIEF)*sizeof(vec3f)));
    CUDA_CALL(hipMemcpy(pt_d,nod,numNod*sizeof(vec3f),hipMemcpyHostToDevice));
    CUDA_CALL(hipMemcpy(pt_d+numNod,chief,numCHIEF*sizeof(vec3f),hipMemcpyHostToDevice));
    
    CUDA_CALL(hipEventRecord(start));
    //Generate the system
    hipFloatComplex *A = (hipFloatComplex*)malloc((numNod+numCHIEF)*numNod*sizeof(hipFloatComplex));
    memset(A,0,(numNod+numCHIEF)*numNod*sizeof(hipFloatComplex));

    for(i=0;i<numNod;i++) 
    {
        A[IDXC0(i,i,numNod+numCHIEF)] = make_hipFloatComplex(1,0);
    }
    
    //Initialization of B
    for(i=0;i<numNod+numCHIEF;i++) 
    {
        for(j=0;j<numSrc;j++) 
        {
            if(i < numNod)
                //B[IDXC0(i,j,ldb)] = ptSrc(k,STRENGTH,src[j],nod[i]);
                B[IDXC0(i,j,ldb)] = dirSrc(k,STRENGTH,dir[j],nod[i]);
            else
                //B[IDXC0(i,j,ldb)] = ptSrc(k,STRENGTH,src[j],chief[i - numNod]);
                B[IDXC0(i,j,ldb)] = dirSrc(k,STRENGTH,dir[j],chief[i-numNod]);
        }
    }
    
    hipFloatComplex *A_d, *B_d;
    CUDA_CALL(hipMalloc(&A_d,(numNod+numCHIEF)*numNod*sizeof(hipFloatComplex)));
    CUDA_CALL(hipMemcpy(A_d,A,(numNod+numCHIEF)*numNod*sizeof(hipFloatComplex),hipMemcpyHostToDevice));
    
    CUDA_CALL(hipMalloc(&B_d,(numNod+numCHIEF)*numSrc*sizeof(hipFloatComplex)));
    CUDA_CALL(hipMemcpy(B_d,B,(numNod+numCHIEF)*numSrc*sizeof(hipFloatComplex),hipMemcpyHostToDevice));
    
    int xNumBlocks, xWidth = 16, yNumBlocks, yWidth = 16;
    xNumBlocks = (numNod+numCHIEF+xWidth-1)/xWidth;
    yNumBlocks = (numElem+yWidth-1)/yWidth;
    dim3 gridLayout, blockLayout;
    gridLayout.x = xNumBlocks;
    gridLayout.y = yNumBlocks;
    
    blockLayout.x = xWidth;
    blockLayout.y = yWidth;
    
    atomicPtsElems_nsgl<<<gridLayout,blockLayout>>>(k,pt_d,numNod,0,numNod+numCHIEF-1,
            elem_d,numElem,A_d,numNod+numCHIEF,B_d,numSrc,ldb);
    atomicPtsElems_sgl<<<yNumBlocks,yWidth>>>(k,pt_d,elem_d,numElem,A_d,numNod+numCHIEF,
            B_d,numSrc,ldb);
    
    //Solving the system
    hipsolverHandle_t cusolverH = NULL;
    CUSOLVER_CALL(hipsolverDnCreate(&cusolverH));
    
    //A = QR
    int lwork;
    CUSOLVER_CALL(hipsolverDnCgeqrf_bufferSize(cusolverH,numNod+numCHIEF,numNod,A_d
            ,numNod+numCHIEF,&lwork));
    
    hipFloatComplex *workspace_d;
    CUDA_CALL(hipMalloc(&workspace_d,lwork*sizeof(hipFloatComplex)));
    hipFloatComplex *tau_d;
    CUDA_CALL(hipMalloc(&tau_d,(numNod+numCHIEF)*sizeof(hipFloatComplex)));
    int *deviceInfo_d, deviceInfo;
    CUDA_CALL(hipMalloc(&deviceInfo_d,sizeof(int)));
    
    
    CUSOLVER_CALL(hipsolverDnCgeqrf(cusolverH,numNod+numCHIEF,numNod,A_d,numNod+numCHIEF,
            tau_d,workspace_d,lwork,deviceInfo_d));
    CUDA_CALL(hipMemcpy(&deviceInfo,deviceInfo_d,sizeof(int),hipMemcpyDeviceToHost));
    
    //B = (Q^H)*B
    CUSOLVER_CALL(hipsolverDnCunmqr(cusolverH,HIPBLAS_SIDE_LEFT,HIPBLAS_OP_C,numNod+numCHIEF,numSrc,
            numNod,A_d,numNod+numCHIEF,tau_d,B_d,ldb,workspace_d,lwork,deviceInfo_d));
    CUDA_CALL(hipMemcpy(&deviceInfo,deviceInfo_d,sizeof(int),hipMemcpyDeviceToHost));
    
    //Solve Rx = B
    hipFloatComplex alpha = make_hipFloatComplex(1,0);
    hipblasHandle_t cublasH;
    CUBLAS_CALL(hipblasCreate(&cublasH));
    CUBLAS_CALL(hipblasCtrsm(cublasH,HIPBLAS_SIDE_LEFT,HIPBLAS_FILL_MODE_UPPER,
            HIPBLAS_OP_N,HIPBLAS_DIAG_NON_UNIT,numNod,numSrc,&alpha,A_d,numNod+numCHIEF,B_d,ldb));
    CUDA_CALL(hipEventRecord(stop));
    CUDA_CALL(hipEventSynchronize(stop));
    CUDA_CALL(hipMemcpy(B,B_d,ldb*numSrc*sizeof(hipFloatComplex),hipMemcpyDeviceToHost));
    
    float milliseconds = 0;
    CUDA_CALL(hipEventElapsedTime(&milliseconds,start,stop));
    printf("Elapsed system solving time: %f milliseconds.\n",milliseconds);
    
    //release memory
    CUDA_CALL(hipEventDestroy(start));
    CUDA_CALL(hipEventDestroy(stop));
    CUDA_CALL(hipFree(A_d));
    CUDA_CALL(hipFree(B_d));
    CUDA_CALL(hipFree(tau_d));
    CUDA_CALL(hipFree(workspace_d));
    CUDA_CALL(hipFree(deviceInfo_d));
    CUBLAS_CALL(hipblasDestroy(cublasH));
    CUSOLVER_CALL(hipsolverDnDestroy(cusolverH));
    CUDA_CALL(hipFree(elem_d));
    CUDA_CALL(hipFree(pt_d));
    free(A);
    return EXIT_SUCCESS;
}

__host__ gsl_complex gsl_sf_bessel_hl(const int l, const double s)
{
    double x = gsl_sf_bessel_jl(l,s);
    double y = gsl_sf_bessel_yl(l,s);
    gsl_complex z = gsl_complex_rect(x,y);
    return z;
}

double jprime(const int n, const double r)
{
    double result;
    if(n == 0) {
        result = -gsl_sf_bessel_jl(1,r);
    } else {
        result = gsl_sf_bessel_jl(n-1,r)-(n+1)*gsl_sf_bessel_jl(n,r)/r;
    }
    return result;
}

gsl_complex hprime(const int n, const double r)
{
    gsl_complex result;
    if(n == 0) {
        result = gsl_complex_negative(gsl_sf_bessel_hl(1,r));
    } else {
        result = gsl_complex_sub(gsl_sf_bessel_hl(n-1,r),gsl_complex_mul_real(gsl_sf_bessel_hl(n,r),(n+1)/r));
    }
    return result;
}

__host__ __device__ vec3f sph2vec(const sph3f s)
{
    float r = s.coords[0], theta = s.coords[1], phi = s.coords[2];
    float x = r*sinf(theta)*cosf(phi), y = r*sinf(theta)*sinf(phi), z = r*cosf(theta);
    vec3f result;
    result.coords[0] = x;
    result.coords[1] = y;
    result.coords[2] = z;
    return result;
}

__host__ __device__ vec3d sph2vec(const sph3d s)
{
    double r = s.coords[0], theta = s.coords[1], phi = s.coords[2];
    double x = r*sin(theta)*cos(phi), y = r*sin(theta)*sin(phi), z = r*cos(theta);
    vec3d result;
    result.coords[0] = x;
    result.coords[1] = y;
    result.coords[2] = z;
    return result;
}

__host__ __device__ sph3f vec2sph(const vec3f s)
{
    sph3f temp;
    temp.coords[0] = sqrtf(powf(s.coords[0],2)+powf(s.coords[1],2)+powf(s.coords[2],2));
    temp.coords[1] = acosf(s.coords[2]/(temp.coords[0]));
    temp.coords[2] = atan2f(s.coords[1],s.coords[0]);
    return temp;
}

__host__ __device__ sph3d vec2sph(const vec3d s)
{
    sph3d temp;
    temp.coords[0] = sqrt(pow(s.coords[0],2)+pow(s.coords[1],2)+pow(s.coords[2],2));
    temp.coords[1] = acos(s.coords[2]/(temp.coords[0]));
    temp.coords[2] = atan2(s.coords[1],s.coords[0]);
    return temp;
}

__device__ hipFloatComplex extrapolation_dir(const float wavNum, const vec3f x, 
        const tri_elem* elem, const int numElem, const vec3f* pt, 
        const hipFloatComplex* p, const float strength, const vec3f dir)
{
    /*field extrapolation from the surface to a single point in free space
     wavNum: wave number
     elem: pointer for all elements
     pt: pointer for all points
     x: the point in free space
     dir: the direction of the plane wave*/
    hipFloatComplex result = dirSrc(wavNum,strength,dir,x);
    hipFloatComplex temp;
    for(int i=0;i<numElem;i++) {
        vec3f nod[3];
        for(int j=0;j<3;j++) {
            nod[j] = pt[elem[i].nod[j]];
        }
        hipFloatComplex gCoeff[3], hCoeff[3]; 
        float cCoeff[3];
        g_h_c_nsgl(wavNum,x,nod,gCoeff,hCoeff,cCoeff);
        for(int j=0;j<3;j++) {
            temp = hipCdivf(elem[i].bc[2],elem[i].bc[1]);
            temp = hipCmulf(temp,gCoeff[j]);
            result = hipCsubf(result,temp);
            temp = hipCdivf(elem[i].bc[0],elem[i].bc[1]);
            temp = hipCmulf(temp,gCoeff[j]);
            temp = hipCsubf(hCoeff[j],temp);
            temp = hipCmulf(temp,p[elem[i].nod[j]]);
            result = hipCsubf(result,temp);
        }
    }
    return result;
}

__device__ hipFloatComplex extrapolation_pt(const float wavNum, const vec3f x, 
        const tri_elem* elem, const int numElem, const vec3f* pt, 
        const hipFloatComplex* p, const float strength, const vec3f src)
{
    /*field extrapolation from the surface to a single point in free space
     x: the single point in free space
     elem: pointer to mesh elements
     pt: pointer to mesh nod and chief points
     p: surface pressure
     strength: intensity of the source
     src: source location*/
    hipFloatComplex result = ptSrc(wavNum,strength,src,x);
    hipFloatComplex temp;
    for(int i=0;i<numElem;i++) {
        vec3f nod[3];
        for(int j=0;j<3;j++) {
            nod[j] = pt[elem[i].nod[j]];
        }
        hipFloatComplex gCoeff[3], hCoeff[3]; 
        float cCoeff[3];
        g_h_c_nsgl(wavNum,x,nod,gCoeff,hCoeff,cCoeff);
        for(int j=0;j<3;j++) {
            temp = hipCdivf(elem[i].bc[2],elem[i].bc[1]);
            temp = hipCmulf(temp,gCoeff[j]);
            result = hipCsubf(result,temp);
            temp = hipCdivf(elem[i].bc[0],elem[i].bc[1]);
            temp = hipCmulf(temp,gCoeff[j]);
            temp = hipCsubf(hCoeff[j],temp);
            temp = hipCmulf(temp,p[elem[i].nod[j]]);
            result = hipCsubf(result,temp);
        }
    }
    return result;
}

__device__ hipFloatComplex extrapolation_mp(const float wavNum, const vec3f x, 
        const tri_elem* elem, const int numElem, const vec3f* pt, 
        const hipFloatComplex* p, const float strength, const vec3f src)
{
    /*field extrapolation from the surface to a single monopole in free space
     x: the single point in free space
     elem: pointer to mesh elements
     pt: pointer to mesh nod and chief points
     p: surface pressure
     strength: intensity of the source
     src: source location
     return: sound pressure at the extrapolation point*/
    hipFloatComplex result = mpSrc(wavNum,strength,src,x);
    hipFloatComplex temp;
    for(int i=0;i<numElem;i++) {
        vec3f nod[3];
        for(int j=0;j<3;j++) {
            nod[j] = pt[elem[i].nod[j]];
        }
        hipFloatComplex gCoeff[3], hCoeff[3]; 
        float cCoeff[3];
        g_h_c_nsgl(wavNum,x,nod,gCoeff,hCoeff,cCoeff);
        for(int j=0;j<3;j++) {
            temp = hipCdivf(elem[i].bc[2],elem[i].bc[1]);
            temp = hipCmulf(temp,gCoeff[j]);
            result = hipCsubf(result,temp);
            temp = hipCdivf(elem[i].bc[0],elem[i].bc[1]);
            temp = hipCmulf(temp,gCoeff[j]);
            temp = hipCsubf(hCoeff[j],temp);
            temp = hipCmulf(temp,p[elem[i].nod[j]]);
            result = hipCsubf(result,temp);
        }
    }
    return result;
}

__global__ void extrapolations_dir(const float wavNum, const vec3f* expPt, const int numExpPt,
        const tri_elem* elem, const int numElem, const vec3f* pt, const hipFloatComplex* p, 
        const float strength, const vec3f dir, hipFloatComplex *p_exp)
{
    /*
     extrapolation from surface pressure to multiple points in free space
     wavNum: wave number
     expPt: extrapolation points in free space
     p: surface pressure
     dir: direction of the plane wave
     p_exp: pressure at the extrapolation points
     */
    int idx = blockIdx.x*blockDim.x+threadIdx.x;
    if(idx < numExpPt) {
        p_exp[idx] = extrapolation_dir(wavNum,expPt[idx],elem,numElem,pt,p,strength,dir);
    }
}

__global__ void extrapolations_pt(const float wavNum, const vec3f* expPt, const int numExpPt,
        const tri_elem* elem, const int numElem, const vec3f* pt, const hipFloatComplex* p, 
        const float strength, const vec3f src, hipFloatComplex *p_exp)
{
    /*extrapolation from surface pressure to multiple points in free space
     wavNum: wave number
     expPt:  extrapolation  points in free space
     p: surface pressure
     src: location of the source
     p_exp: pressure at the extrapolation points*/
    int idx = blockIdx.x*blockDim.x+threadIdx.x;
    if(idx < numExpPt) {
        p_exp[idx] = extrapolation_pt(wavNum,expPt[idx],elem,numElem,pt,p,strength,src);
    }
}

__global__ void extrapolations_mp(const float wavNum, const vec3f* expPt, const int numExpPt,
        const tri_elem* elem, const int numElem, const vec3f* pt, const hipFloatComplex* p, 
        const float strength, const vec3f src, hipFloatComplex *p_exp)
{
    int idx = blockIdx.x*blockDim.x+threadIdx.x;
    if(idx < numExpPt) {
        p_exp[idx] = extrapolation_mp(wavNum,expPt[idx],elem,numElem,pt,p,strength,src);
    }
}

int field_extrapolation_single_dir(const float wavNum, const vec3f* expPt, const int numExpPt, 
        const tri_elem* elem, const int numElem, const vec3f* pt, const int numPt, 
        const hipFloatComplex* p, const float strength, const vec3f dir, hipFloatComplex *pExp)
{
    /*extrapolation of acoustic field from surface pressure
     wavNum: wave number
     expPt: extrapolation points in free space
     elem: pointer to mesh elements
     pt: pointer to mesh nod and chief points
     p: surface pressure
     strength: intensity of the sound source
     dir: direction of the plane wave
     pExp: pressure at extrapolation points*/
    int width = 16, numBlock = (numExpPt+width-1)/width;
    
    // allocate memory on GPU and copy data to GPU memory
    vec3f *expPt_d, *pt_d;
    tri_elem *elem_d;
    hipFloatComplex *p_d, *pExp_d;
    
    CUDA_CALL(hipMalloc(&expPt_d,numExpPt*sizeof(vec3f)));
    CUDA_CALL(hipMemcpy(expPt_d,expPt,numExpPt*sizeof(vec3f),hipMemcpyHostToDevice));
    
    CUDA_CALL(hipMalloc(&pt_d,numPt*sizeof(vec3f)));
    CUDA_CALL(hipMemcpy(pt_d,pt,numPt*sizeof(vec3f),hipMemcpyHostToDevice));
    
    CUDA_CALL(hipMalloc(&elem_d,numElem*sizeof(tri_elem)));
    CUDA_CALL(hipMemcpy(elem_d,elem,numElem*sizeof(tri_elem),hipMemcpyHostToDevice));
    
    CUDA_CALL(hipMalloc(&p_d,numPt*sizeof(hipFloatComplex)));
    CUDA_CALL(hipMemcpy(p_d,p,numPt*sizeof(hipFloatComplex),hipMemcpyHostToDevice));
    
    CUDA_CALL(hipMalloc(&pExp_d,numExpPt*sizeof(hipFloatComplex)));
    
    extrapolations_dir<<<numBlock,width>>>(wavNum,expPt_d,numExpPt,elem_d,numElem,pt_d,p_d,
            strength,dir,pExp_d);
    
    CUDA_CALL(hipMemcpy(pExp,pExp_d,numExpPt*sizeof(hipFloatComplex),hipMemcpyDeviceToHost));
    
    CUDA_CALL(hipFree(expPt_d));
    CUDA_CALL(hipFree(pt_d));
    CUDA_CALL(hipFree(elem_d));
    CUDA_CALL(hipFree(p_d));
    CUDA_CALL(hipFree(pExp_d));
    
    return EXIT_SUCCESS;
}

int field_extrapolation_single_pt(const float wavNum, const vec3f* expPt, const int numExpPt, 
        const tri_elem* elem, const int numElem, const vec3f* pt, const int numPt, 
        const hipFloatComplex* p, const float strength, const vec3f src, hipFloatComplex *pExp)
{
    /*Extrapolation of an acoustic field from surface pressure and a single point source
     wavNum: wave number
     expPt: pointer for extrapolation points
     elem: mesh elements
     pt: nod and chief points
     p: surface pressure
     strength: intensity of a source
     src: location of the point source
     pExp: pressure at extrapolation points*/
    int width = 16, numBlock = (numExpPt+width-1)/width;
    
    // allocate memory on GPU and copy data to GPU memory
    vec3f *expPt_d, *pt_d;
    tri_elem *elem_d;
    hipFloatComplex *p_d, *pExp_d;
    
    CUDA_CALL(hipMalloc(&expPt_d,numExpPt*sizeof(vec3f)));
    CUDA_CALL(hipMemcpy(expPt_d,expPt,numExpPt*sizeof(vec3f),hipMemcpyHostToDevice));
    
    CUDA_CALL(hipMalloc(&pt_d,numPt*sizeof(vec3f)));
    CUDA_CALL(hipMemcpy(pt_d,pt,numPt*sizeof(vec3f),hipMemcpyHostToDevice));
    
    CUDA_CALL(hipMalloc(&elem_d,numElem*sizeof(tri_elem)));
    CUDA_CALL(hipMemcpy(elem_d,elem,numElem*sizeof(tri_elem),hipMemcpyHostToDevice));
    
    CUDA_CALL(hipMalloc(&p_d,numPt*sizeof(hipFloatComplex)));
    CUDA_CALL(hipMemcpy(p_d,p,numPt*sizeof(hipFloatComplex),hipMemcpyHostToDevice));
    
    CUDA_CALL(hipMalloc(&pExp_d,numExpPt*sizeof(hipFloatComplex)));
    
    extrapolations_pt<<<numBlock,width>>>(wavNum,expPt_d,numExpPt,elem_d,numElem,pt_d,p_d,
            strength,src,pExp_d);
    
    CUDA_CALL(hipMemcpy(pExp,pExp_d,numExpPt*sizeof(hipFloatComplex),hipMemcpyDeviceToHost));
    
    CUDA_CALL(hipFree(expPt_d));
    CUDA_CALL(hipFree(pt_d));
    CUDA_CALL(hipFree(elem_d));
    CUDA_CALL(hipFree(p_d));
    CUDA_CALL(hipFree(pExp_d));
    
    return EXIT_SUCCESS;
}

int field_extrapolation_single_mp(const float wavNum, const vec3f* expPt, const int numExpPt, 
        const tri_elem* elem, const int numElem, const vec3f* pt, const int numPt, 
        const hipFloatComplex* p, const float strength, const vec3f src, hipFloatComplex *pExp)
{
    int width = 16, numBlock = (numExpPt+width-1)/width;
    
    // allocate memory on GPU and copy data to GPU memory
    vec3f *expPt_d, *pt_d;
    tri_elem *elem_d;
    hipFloatComplex *p_d, *pExp_d;
    
    CUDA_CALL(hipMalloc(&expPt_d,numExpPt*sizeof(vec3f)));
    CUDA_CALL(hipMemcpy(expPt_d,expPt,numExpPt*sizeof(vec3f),hipMemcpyHostToDevice));
    
    CUDA_CALL(hipMalloc(&pt_d,numPt*sizeof(vec3f)));
    CUDA_CALL(hipMemcpy(pt_d,pt,numPt*sizeof(vec3f),hipMemcpyHostToDevice));
    
    CUDA_CALL(hipMalloc(&elem_d,numElem*sizeof(tri_elem)));
    CUDA_CALL(hipMemcpy(elem_d,elem,numElem*sizeof(tri_elem),hipMemcpyHostToDevice));
    
    CUDA_CALL(hipMalloc(&p_d,numPt*sizeof(hipFloatComplex)));
    CUDA_CALL(hipMemcpy(p_d,p,numPt*sizeof(hipFloatComplex),hipMemcpyHostToDevice));
    
    CUDA_CALL(hipMalloc(&pExp_d,numExpPt*sizeof(hipFloatComplex)));
    
    extrapolations_mp<<<numBlock,width>>>(wavNum,expPt_d,numExpPt,elem_d,numElem,pt_d,p_d,
            strength,src,pExp_d);
    
    CUDA_CALL(hipMemcpy(pExp,pExp_d,numExpPt*sizeof(hipFloatComplex),hipMemcpyDeviceToHost));
    
    CUDA_CALL(hipFree(expPt_d));
    CUDA_CALL(hipFree(pt_d));
    CUDA_CALL(hipFree(elem_d));
    CUDA_CALL(hipFree(p_d));
    CUDA_CALL(hipFree(pExp_d));
    
    return EXIT_SUCCESS;
}

vec3f rectCoordDbl2rectCoordFlt(const vec3d t)
{
    vec3f result;
    for(int i=0;i<3;i++) {
        result.coords[i] = t.coords[i];
    }
    return result;
}

void rectCoordDblArr2rectCoordFltArr(const vec3d* dArr, 
        const int num, vec3f* fArr)
{
    for(int i=0;i<num;i++) {
        fArr[i] = rectCoordDbl2rectCoordFlt(dArr[i]);
    }
}

void reorgField(hipFloatComplex* field, const int l)
{
    /*re-organize the acoustic fields from the order of z, y, x to x, y, z*/
    int totalNum = pow(8,l), dimNum = pow(2,l);
    hipFloatComplex *temp = (hipFloatComplex*)malloc(totalNum*sizeof(hipFloatComplex));
    memcpy(temp,field,totalNum*sizeof(hipFloatComplex));
    
    // reorganize
    for(int x=0;x<dimNum;x++) {
        for(int y=0;y<dimNum;y++) {
            for(int z=0;z<dimNum;z++) {
                int idx_old = x*dimNum*dimNum+y*dimNum+z;
                int idx_new = z*dimNum*dimNum+y*dimNum+x;
                field[idx_new] = temp[idx_old];
            }
        }
    }
    free(temp);
}

int genFields_MultiPtSrcSglObj(const float strength, const float wavNum, 
        const vec3f* srcs, const int numSrcs, const vec3d* pts, const int numPts, 
        const tri_elem* elems, const int numElems, const vec3d cnr, const double d, 
        const int level, hipFloatComplex* fields)
{
    /*generate an acoustic field with a given boundary
     level: octree level
     cnr: lowest corner of the bounding box
     d: side length of the bounding box
     fields: pressure array equal to the number of boxes at level l times number of sources*/
    vec3f *pts_f = (vec3f*)malloc(numPts*sizeof(vec3f));
    rectCoordDblArr2rectCoordFltArr(pts,numPts,pts_f);
    
    // generate chief points
    vec3f chief[NUMCHIEF];
    genCHIEF(pts_f,numPts,elems,numElems,chief,NUMCHIEF);
    
    // allocate memory for the right-hand side of the linear system
    hipFloatComplex *B = (hipFloatComplex*)malloc((numPts+NUMCHIEF)*numSrcs*sizeof(hipFloatComplex));
    // solve the linear system to get the surface pressure
    HOST_CALL(bemSolver_mp(wavNum,elems,numElems,pts_f,numPts,chief,NUMCHIEF,srcs,numSrcs,B,numPts+NUMCHIEF));
    
    // compute the extrapolation points of the field
    // note that the indices first increase in z, then in y and at last in x
    int numExpPts = (int)pow(8,level);
    vec3d *expPts = (vec3d*)malloc(numExpPts*sizeof(vec3d));
    for(int i=0;i<numExpPts;i++) {
        vec3d pt_scaled = boxCenter(i,level);
        vec3d pt_descaled = descale(pt_scaled,cnr,d);
        expPts[i] = pt_descaled;
    }
    vec3f *expPts_f = (vec3f*)malloc(numExpPts*sizeof(vec3f));
    rectCoordDblArr2rectCoordFltArr(expPts,numExpPts,expPts_f);
    free(expPts);
    
    // extrapolate the acoustic field from the surface to free space
    hipFloatComplex *field = (hipFloatComplex*)malloc(numExpPts*sizeof(hipFloatComplex));
    for(int i=0;i<numSrcs;i++) {
        HOST_CALL(field_extrapolation_single_pt(wavNum,expPts_f,numExpPts,elems,numElems,
                pts_f,numPts,&B[i*(numPts+NUMCHIEF)],strength,srcs[i],field));
        reorgField(field,level);
        memcpy(&fields[i*numExpPts],field,numExpPts*sizeof(hipFloatComplex));
    }
    
    return EXIT_SUCCESS;
}

gsl_complex rigid_sphere_plane(const double wavNum, const double strength, const double a, 
        const double r, const double theta)
{
    gsl_complex result = gsl_complex_rect(0,0), temp_c;
    const int numTrunc = 70;
    for(int n=0;n<numTrunc;n++)
    {
        temp_c = gsl_complex_div(gsl_complex_rect(jprime(n,wavNum*a),0),hprime(n,wavNum*a));
        temp_c = gsl_complex_mul(temp_c,gsl_sf_bessel_hl(n,wavNum*r));
        temp_c = gsl_complex_sub(gsl_complex_rect(gsl_sf_bessel_jl(n,wavNum*r),0),temp_c);
        temp_c = gsl_complex_mul(gsl_complex_pow_real(gsl_complex_rect(0,1),n),temp_c);
        temp_c = gsl_complex_mul_real(temp_c,2*n+1);
        temp_c = gsl_complex_mul_real(temp_c,gsl_sf_legendre_Pl(n,cos(theta)));
        result = gsl_complex_add(result,temp_c);
    }
    result = gsl_complex_mul_real(result,strength);
    return result;
}

gsl_complex rigid_sphere_point(const double wavNum, const double strength, const double rs, 
        const double a, const vec3d y)
{
    const int truncNum = 100;
    const vec3d src = {0,0,rs};
    vec3d temp_cart_coord = vecSub(y,src);
    sph3d temp_sph_coord = vec2sph(temp_cart_coord);
    double R = temp_sph_coord.coords[0];
    temp_sph_coord = vec2sph(y);
    double r = temp_sph_coord.coords[0];
    double theta = temp_sph_coord.coords[1];
    gsl_complex result = gsl_complex_rect(strength*cos(wavNum*R)/(4*PI*R),strength*sin(wavNum*R)/(4*PI*R));
    for(int n=0;n<truncNum;n++) {
        gsl_complex temp[2];
        double t = (n+0.5)*jprime(n,wavNum*a)*wavNum*strength/(2*PI)*gsl_sf_legendre_Pl(n,cos(theta));
        temp[0] = gsl_complex_rect(0,t);
        temp[1] = gsl_complex_mul(gsl_sf_bessel_hl(n,wavNum*rs),gsl_sf_bessel_hl(n,wavNum*r));
        temp[0] = gsl_complex_mul(temp[0],temp[1]);
        temp[0] = gsl_complex_div(temp[0],hprime(n,wavNum*a));
        result = gsl_complex_sub(result,temp[0]);
    }
    return result;
}

gsl_complex rigid_sphere_monopole(const double wavNum, const double strength, const double rs, 
        const double a, const vec3d y)
{
    const int truncNum = 100;
    const vec3d src = {0,0,rs};
    vec3d temp_cart_coord = vecSub(y,src);
    sph3d temp_sph_coord = vec2sph(temp_cart_coord);
    double R = temp_sph_coord.coords[0];
    temp_sph_coord = vec2sph(y);
    double r = temp_sph_coord.coords[0];
    double theta = temp_sph_coord.coords[1];
    gsl_complex result = gsl_complex_rect(cos(wavNum*R)/(4*PI*R),sin(wavNum*R)/(4*PI*R));
    result = gsl_complex_mul(result,gsl_complex_rect(0,-RHO_AIR*SPEED_SOUND*wavNum*strength));
    for(int n=0;n<truncNum;n++) {
        gsl_complex temp[2];
        double t = RHO_AIR*SPEED_SOUND*strength*pow(wavNum,2)/(2*PI)*(n+0.5)*jprime(n,wavNum*a)*gsl_sf_legendre_Pl(n,cos(theta));
        temp[0] = gsl_complex_rect(t,0);
        temp[1] = gsl_complex_mul(gsl_sf_bessel_hl(n,wavNum*rs),gsl_sf_bessel_hl(n,wavNum*r));
        temp[0] = gsl_complex_mul(temp[0],temp[1]);
        temp[0] = gsl_complex_div(temp[0],hprime(n,wavNum*a));
        result = gsl_complex_sub(result,temp[0]);
    }
    return result;
}
