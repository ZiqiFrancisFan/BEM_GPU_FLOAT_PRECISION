#include "hip/hip_runtime.h"
/*
 * To change this license header, choose License Headers in Project Properties.
 * To change this template file, choose Tools | Templates
 * and open the template in the editor.
 */
#include "geometry.h"
#include "octree.h"
#include <stdio.h>
#include <stdlib.h>
#include <float.h>

__constant__ vec3d BASES[3];

vec3d bases[3];

__host__ int CopyBasesToConstant()
{
    vec3d bases[3];
    bases[0].coords[0] = 1;
    bases[0].coords[1] = 0;
    bases[0].coords[2] = 0;
    
    bases[1].coords[0] = 0;
    bases[1].coords[1] = 1;
    bases[1].coords[2] = 0;
    
    bases[2].coords[0] = 0;
    bases[2].coords[1] = 0;
    bases[2].coords[2] = 1;
    
    CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(BASES),bases,3*sizeof(vec3d),0,hipMemcpyHostToDevice));
    return EXIT_SUCCESS;
}

__host__ void SetHostBases()
{
    bases[0].coords[0] = 1;
    bases[0].coords[1] = 0;
    bases[0].coords[2] = 0;
    
    bases[1].coords[0] = 0;
    bases[1].coords[1] = 1;
    bases[1].coords[2] = 0;
    
    bases[2].coords[0] = 0;
    bases[2].coords[1] = 0;
    bases[2].coords[2] = 1;
}

__host__ __device__ int deterPtPlaneRel(const vec3d pt, const plane3d plane)
{
    vec3d vec = vecSub(pt,plane.pt);
    double result = vecDotMul(plane.n,vec);
    if(result>=0) {
        // on the positive side of the plane normal
        return 1;
    } else {
        return 0;
    }
}

__host__ __device__ int deterPtCubeEdgeVolRel(const vec3d pt, const aacb3d cb)
{
    /*determine the relationship between a point and the volume bounded by edge faces
     of a cube*/
    
    //declare two vec3d arrays for nod at the bottom and the top face 
    vec3d btm[4], top[4], left[4], right[4], back[4], front[4];
    
    // declare the basis unit vectors
    vec3d dir_x = {1.0,0.0,0.0}, dir_y = {0.0,1.0,0.0}, dir_z = {0.0,0.0,1.0};
    
    //set up btm and top nod
    btm[0] = cb.cnr;
    btm[1] = vecAdd(btm[0],scaVecMul(cb.len,dir_x));
    btm[2] = vecAdd(btm[1],scaVecMul(cb.len,dir_y));
    btm[3] = vecAdd(btm[0],scaVecMul(cb.len,dir_y));
    //printVec(btm,4);
    
    top[0] = vecAdd(btm[0],scaVecMul(cb.len,dir_z));
    top[1] = vecAdd(top[0],scaVecMul(cb.len,dir_x));
    top[2] = vecAdd(top[1],scaVecMul(cb.len,dir_y));
    top[3] = vecAdd(top[0],scaVecMul(cb.len,dir_y));
    //printVec(top,4);
    
    //set up left and right nod
    left[0] = cb.cnr;
    left[1] = vecAdd(left[0],scaVecMul(cb.len,dir_x));
    left[2] = vecAdd(left[1],scaVecMul(cb.len,dir_z));
    left[3] = vecAdd(left[0],scaVecMul(cb.len,dir_z));
    //printVec(left,4);
    
    right[0] = vecAdd(left[0],scaVecMul(cb.len,dir_y));
    right[1] = vecAdd(right[0],scaVecMul(cb.len,dir_x));
    right[2] = vecAdd(right[1],scaVecMul(cb.len,dir_z));
    right[3] = vecAdd(right[0],scaVecMul(cb.len,dir_z));
    //printVec(right,4);
    
    //set up back and front nod
    back[0] = cb.cnr;
    back[1] = vecAdd(back[0],scaVecMul(cb.len,dir_y));
    back[2] = vecAdd(back[1],scaVecMul(cb.len,dir_z));
    back[3] = vecAdd(back[0],scaVecMul(cb.len,dir_z));
    //printVec(back,4);
    
    front[0] = vecAdd(back[0],scaVecMul(cb.len,dir_x));
    front[1] = vecAdd(front[0],scaVecMul(cb.len,dir_y));
    front[2] = vecAdd(front[1],scaVecMul(cb.len,dir_z));
    front[3] = vecAdd(front[0],scaVecMul(cb.len,dir_z));
    //printVec(front,4);
    
    //declare an array nrml for determining the normal of the new plane
    vec3d nrml[3];
    plane3d plane;
    int result;
    
    //deal with the bottom face
    for(int i=0;i<3;i++) {
        nrml[1] = dir_z;
        switch(i) {
            case 0: // edge determined by btm[0] and btm[1]
                nrml[0] = dir_y;
                break;
            case 1: // edge determined by btm[1] and btm[2]
                nrml[0] = scaVecMul(-1,dir_x);
                break;
            case 2: // edge determined by btm[2] and btm[3]
                nrml[0] = scaVecMul(-1,dir_y);
                break;
            case 3: // edge determined by btm[3] and btm[0]
                nrml[0] = dir_x;
                break;
            default:
                printf("Entered the wrong branch.\n");
        }
        nrml[3] = nrmlzVec(vecAdd(nrml[0],nrml[1]));
        plane.n = nrml[3];
        plane.pt = btm[i];
        result = deterPtPlaneRel(pt,plane);
        if(result == 0) {
            //printf("bottom face: %dth node\n",i);
            return 0;
        }
    }
    
    //deal with the top face
    for(int i=0;i<3;i++) {
        nrml[1] = scaVecMul(-1,dir_z);
        switch(i) {
            case 0: // edge determined by top[0] and top[1]
                nrml[0] = dir_y;
                break;
            case 1: // edge determined by top[1] and top[2]
                nrml[0] = scaVecMul(-1,dir_x);
                break;
            case 2: // edge determined by top[2] and top[3]
                nrml[0] = scaVecMul(-1,dir_y);
                break;
            case 3: // edge determined by top[3] and top[0]
                nrml[0] = dir_x;
                break;
            default:
                printf("Entered the wrong branch.\n");
        }
        nrml[3] = nrmlzVec(vecAdd(nrml[0],nrml[1]));
        plane.n = nrml[3];
        plane.pt = top[i];
        result = deterPtPlaneRel(pt,plane);
        if(result==0) {
            //printf("top face: %dth node\n",i);
            return 0;
        }
    }
    
    //deal with the left face
    for(int i=0;i<3;i++) {
        nrml[1] = dir_y;
        switch(i) {
            case 0: // edge determined by left[0] and left[1]
                nrml[0] = dir_z;
                break;
            case 1: // edge determined by left[1] and left[2]
                nrml[0] = scaVecMul(-1,dir_x);
                break;
            case 2: // edge determined by left[2] and left[3]
                nrml[0] = scaVecMul(-1,dir_z);
                break;
            case 3: // edge determined by left[3] and left[0]
                nrml[0] = dir_x;
                break;
            default:
                printf("Entered the wrong branch.\n");
        }
        nrml[3] = nrmlzVec(vecAdd(nrml[0],nrml[1]));
        plane.n = nrml[3];
        plane.pt = left[i];
        result = deterPtPlaneRel(pt,plane);
        if(result==0) {
            //printf("left face: %dth node\n",i);
            return 0;
        }
    }
    
    //deal with the right face
    for(int i=0;i<3;i++) {
        nrml[1] = scaVecMul(-1,dir_y);
        switch(i) {
            case 0: // edge determined by right[0] and right[1]
                nrml[0] = dir_z;
                break;
            case 1: // edge determined by right[1] and right[2]
                nrml[0] = scaVecMul(-1,dir_x);
                break;
            case 2: // edge determined by right[2] and right[3]
                nrml[0] = scaVecMul(-1,dir_z);
                break;
            case 3: // edge determined by btm[3] and btm[0]
                nrml[0] = dir_x;
                break;
            default:
                printf("Entered the wrong branch.\n");
        }
        nrml[3] = nrmlzVec(vecAdd(nrml[0],nrml[1]));
        plane.n = nrml[3];
        plane.pt = right[i];
        result = deterPtPlaneRel(pt,plane);
        if(result==0) {
            //printf("right face: %dth node\n",i);
            return 0;
        }
    }
    
    //deal with the back face
    for(int i=0;i<3;i++) {
        nrml[1] = dir_x;
        switch(i) {
            case 0: // edge determined by back[0] and back[1]
                nrml[0] = dir_z;
                break;
            case 1: // edge determined by btm[1] and btm[2]
                nrml[0] = scaVecMul(-1,dir_y);
                break;
            case 2: // edge determined by btm[2] and btm[3]
                nrml[0] = scaVecMul(-1,dir_z);
                break;
            case 3: // edge determined by btm[3] and btm[0]
                nrml[0] = dir_y;
                break;
            default:
                printf("Entered the wrong branch.\n");
        }
        nrml[3] = nrmlzVec(vecAdd(nrml[0],nrml[1]));
        plane.n = nrml[3];
        plane.pt = back[i];
        result = deterPtPlaneRel(pt,plane);
        if(result==0) {
            //printf("back face: %dth node\n",i);
            return 0;
        }
    }
    
    //deal with the front face
    for(int i=0;i<3;i++) {
        nrml[1] = scaVecMul(-1,dir_x);
        switch(i) {
            case 0: // edge determined by front[0] and front[1]
                nrml[0] = dir_z;
                break;
            case 1: // edge determined by front[1] and front[2]
                nrml[0] = scaVecMul(-1,dir_y);
                break;
            case 2: // edge determined by front[2] and front[3]
                nrml[0] = scaVecMul(-1,dir_z);
                break;
            case 3: // edge determined by front[3] and front[0]
                nrml[0] = dir_y;
                break;
            default:
                printf("Entered the wrong branch.\n");
        }
        nrml[3] = nrmlzVec(vecAdd(nrml[0],nrml[1]));
        plane.n = nrml[3];
        plane.pt = front[i];
        result = deterPtPlaneRel(pt,plane);
        if(result==0) {
            //printf("front face: %dth node\n",i);
            return 0;
        }
    }
    
    // if not returned 0, then return 1, the point is inside the volume
    return 1;
}

__host__ __device__ int deterPtCubeVtxVolRel(const vec3d pt, const aacb3d cb)
{
    // declare the basis unit vectors
    vec3d dir_x = {1.0,0.0,0.0}, dir_y = {0.0,1.0,0.0}, dir_z = {0.0,0.0,1.0}, 
            nrml[4], tempPt;
    plane3d plane;
    int result;
    // deal with the eight nod in order
    for(int i=0;i<8;i++) {
        switch(i) {
            case 0: //the first vertex
                tempPt = cb.cnr;
                nrml[0] = dir_x;
                nrml[1] = dir_y;
                nrml[2] = dir_z;
                break;
            case 1: //the second vertex
                tempPt = vecAdd(cb.cnr,scaVecMul(cb.len,dir_x));
                nrml[0] = scaVecMul(-1,dir_x);
                nrml[1] = dir_y;
                nrml[2] = dir_z;
                break;
            case 2: //the third vertex
                tempPt = vecAdd(vecAdd(cb.cnr,scaVecMul(cb.len,dir_x)),scaVecMul(cb.len,dir_y));
                nrml[0] = scaVecMul(-1,dir_x);
                nrml[1] = scaVecMul(-1,dir_y);
                nrml[2] = dir_z;
                break;
            case 3: //the fourth vertex
                tempPt = vecAdd(cb.cnr,scaVecMul(cb.len,dir_y));
                nrml[0] = dir_x;
                nrml[1] = scaVecMul(-1,dir_y);
                nrml[2] = dir_z;
                break;
            case 4: //the fifth vertex
                tempPt = vecAdd(cb.cnr,scaVecMul(cb.len,dir_z));
                nrml[0] = dir_x;
                nrml[1] = dir_y;
                nrml[2] = scaVecMul(-1,dir_z);
                break;
            case 5: //the sixth vertex
                tempPt = vecAdd(vecAdd(cb.cnr,scaVecMul(cb.len,dir_z)),scaVecMul(cb.len,dir_x));
                nrml[0] = scaVecMul(-1,dir_x);
                nrml[1] = dir_y;
                nrml[2] = scaVecMul(-1,dir_z);
                break;
            case 6: //the seventh vertex
                tempPt = vecAdd(vecAdd(vecAdd(cb.cnr,scaVecMul(cb.len,dir_z)),
                        scaVecMul(cb.len,dir_x)),scaVecMul(cb.len,dir_y));
                nrml[0] = scaVecMul(-1,dir_x);
                nrml[1] = scaVecMul(-1,dir_y);
                nrml[2] = scaVecMul(-1,dir_z);
                break;
            case 7: //the eighth vertex
                tempPt = vecAdd(vecAdd(cb.cnr,scaVecMul(cb.len,dir_z)),scaVecMul(cb.len,dir_y));
                nrml[0] = dir_x;
                nrml[1] = scaVecMul(-1,dir_y);
                nrml[2] = scaVecMul(-1,dir_z);
                break;
            default:
                printf("safety purpose.\n");
        }
        nrml[3] = nrmlzVec(vecAdd(vecAdd(nrml[0],nrml[1]),nrml[2]));
        plane.n = nrml[3];
        plane.pt = tempPt;
        result = deterPtPlaneRel(pt,plane);
        if(result == 0) {
            return 0;
        }
    }
    return 1;
}

__host__ __device__ int deterPtCubeRel(const vec3d pt, const aacb3d cube)
{
    vec3d cnr_fru = cube.cnr;
    cnr_fru = vecAdd(cnr_fru,scaVecMul(cube.len,{1,0,0}));
    cnr_fru = vecAdd(cnr_fru,scaVecMul(cube.len,{0,1,0}));
    cnr_fru = vecAdd(cnr_fru,scaVecMul(cube.len,{0,0,1}));
    double x_min = cube.cnr.coords[0], y_min = cube.cnr.coords[1], z_min = cube.cnr.coords[2], 
            x_max = cnr_fru.coords[0], y_max = cnr_fru.coords[1], z_max = cnr_fru.coords[2],
            x = pt.coords[0], y = pt.coords[1], z = pt.coords[2];
    if(x >= x_min && x<= x_max && y >= y_min && y<= y_max && z >= z_min && z<= z_max) {
        return 1;
    } else {
        return 0;
    }
}

__host__ __device__ int deterLinePlaneRel(const line_dbl ln, const plane3d pln, double* t)
{
    if(abs(vecDotMul(ln.dir,pln.n))<EPS) {
        //line parallel to plane
        if(abs(vecDotMul(pln.n,vecSub(ln.pt,pln.pt)))<EPS) {
            return 2;
        } else {
            return 0;
        }
    } else {
        double temp = vecDotMul(pln.n,vecSub(pln.pt,ln.pt))/vecDotMul(pln.n,ln.dir);
        *t = temp;
        return 1;
    }
}

__host__ __device__ double triArea(const tri_dbl s)
{
    vec3d vec[2];
    vec[0] = vecSub(s.nod[1],s.nod[0]);
    vec[1] = vecSub(s.nod[2],s.nod[0]);
    return 0.5*vecNorm(vecCrossMul(vec[0],vec[1]));
}

__host__ __device__ double quadArea(const quad_dbl s)
{
    vec3d vec[2];
    vec[0] = vecSub(s.nod[1],s.nod[0]);
    vec[1] = vecSub(s.nod[2],s.nod[0]);
    return vecNorm(vecCrossMul(vec[0],vec[1]));
}

__host__ __device__ plane3d tri2plane(const tri_dbl tri)
{
    plane3d pln;
    pln.pt = tri.nod[0];
    vec3d vec[2];
    vec[0] = vecSub(tri.nod[1],tri.nod[0]);
    vec[1] = vecSub(tri.nod[2],tri.nod[0]);
    pln.n = nrmlzVec(vecCrossMul(vec[0],vec[1]));
    return pln;
}

__host__ __device__ plane3d quad2plane(const quad_dbl qd)
{
    /*get the plane containing a quad*/
    plane3d pln;
    pln.pt = qd.nod[0];
    vec3d vec[2];
    vec[0] = vecSub(qd.nod[1],qd.nod[0]);
    vec[1] = vecSub(qd.nod[2],qd.nod[0]);
    pln.n = nrmlzVec(vecCrossMul(vec[0],vec[1]));
    return pln;
}

__host__ __device__ line_dbl lnSeg2ln(const lnseg3d ls)
{
    line_dbl l;
    l.pt = ls.nod[0];
    l.dir = vecSub(ls.nod[1],ls.nod[0]);
    return l;
}

__host__ __device__ int deterPtTriRel(const vec3d pt, const tri_dbl tri)
{
    /*determine the relationship between a point and a quad on the same plane
     return: 
     1: pt in tri
     0: pt outsidie tri*/
    double area = 0.0;
    vec3d vec[2];
    for(int i=0;i<3;i++) {
        vec[0] = vecSub(tri.nod[i%3],pt);
        vec[1] = vecSub(tri.nod[(i+1)%3],pt);
        area += 0.5*vecNorm(vecCrossMul(vec[0],vec[1]));
    }
    double area_tri = triArea(tri);
    if(abs(area-area_tri)<EPS) {
        return 1; // in
    } else {
        return 0; // out
    }
}

__host__ __device__ int deterPtQuadRel(const vec3d pt, const quad_dbl qd)
{
    /*determine the relationship between a point and a quad on the same plane
     return: 
     1: pt in qd
     0: pt outside qd*/
    double area = 0.0;
    vec3d vec[2];
    for(int i=0;i<4;i++) {
        vec[0] = vecSub(qd.nod[i%4],pt);
        vec[1] = vecSub(qd.nod[(i+1)%4],pt);
        area += 0.5*vecNorm(vecCrossMul(vec[0],vec[1]));
    }
    double area_quad = quadArea(qd);
    if(abs(area-area_quad)<EPS) {
        return 1; // in
    } else {
        return 0; // out
    }
}

__host__ __device__ double rectCoordDet(const vec3d vec[3])
{
    double result, v1x, v1y, v1z, v2x, v2y, v2z, v3x, v3y, v3z;
    
    v1x = vec[0].coords[0];
    v1y = vec[0].coords[1];
    v1z = vec[0].coords[2];
    
    v2x = vec[1].coords[0];
    v2y = vec[1].coords[1];
    v2z = vec[1].coords[2];
    
    v3x = vec[2].coords[0];
    v3y = vec[2].coords[1];
    v3z = vec[2].coords[2];
    
    result = v1x*(v2y*v3z-v3y*v2z)-v2x*(v1y*v3z-v3y*v1z)+v3x*(v1y*v2z-v2y*v1z);
    
    return result;
}

__host__ __device__ int deterLnLnRel(const line_dbl ln1, const line_dbl ln2, double* t1, double* t2)
{   
    if(abs(vecNorm(vecCrossMul(ln1.dir,ln2.dir)))<EPS) {
        // the two lines are either parallel or the same line
        
        // check if a point on line 1 is on line 2
        vec3d vec = vecSub(ln1.pt,ln2.pt);
        if(vecNorm(vec)<EPS) {
            //the points are the same
            return 2; 
        } 
        else {
            if(vecNorm(vecCrossMul(vec,ln2.dir))<EPS) {
                // vec is a multiple of ln2.dir
                return 2; 
            } 
            else {
                // the two lines are parallel
                return 0;
            }
        }
    } 
    else {
        //the two lines either are skew or intersect
        vec3d pt[4];
        pt[0] = ln1.pt;
        pt[1] = vecAdd(ln1.pt,scaVecMul(1.0,ln1.dir));
        pt[2] = ln2.pt;
        pt[3] = vecAdd(ln2.pt,scaVecMul(1.0,ln2.dir));
        //printVec(pt,4);
        if(vecEqual(pt[0],pt[2]) || vecEqual(pt[0],pt[3]) || 
                vecEqual(pt[1],pt[2]) || vecEqual(pt[1],pt[3])) {
            //the two points on the line is the same point
            if(vecEqual(pt[0],pt[2])) {
                *t1 = 0;
                *t2 = 0;
            } 
            else {
                if(vecEqual(pt[0],pt[3])) {
                    *t1 = 0;
                    *t2 = 1.0;
                } 
                else {
                    if(vecEqual(pt[1],pt[2])) {
                        *t1 = 1.0;
                        *t2 = 0.0;
                    } 
                    else {
                        *t1 = 1.0;
                        *t2 = 1.0;
                    }
                }
            }
            return 1;
        } 
        else {
            //
            vec3d vec[3];
            vec[0] = vecSub(pt[1],pt[0]);
            vec[1] = vecSub(pt[2],pt[0]);
            vec[2] = vecSub(pt[3],pt[0]);
            
            //printf("The determinant is: %f\n",rectCoordDet(vec));
            if(abs(rectCoordDet(vec))>EPS) {
                //skew lines
                return 0;
            } 
            else {
                // the two lines intersects. compute it.
                // first find the valid sub-system
                double coeff1[2], coeff2[2];
                for(int i=0;i<3;i++) {
                    coeff1[0] = ln1.dir.coords[i%3];
                    coeff1[1] = ln1.dir.coords[(i+1)%3];
                    coeff2[0] = ln2.dir.coords[i%3];
                    coeff2[1] = ln2.dir.coords[(i+1)%3];
                    //check the determinant of the current system;
                    double det = coeff1[0]*coeff2[1]-coeff1[1]*coeff2[0];
                    if(abs(det)>EPS) {
                        // get the right-hand side
                        double rhs1[2], rhs2[2];
                        rhs1[0] = ln1.pt.coords[i%3];
                        rhs1[1] = ln1.pt.coords[(i+1)%3];
                        rhs2[0] = ln2.pt.coords[i%3];
                        rhs2[1] = ln2.pt.coords[(i+1)%3];
                        double rhs;
                        rhs = (rhs2[0]-rhs1[0])*coeff2[1]-(rhs2[1]-rhs1[1])*coeff2[0];
                        *t1 = rhs/det;
                        rhs = (rhs2[0]-rhs1[0])*coeff1[1]-(rhs2[1]-rhs1[1])*coeff1[0];
                        *t2 = rhs/det;
                        break;
                    }
                }
                return 1;
            }
        }
        
        
    }
}

__host__ __device__ int deterPtLnRel(const vec3d pt, const line_dbl ln)
{
    /*determines the relation between a point and a line*/
    vec3d vec = vecSub(pt,ln.pt);
    if(vecNorm(vecCrossMul(vec,ln.dir))<EPS) {
        return 1;
    } 
    else {
        return 0;
    }
}

__host__ __device__ int deterPtLnSegRel(const vec3d pt, const lnseg3d lnSeg)
{
    /*determines the relation between a point and a line segment*/
    line_dbl ln = lnSeg2ln(lnSeg);
    if(deterPtLnRel(pt,ln)==0) {
        //point not on the line containing the line segment
        return 0;
    } 
    else {
        double t;
        vec3d vec = vecSub(pt,ln.pt);
        for(int i=0;i<3;i++) {
            if(abs(ln.dir.coords[i])>EPS) {
                t = vec.coords[i]/ln.dir.coords[i];
                break;
            }
        }
        if(t>=0 && t<=1) {
            return 1;
        }
        else {
            return 0;
        }
    }
}

__host__ __device__ int deterLnSegLnSegRel(const lnseg3d seg1, const lnseg3d seg2)
{
    /*determines the relation between two line segments
     seg1: a line segment
     seg2: a line segment
     return: 
     0: no intersection
     1: intersection
     2: infinitely many intersections*/
    line_dbl ln1 = lnSeg2ln(seg1), ln2 = lnSeg2ln(seg2);
    double t1, t2;
    int relLnLn = deterLnLnRel(ln1,ln2,&t1,&t2);
    if(relLnLn==0) {
        // the two lines are skew to each other
        return 0;
    }
    else {
        if(relLnLn==1) {
            // the two lines have one intersection
            if(t1>=0 && t1<=1 && t2>=0 && t2<=1) {
                return 1;
            }
            else {
                return 0;
            }
        }
        else {
            // the two lines are the same line
            if(deterPtLnSegRel(seg1.nod[0],seg2)==0 
                    && deterPtLnSegRel(seg1.nod[1],seg2)==0) {
                // no intersection
                return 0;
            }
            else {
                //determine if one or infinitely many intersection points
                for(int i=0;i<2;i++) {
                    for(int j=0;j<2;j++) {
                        if(vecEqual(seg1.nod[i],seg2.nod[j])) {
                            vec3d vec[2];
                            vec[0] = vecSub(seg1.nod[(i+1)%2],seg1.nod[i]);
                            vec[1] = vecSub(seg2.nod[(j+1)%2],seg1.nod[j]);
                            if(vecDotMul(vec[0],vec[1])<0) {
                                return 1;
                            }
                        }
                    }
                }
                return 2;
            }
        }
    }
}

__host__ __device__ int deterLnSegQuadRel(const lnseg3d lnSeg, const quad_dbl qd)
{
    /*determine if a line segment intersects a quad
     the difference between single intersection and infinitely many intersections 
     is not made.
     0: no intersection
     1: intersection*/
    int flag;
    
    //make a line containing the line segment   
    line_dbl ln = lnSeg2ln(lnSeg);
    
    // define a plane containing the quad
    plane3d pln = quad2plane(qd);
    
    // determine the intersection between the line and the plane
    double t;
    flag = deterLinePlaneRel(ln,pln,&t);
    
    // differentiate between different cases
    if(flag==0) {
        // no intersection between the line and the plane
        return 0;
    } 
    else {
        if(flag==2) {
            // infinitely many intersections between the line and plane
            if(deterPtQuadRel(lnSeg.nod[0],qd)==1 || deterPtQuadRel(lnSeg.nod[1],qd)==1) {
                //oen of the nodes is within the quad
                return 1;
            } 
            else {
                // none of the nodes is within the quad, test if segments intersect
                for(int i=0;i<4;i++) {
                    lnseg3d qdLnSeg;
                    qdLnSeg.nod[0] = qd.nod[i%4];
                    qdLnSeg.nod[1] = qd.nod[(i+1)%4];
                    if(deterLnSegLnSegRel(lnSeg,qdLnSeg)!=0) {
                        // the line segment intersects a quad line segment
                        return 1;
                    }
                }
                return 0;
            }
        }
        else {
            //determines if a point is within a quad
            if(t<0 || t>1) {
                // intersection not on the line segment
                return 0;
            } 
            else {
                vec3d intersection = vecAdd(ln.pt,scaVecMul(t,ln.dir));
                if(deterPtQuadRel(intersection,qd)==1) {
                    // intersection in the quad
                    return 1;
                }
                else {
                    return  0;
                }
            }
            
        }
    }
}

__host__ __device__ int deterLnSegTriRel(const lnseg3d lnSeg, const tri_dbl tri)
{
    /*determine if a line segment intersects a quad
     0: no intersection
     1: intersection*/
    int flag;
    
    //make a line containing the line segment    
    line_dbl ln = lnSeg2ln(lnSeg);
    
    // define a plane containing the triangle
    plane3d pln = tri2plane(tri);
    
    // determine the intersection between the line and the plane
    double t;
    flag = deterLinePlaneRel(ln,pln,&t);
    if(flag==0) {
        // no intersection between the line and the plane
        return 0;
    } 
    else {
        if(flag==2) {
            // infinitely many intersections between the line and plane
            if(deterPtTriRel(lnSeg.nod[0],tri)==1 || deterPtTriRel(lnSeg.nod[1],tri)==1) {
                //oen of the nodes is within the quad
                return 1;
            } 
            else {
                // none of the nodes is within the triangle, test if segments intersect
                for(int i=0;i<3;i++) {
                    lnseg3d triLnSeg;
                    triLnSeg.nod[0] = tri.nod[i%3];
                    triLnSeg.nod[1] = tri.nod[(i+1)%3];
                    if(deterLnSegLnSegRel(lnSeg,triLnSeg)!=0) {
                        // the line segment intersects a trianagle line segment
                        return 1;
                    }
                }
                return 0;
            }
        }
        else {
            //determines if a point is within a quad
            if(t<0 || t>1) {
                // intersection not on the line segment
                return 0;
            } 
            else {
                vec3d intersection = vecAdd(ln.pt,scaVecMul(t,ln.dir));
                if(deterPtTriRel(intersection,tri)==1) {
                    // intersection in the tri
                    return 1;
                }
                else {
                    return  0;
                }
            }
            
        }
    }
}

__host__ __device__ int deterTriCubeInt(const tri_dbl tri, const aacb3d cb)
{
    /*this function determines if a triangle intersects with a cube
     tri: an triangle
     cb: a cube
     return: 
     1: intersection
     0: no intersection*/
    
    //test nodes of the triangle against the cube
    int nodRel[3];
    for(int i=0;i<3;i++) {
        nodRel[i] = deterPtCubeRel(tri.nod[i],cb);
        if(nodRel[i]==1) {
            // node i is in the cube, thus the cube is occupied
            return 1;
        }
    }
    
    //test the intersection between edges of the triangle and the six faces of the cube
    int rel = 0;
    lnseg3d triEdge[3];
    quad_dbl cbFace[6];
    lnseg3d cbDiag[4];
    
    //set up translation vectors
    vec3d dir_x = {1,0,0}, dir_y = {0,1,0}, dir_z = {0,0,1};
    dir_x = scaVecMul(cb.len,dir_x);
    dir_y = scaVecMul(cb.len,dir_y);
    dir_z = scaVecMul(cb.len,dir_z);
    
    //set the edges, faces and diagonals
    for(int i=0;i<3;i++) {
        triEdge[i].nod[0] = tri.nod[i];
        triEdge[i].nod[1] = tri.nod[(i+1)%3];
    }
    
    for(int i=0;i<6;i++) {
        vec3d pt;
        switch(i) {
            case 0: //bottom x-y plane
                pt = cb.cnr;
                cbFace[i].nod[0] = pt;
                cbFace[i].nod[1] = vecAdd(cbFace[i].nod[0],dir_x);
                cbFace[i].nod[2] = vecAdd(cbFace[i].nod[1],dir_y);
                cbFace[i].nod[3] = vecAdd(cbFace[i].nod[2],scaVecMul(-1,dir_x));
                break;
            case 1: //up x-y plane
                pt = vecAdd(pt,dir_z);
                cbFace[i].nod[0] = pt;
                cbFace[i].nod[1] = vecAdd(cbFace[i].nod[0],dir_x);
                cbFace[i].nod[2] = vecAdd(cbFace[i].nod[1],dir_y);
                cbFace[i].nod[3] = vecAdd(cbFace[i].nod[2],scaVecMul(-1,dir_x));
                break;
            case 2: //left y-z plane
                pt = cb.cnr;
                cbFace[i].nod[0] = pt;
                cbFace[i].nod[1] = vecAdd(cbFace[i].nod[0],dir_x);
                cbFace[i].nod[2] = vecAdd(cbFace[i].nod[1],dir_z);
                cbFace[i].nod[3] = vecAdd(cbFace[i].nod[2],scaVecMul(-1,dir_x));
                break;
            case 3: //right y-z plane
                pt = vecAdd(cb.cnr,dir_y);
                cbFace[i].nod[0] = pt;
                cbFace[i].nod[1] = vecAdd(cbFace[i].nod[0],dir_x);
                cbFace[i].nod[2] = vecAdd(cbFace[i].nod[1],dir_z);
                cbFace[i].nod[3] = vecAdd(cbFace[i].nod[2],scaVecMul(-1,dir_x));
                break;
            case 4: //back z-x plane
                pt = cb.cnr;
                cbFace[i].nod[0] = pt;
                cbFace[i].nod[1] = vecAdd(cbFace[i].nod[0],dir_y);
                cbFace[i].nod[2] = vecAdd(cbFace[i].nod[1],dir_z);
                cbFace[i].nod[3] = vecAdd(cbFace[i].nod[2],scaVecMul(-1,dir_y));
                break;
            case 5: //front z-x plane
                pt = vecAdd(cb.cnr,dir_x);
                cbFace[i].nod[0] = pt;
                cbFace[i].nod[1] = vecAdd(cbFace[i].nod[0],dir_y);
                cbFace[i].nod[2] = vecAdd(cbFace[i].nod[1],dir_z);
                cbFace[i].nod[3] = vecAdd(cbFace[i].nod[2],scaVecMul(-1,dir_y));
                break;
            default:
                printf("Should not enter this.\n");
        }
    }
    
    // first diagnonal
    cbDiag[0].nod[0] = cb.cnr;
    cbDiag[0].nod[1] = vecAdd(vecAdd(vecAdd(cbDiag[0].nod[0],dir_x),dir_y),dir_z);
    
    // second diagnonal
    cbDiag[1].nod[0] = vecAdd(cbDiag[0].nod[0],dir_x);
    cbDiag[1].nod[1] = vecAdd(vecAdd(vecAdd(cbDiag[1].nod[0],dir_y),scaVecMul(-1,dir_x)),dir_z);
    
    // third diagnoal
    cbDiag[2].nod[0] = vecAdd(cbDiag[1].nod[0],dir_y);
    cbDiag[2].nod[1] = vecAdd(vecAdd(vecAdd(cbDiag[2].nod[0],scaVecMul(-1,dir_x)),scaVecMul(-1,dir_y)),dir_z);
    
    // fourth diagonal
    cbDiag[3].nod[0] = vecAdd(cbDiag[2].nod[0],scaVecMul(-1,dir_x));
    cbDiag[3].nod[1] = vecAdd(vecAdd(vecAdd(cbDiag[3].nod[0],scaVecMul(-1,dir_y)),dir_x),dir_z);
    
    // determine if any of the three edges of the triangle intersects the faces;
    //printf("Entered diagonal test.\n");
    for(int i=0;i<3;i++) {
        for(int j=0;j<6;j++) {
            rel = deterLnSegQuadRel(triEdge[i],cbFace[j]);
            if(rel==1) {
                return 1;
            }
        }
    }
    
    for(int i=0;i<4;i++) {
        rel = deterLnSegTriRel(cbDiag[i],tri);
        if(rel==1) {
            return 1;
        }
    }
    
    return 0;
}

__global__ void testTriCbInt(const tri_dbl* tri, const int numTri, const aacb3d* cb, 
        const int numCb, int* flag)
{
    /*the global function for testing triangle-cube intersection
     tri: an array of triangles
     numTri: the number of triangles
     cb: an array of cubes
     numCb: the number of cubes
     flag: an array of flags, initialized to zero, of size numCb*/
    int idx_x = blockIdx.x*blockDim.x+threadIdx.x; // triangle index
    int idx_y = blockIdx.y*blockDim.y+threadIdx.y; // cube index
    
    if(idx_x < numTri && idx_y < numCb) {
        if(idx_x==0 && idx_y==0) {
            printf("the first node. cube: (%lf,%lf,%lf),%lf\n",cb[idx_y].cnr.coords[0],
                    cb[idx_y].cnr.coords[1],cb[idx_y].cnr.coords[2],cb[idx_y].len);
        }
        int rel = deterTriCubeInt(tri[idx_x],cb[idx_y]);
        if(idx_x==0 && idx_y==0) {
            printf("Completed determination.\n");
        }
        atomicAdd(&flag[idx_y],rel);
    }
}

__host__ int getTriCbRel(const tri_dbl* tri, const int numTri, const aacb3d* cb, 
        const int numCb, int* flag)
{
    /*voxelize a space into occupance grids
     tri: an array of triangles
     numTri: the number of triangles
     cb: an array of cubes
     numCb: number of cubes
     flag: an array of flags for cube occupancy*/
    printf("Entered getTriCbRel.\n");
    tri_dbl *tri_d;
    CUDA_CALL(hipMalloc(&tri_d,numTri*sizeof(tri_dbl)));
    CUDA_CALL(hipMemcpy(tri_d,tri,numTri*sizeof(tri_dbl),hipMemcpyHostToDevice));
    printf("Allocated and copied memory for triangles\n");
    
    aacb3d *cb_d;
    CUDA_CALL(hipMalloc(&cb_d,numCb*sizeof(aacb3d)));
    CUDA_CALL(hipMemcpy(cb_d,cb,numCb*sizeof(aacb3d),hipMemcpyHostToDevice));
    printf("Allocated and copied memory for cubes\n");
    
    memset(flag,0,numCb*sizeof(int));
    printf("Initialized flag\n");
    int *flag_d;
    CUDA_CALL(hipMalloc(&flag_d,numCb*sizeof(int)));
    CUDA_CALL(hipMemcpy(flag_d,flag,numCb*sizeof(int),hipMemcpyHostToDevice));
    
    printf("Device memory allocated.\n");
    
    int xNumBlocks, xWidth = 1, yNumBlocks, yWidth = 1;
    xNumBlocks = (numTri+xWidth-1)/xWidth;
    yNumBlocks = (numCb+yWidth-1)/yWidth;
    
    dim3 gridLayout, blockLayout;
    gridLayout.x = xNumBlocks;
    gridLayout.y = yNumBlocks;
    
    blockLayout.x = xWidth;
    blockLayout.y = yWidth;
    
    testTriCbInt<<<gridLayout,blockLayout>>>(tri_d,numTri,cb_d,numCb,flag_d);
    HOST_CALL(hipMemcpy(flag,flag_d,numCb*sizeof(int),hipMemcpyDeviceToHost));
    
    CUDA_CALL(hipFree(flag_d));
    CUDA_CALL(hipFree(cb_d));
    CUDA_CALL(hipFree(tri_d));
    
    for(int i=0;i<numCb;i++) {
        if(flag[i]!=0) {
            flag[i] = 1;
        }
    }
    
    return EXIT_SUCCESS;
}

void reorgGrid_zyx2xyz(int* grid, const int l)
{
    /*re-organize the voxel grids from the order of significance of z, y, x to x, y, z*/
    int totalNum = pow(8,l), dimNum = pow(2,l);
    int *temp = (int*)malloc(totalNum*sizeof(int));
    memcpy(temp,grid,totalNum*sizeof(int));
    
    // reorganize
    for(int x=0;x<dimNum;x++) {
        for(int y=0;y<dimNum;y++) {
            for(int z=0;z<dimNum;z++) {
                int idx_old = x*dimNum*dimNum+y*dimNum+z;
                int idx_new = z*dimNum*dimNum+y*dimNum+x;
                grid[idx_new] = temp[idx_old];
            }
        }
    }
    free(temp);
}

__host__ __device__ void printCube(const aacb3d cb)
{
    printf("corner: (%lf,%lf,%lf), length: %lf\n",cb.cnr.coords[0],cb.cnr.coords[1],
            cb.cnr.coords[2],cb.len);
}

__host__ __device__ void printTriangle(const tri_dbl tri)
{
    printf("nodes: (%lf,%lf,%lf), (%lf,%lf,%lf), (%lf,%lf,%lf)\n",
            tri.nod[0].coords[0],tri.nod[0].coords[1],tri.nod[0].coords[2],
            tri.nod[1].coords[0],tri.nod[1].coords[1],tri.nod[1].coords[2],
            tri.nod[2].coords[0],tri.nod[2].coords[1],tri.nod[2].coords[2]);
}

__host__ __device__ void PrintVec(const vec2d* vec, const int num)
{
    for(int i=0;i<num;i++) {
        printf("(%lf,%lf)\n",vec[i].coords[0],vec[i].coords[1]);
    }
}

__host__ __device__ void PrintVec(const vec2f* vec, const int num)
{
    for(int i=0;i<num;i++) {
        printf("(%f,%f)\n",vec[i].coords[0],vec[i].coords[1]);
    }
}

__host__ int voxelSpace(const aacb3d sp, const int numEachDim, const vec3d* pt, 
        const tri_elem* elem, const int numElem, int* flag)
{
    /*voxelize the a space of objects composed of triangles
     sp: a cube representing the whole space
     pt: an array of points
     numPt: the number of points
     elem: an array of elements
     numElem: the number of elements
     flag: an array of flags
     octLevel: the level of the octree*/
    
    printf("Entered voxSpace.\n");
    // save all the triangles in a triangle array
    tri_dbl *tri = (tri_dbl*)malloc(numElem*sizeof(tri_dbl));
    for(int i=0;i<numElem;i++) {
        for(int j=0;j<3;j++) {
            tri[i].nod[j] = pt[elem[i].nod[j]];
        }
    }
    printf("Initialized triangles.\n");
    //for(int i=0;i<numElem;i++) {
    //    printf("Current triangle: (%lf,%lf,%f), (%lf,%lf,%lf), (%lf,%lf,%f)\n",
    //            tri[i].nod[0].coords[0],tri[i].nod[0].coords[1],tri[i].nod[0].coords[2],
    //            tri[i].nod[1].coords[0],tri[i].nod[1].coords[1],tri[i].nod[1].coords[2],
    //            tri[i].nod[2].coords[0],tri[i].nod[2].coords[1],tri[i].nod[2].coords[2]);
    //}
    // save all the unit boxes in a cube array
    int numVox = numEachDim*numEachDim*numEachDim;
    memset(flag,0,numVox*sizeof(int));
    
    aacb3d *cb = (aacb3d*)malloc(numVox*sizeof(aacb3d));
    double unitLen = sp.len/numEachDim;
    vec3d dir_x = {unitLen,0,0}, dir_y = {0,unitLen,0}, dir_z = {0,0,unitLen}, 
            xOffset, yOffset, zOffset;
    int idx, rel;
    for(int i=0;i<numEachDim;i++) {
        // z dimension
        zOffset = scaVecMul(i,dir_z);
        for(int j=0;j<numEachDim;j++) {
            // y dimension
            yOffset = scaVecMul(j,dir_y);
            for(int k=0;k<numEachDim;k++) {
                // x dimension
                xOffset = scaVecMul(k,dir_x);
                idx = i*(numEachDim*numEachDim)+j*numEachDim+k;
                cb[idx].cnr = vecAdd(vecAdd(vecAdd(sp.cnr,xOffset),yOffset),zOffset);
                cb[idx].len = unitLen;
            }
        }
    }
    for(int i=0;i<numVox;i++) {
        for(int j=0;j<numElem;j++) {
            rel = deterTriCubeInt(tri[j],cb[i]);
            if(rel==1) {
                flag[i] = 1;
                break;
            }
        }
    }
    free(cb);
    free(tri);
    return EXIT_SUCCESS;
}

__host__ int write_voxels(const int* flag, const int numEachDim, const char* file_path)
{
    FILE *file = fopen(file_path,"w");
    if(file==NULL) {
        printf("Failed to open file.\n");
        return EXIT_FAILURE;
    }
    else {
        int status;
        for(int i=0;i<numEachDim*numEachDim*numEachDim;i++) {
            status = fprintf(file,"%d ",flag[i]);
            if((i+1)%numEachDim == 0) {
                status = fprintf(file,"\n");
            }
            if((i+1)%(numEachDim*numEachDim) == 0) {
                status = fprintf(file,"\n");
            }
            if(status<0) {
                printf("Failed to write the %dth line to file\n",i);
                return EXIT_FAILURE;
            }
        }
        fclose(file);
        return EXIT_SUCCESS;
    }
}

__host__ __device__ vec2d GetMin(const aarect2d rect)
{
    return rect.cnr;
}

__host__ __device__ vec2d GetMax(const aarect2d rect)
{
    vec2d dir_x = {1,0}, dir_y = {0,1};
    vec2d nod = vecAdd(vecAdd(rect.cnr,scaVecMul(rect.len[0],dir_x)),scaVecMul(rect.len[1],dir_y));
    return nod;
}

__host__ __device__ bool IntvlIntvlOvlp(const intvl2d intvl1, const intvl2d intvl2)
{
    /*returns true if the two intervals overlap and false if not*/
    if(intvl1.min<=intvl2.max && intvl2.min<=intvl1.max) {
        return true;
    }
    else {
        return false;
    }
}

__host__ __device__ bool AaRectAaRectOvlp(const aarect2d rect1, const aarect2d rect2)
{
    /*determines if two axis-aligned rectangles overlap
     rect1: the first rectangle
     rect2: the second rectangle
     return: 
     true: the two rectangles overlap
     false: the two rectangles do not overlap*/
    // first check if the projections on the x axis overlap
    vec2d minNod1 = GetMin(rect1), maxNod1 = GetMax(rect1), minNod2 = GetMin(rect2), 
            maxNod2 = GetMax(rect2);
    intvl2d intvl1x = {minNod1.coords[0],maxNod1.coords[0]}, intvl2x = {minNod2.coords[0],maxNod2.coords[0]},
            intvl1y = {minNod1.coords[1],maxNod1.coords[1]}, intvl2y = {minNod2.coords[1],maxNod2.coords[1]};
    
    if(IntvlIntvlOvlp(intvl1x,intvl2x) && IntvlIntvlOvlp(intvl1y,intvl2y)) {
        return true;
    }
    else {
        return false;
    }
}

__host__ __device__ vec3d GetMin(const aarect3d& rect)
{
    return rect.cnr;
}

#ifdef __CUDA_ARCH__

vec3d GetMax(const aarect3d& rect)
{
    vec3d cnr_max = rect.cnr;
    for(int i=0;i<3;i++) {
        cnr_max = vecAdd(cnr_max,scaVecMul(rect.len[i],BASES[i]));
    }
    return cnr_max;
}

#else

vec3d GetMax(const aarect3d& rect)
{   
    vec3d cnr_max = rect.cnr;
    for(int i=0;i<3;i++) {
        cnr_max = vecAdd(cnr_max,scaVecMul(rect.len[i],bases[i]));
    }
    return cnr_max;
}

#endif

__host__ __device__ intvl3d GetInterval(const aarect3d& rect, const vec3d& axis)
{
    vec3d cnrs[2], vertex;
    cnrs[0] = GetMin(rect);
    cnrs[1] = GetMax(rect);
    intvl3d intvl;
    double projection;
    intvl.max = -DBL_MAX;
    intvl.min = DBL_MAX;
    for(int i=0;i<2;i++) {
        vertex.coords[0] = cnrs[i].coords[0];
        for(int j=0;j<2;j++) {
            vertex.coords[1] = cnrs[j].coords[1];
            for(int k=0;k<2;k++) {
                vertex.coords[2] = cnrs[k].coords[2];
                //printVec(&vertex,1);
                projection = vecDotMul(vertex,axis);
                intvl.max = (projection>intvl.max) ? projection : intvl.max;
                intvl.min = (projection<intvl.min) ? projection : intvl.min;
            }
        }
    }
    return intvl;
}

__host__ __device__ bool IntvlIntvlOvlp(const intvl3d& intvl1, const intvl3d& intvl2)
{
    /*returns true if the two intervals overlap and false if not*/
    if(intvl1.min<=intvl2.max && intvl2.min<=intvl1.max) {
        return true;
    }
    else {
        return false;
    }
}

__host__ __device__ intvl3d GetInterval(const tri3d& tri, const vec3d& ax)
{
    intvl3d intvl;
    intvl.min = DBL_MAX;
    intvl.max = -DBL_MAX;
    double projection;
    
    for(int i=0;i<3;i++) {
        projection = vecDotMul(tri.nod[i],ax);
        intvl.max = (projection>intvl.max) ? projection : intvl.max;
        intvl.min = (projection<intvl.min) ? projection : intvl.min;
    }
    
    return intvl;
}

__host__ __device__ bool OverlapOnAxis(const tri3d& tri, const aarect3d& rect, const vec3d& ax)
{
    intvl3d intvl_tri, intvl_rect;
    intvl_tri = GetInterval(tri,ax);
    intvl_rect = GetInterval(rect,ax);
    
    return IntvlIntvlOvlp(intvl_tri,intvl_rect);
}

__host__ __device__ bool OverlapTriangleAARect(const tri3d& tri, const aarect3d& rect)
{
    return true;
}